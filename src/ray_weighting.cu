#include "hip/hip_runtime.h"
#include "ray_weighting.cuh"

#include "hip/hip_runtime.h"
#include "cuda_utils.h"

#include <stdlib.h>
#include <math.h>
#include <stdio.h>

__global__ void convertARTtoERTkernel(float* g, const float muCoeff, const float muRadius, const float T_u, const float u_0, int3 N)
{
	const int i = threadIdx.x + blockIdx.x * blockDim.x;
	const int j = threadIdx.y + blockIdx.y * blockDim.y;
	const int k = threadIdx.z + blockIdx.z * blockDim.z;
	if (i >= N.x || j >= N.y || k >= N.z)
		return;

	//weight = np.sqrt(np.clip(150.0 * *2 - u * *2, 0.0, 150.0 * *2))
	//weight = np.exp(muCoeff * weight)
	const float u = T_u * k + u_0;
	if (fabs(u) < muRadius)
		g[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] *= expf(muCoeff * sqrt(muRadius*muRadius - u*u));
}

__global__ void applyWeightsKernel(float* g, const float* w_view, const float* w_ray, int3 N)
{
	const int i = threadIdx.x + blockIdx.x * blockDim.x;
	const int j = threadIdx.y + blockIdx.y * blockDim.y;
	const int k = threadIdx.z + blockIdx.z * blockDim.z;
	if (i >= N.x || j >= N.y || k >= N.z)
		return;

	float theWeight = 1.0f;
	if (w_ray != NULL)
		theWeight *= w_ray[j * N.z + k];
	if (w_view != NULL)
		theWeight *= w_view[i * N.z + k];
	g[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] *= theWeight;
}

float FBPscalar(parameters* params)
{
	float magFactor = params->sod / params->sdd;
	if (params->geometry == parameters::CONE)
		return 1.0 / (2.0 * PI) * fabs(params->T_phi() * params->pixelWidth * magFactor * params->pixelHeight * magFactor / (params->voxelWidth * params->voxelWidth * params->voxelHeight));
	else if (params->geometry == parameters::FAN)
		return 1.0 / (2.0 * PI) * fabs(params->T_phi() * params->pixelWidth * magFactor * params->pixelHeight / (params->voxelWidth * params->voxelWidth * params->voxelHeight));
	else
		return 1.0 / (2.0 * PI) * fabs(params->T_phi() * params->pixelWidth / (params->voxelWidth * params->voxelWidth));
}

float* setViewWeights(parameters* params)
{
	float* w = setParkerWeights(params);
	return setRedundantAndNonEquispacedViewWeights(params, w);
}

float* setParkerWeights(parameters* params)
{
	if (params->angularRange >= 359.9999 || params->numAngles == 1)
		return NULL;

	if (params->geometry == parameters::CONE || params->geometry == parameters::FAN)
	{
		bool normalizeConeAndFanCoordinateFunctions_save = params->normalizeConeAndFanCoordinateFunctions;
		params->normalizeConeAndFanCoordinateFunctions = true;
		float* retVal = (float*)malloc(sizeof(float) * params->numAngles * params->numCols);

		double beta_max = params->angularRange*PI / 180.0;

		double gamma = asin(params->tau / sqrt(params->sod * params->sod + params->tau * params->tau));
		double alpha_max;
		if (params->detectorType == parameters::FLAT)
			alpha_max = min(fabs(atan(params->u(0)) - gamma), fabs(atan(params->u(params->numCols - 1)) - gamma));
		else
			alpha_max = min(fabs(params->u(0) - gamma), fabs(params->u(params->numCols - 1) - gamma));

		double shortScanThreshold = PI + 2.0 * alpha_max;

		if (beta_max < shortScanThreshold)
			printf("FDK::calcParkerWeights: Not enough data!\n");

		double thres = (beta_max - PI) / 2.0;
		if (thres < 0.0)
			thres = 0.0;
		double beta, alpha, theWeight;
		double alpha_c;

		double plus_or_minus = 1.0;
		if (params->T_phi() < 0.0)
			plus_or_minus = -1.0;
		//plus_or_minus = 1.0;
		for (int j = 0; j < params->numCols; j++)
		{
			if (params->detectorType == parameters::FLAT)
				alpha = plus_or_minus * atan(params->u(j)) - gamma;
			else
				alpha = plus_or_minus * params->u(j) - gamma;

			alpha_c = -alpha;

			for (int i = 0; i < params->numAngles; i++)
			{
				beta = fabs(params->phis[i] - params->phis[0]);
				//beta = fabs(g->T_phi)*double(i); //g->phi(i) - g->phi_0;
				if (beta < 2.0 * (thres + alpha))
				{
					theWeight = sin(PI / 4.0 * beta / (thres + alpha));
					theWeight = theWeight * theWeight;
				}
				else if (beta < PI + alpha - alpha_c)
					theWeight = 1.0;
				else if (beta < PI + 2.0 * thres)
				{
					theWeight = cos(PI / 4.0 * (beta - alpha + alpha_c - PI) / (thres + alpha_c));
					theWeight = theWeight * theWeight;
				}
				else
					theWeight = 0.0;
				if (theWeight < 1e-8)
					theWeight = 1e-8;
				retVal[i*params->numCols+j] = theWeight;
			}
		}

		params->normalizeConeAndFanCoordinateFunctions = normalizeConeAndFanCoordinateFunctions_save;

		return retVal;
	}
	else if (params->geometry == parameters::PARALLEL)
	{
		float T_phi = params->T_phi();
		float* retVal = (float*)malloc(sizeof(float) * params->numAngles * params->numCols);
		for (int i = 0; i < params->numAngles; i++)
		{
			double theWeight = 0.0;

			double view_1 = fabs(T_phi) * double(i);
			if (view_1 > PI)
				view_1 -= PI;

			for (int j = 0; j < params->numAngles; j++)
			{
				double view_2 = fabs(T_phi) * double(j);

				double viewOffset = min(fabs(view_1 - view_2), fabs(view_1 - (view_2 - PI)));
				viewOffset = min(viewOffset, fabs((view_1 - PI) - view_2));
				if (fabs(viewOffset) < fabs(T_phi))
					theWeight += min(0.5, viewOffset / fabs(T_phi) + 0.5) - max(-0.5, viewOffset / fabs(T_phi) - 0.5);
			}
			theWeight = 1.0 / theWeight;

			for (int j = 0; j < params->numCols; j++)
				retVal[i*params->numCols+j] = theWeight;
		}
		return retVal;
	}
	else
		return NULL;
}

float* setRedundantAndNonEquispacedViewWeights(parameters* params, float* w)
{
	float* retVal = w;
	if (retVal == NULL)
	{
		retVal = (float*)malloc(sizeof(float) * params->numAngles * params->numCols);
		for (int i = 0; i < params->numAngles * params->numCols; i++) retVal[i] = 1.0;
	}
	if (params->numAngles < 2)
		return retVal;

	// First modify weight in cases where we have non-equispaced angles
	float T_phi = fabs(params->T_phi());
	for (int i = 0; i < params->numAngles; i++)
	{
		float theWeight = 1.0;
		if (i == 0)
		{
			theWeight = fabs(params->phis[1] - params->phis[0]) / T_phi;
		}
		else if (i == params->numAngles - 1)
		{
			theWeight = fabs(params->phis[params->numAngles - 1] - params->phis[params->numAngles - 2]) / T_phi;
		}
		else
		{
			theWeight = 0.5 * (fabs(params->phis[i + 1] - params->phis[i]) + fabs(params->phis[i] - params->phis[i - 1])) / T_phi;
		}
		for (int j = 0; j < params->numCols; j++)
			retVal[i * params->numCols + j] *= theWeight;
	}

	// Now apply weights for cases where we have redundant measurements
	if ((params->geometry == parameters::PARALLEL && params->angularRange >= 359.9999) || 
		(params->geometry == parameters::FAN && params->angularRange >= 359.9999) ||
		(params->geometry == parameters::CONE && params->angularRange >= 359.9999))
	{
		float c = 0.5;
		//float c = 1.0;
		//if (params->geometry == parameters::FAN || params->geometry == parameters::CONE)
		//	c = 0.5;
		float T = fabs(params->T_phi());
		for (int i = 0; i < params->numAngles; i++)
		{
			float viewWeight = 0.0;
			for (int j = 0; j < params->numAngles; j++)
			{
				double viewOffset = atan2(sin(double(j - i) * T), cos(double(j - i) * T)); // signed angular distance
				if (fabs(viewOffset) < T)
					viewWeight += min(0.5, viewOffset / T + 0.5) - max(-0.5, viewOffset / T - 0.5);
			}
			//printf("%f\n", viewWeight);
			viewWeight = 1.0 / viewWeight;
			for (int j = 0; j < params->numCols; j++)
				retVal[i * params->numCols + j] *= c * viewWeight;
		}
	}
	return retVal;
}

float* setInverseConeWeight(parameters* params)
{
	if (params->geometry == parameters::CONE)
	{
		bool normalizeConeAndFanCoordinateFunctions_save = params->normalizeConeAndFanCoordinateFunctions;
		params->normalizeConeAndFanCoordinateFunctions = true;

		float* retVal = (float*)malloc(sizeof(float) * params->numRows * params->numCols);
		for (int iv = 0; iv < params->numRows; iv++)
		{
			float v = params->v(iv);
			for (int iu = 0; iu < params->numCols; iu++)
			{
				float u = params->u(iu);
				if (params->detectorType == parameters::FLAT)
					retVal[iv * params->numCols + iu] = 1.0 / sqrt(1.0 + u * u + v * v);
				else
					retVal[iv * params->numCols + iu] = 1.0 / sqrt(1.0 + v * v);
			}
		}
		params->normalizeConeAndFanCoordinateFunctions = normalizeConeAndFanCoordinateFunctions_save;

		return retVal;
	}
	else if (params->geometry == parameters::FAN)
	{
		bool normalizeConeAndFanCoordinateFunctions_save = params->normalizeConeAndFanCoordinateFunctions;
		params->normalizeConeAndFanCoordinateFunctions = true;

		float* retVal = (float*)malloc(sizeof(float) * params->numRows * params->numCols);
		for (int iv = 0; iv < params->numRows; iv++)
		{
			for (int iu = 0; iu < params->numCols; iu++)
			{
				float u = params->u(iu);
				if (params->detectorType == parameters::FLAT)
					retVal[iv * params->numCols + iu] = 1.0 / sqrt(1.0 + u * u);
				else
					retVal[iv * params->numCols + iu] = 1.0;
			}
		}
		params->normalizeConeAndFanCoordinateFunctions = normalizeConeAndFanCoordinateFunctions_save;

		return retVal;
	}
	else
		return NULL;
}

float* setPreRampFilterWeights(parameters* params)
{
	float* w = setInverseConeWeight(params); // numRows X numCols
	if (w != NULL && (params->geometry == parameters::CONE || params->geometry == parameters::FAN))
	{
		bool normalizeConeAndFanCoordinateFunctions_save = params->normalizeConeAndFanCoordinateFunctions;
		params->normalizeConeAndFanCoordinateFunctions = true;
		for (int iv = 0; iv < params->numRows; iv++)
		{
			for (int iu = 0; iu < params->numCols; iu++)
			{
				float u = params->u(iu);
				if (params->detectorType == parameters::FLAT)
					w[iv * params->numCols + iu] *= (1.0 + params->tau / params->sod * u);
				else
					w[iv * params->numCols + iu] *= (cos(u) + params->tau / params->sod * sin(u));
			}
		}
		params->normalizeConeAndFanCoordinateFunctions = normalizeConeAndFanCoordinateFunctions_save;
	}
	return w;
}

bool applyPreRampFilterWeights_CPU(float* g, parameters* params)
{
	float* w = setPreRampFilterWeights(params);
	float* w_view = setViewWeights(params); // numAngles X numCols

	if (w == NULL && w_view == NULL)
		return true;

	for (int iphi = 0; iphi < params->numAngles; iphi++)
	{
		for (int iv = 0; iv < params->numRows; iv++)
		{
			for (int iu = 0; iu < params->numCols; iu++)
			{
				float theWeight = 1.0;
				if (w != NULL)
					theWeight *= w[iv * params->numCols + iu];
				if (w_view != NULL)
					theWeight *= w_view[iphi * params->numCols + iu];
				g[uint64(iphi) * uint64(params->numRows * params->numCols) + uint64(iv * params->numCols + iu)] *= theWeight;
			}
		}
	}
	if (w != NULL)
		free(w);
	if (w_view != NULL)
		free(w_view);
	return true;
}

bool applyPostRampFilterWeights_CPU(float* g, parameters* params)
{
	float* w = setInverseConeWeight(params); // numRows X numCols
	if (w == NULL)
		return true;
	else
	{
		for (int iphi = 0; iphi < params->numAngles; iphi++)
		{
			for (int iv = 0; iv < params->numRows; iv++)
			{
				for (int iu = 0; iu < params->numCols; iu++)
				{
					float theWeight = 1.0;
					if (w != NULL)
						theWeight *= w[iv * params->numCols + iu];
					g[uint64(iphi) * uint64(params->numRows * params->numCols) + uint64(iv * params->numCols + iu)] *= theWeight;
				}
			}
		}
		free(w);
		return true;
	}
}

bool applyPreRampFilterWeights_GPU(float* g, parameters* params, bool cpu_to_gpu)
{
	float* w_ray = setPreRampFilterWeights(params);
	float* w_view = setViewWeights(params); // numAngles X numCols

	if (w_ray == NULL && w_view == NULL)
		return true;
	else
	{
		hipSetDevice(params->whichGPU);
		hipError_t cudaStatus;

		int3 N = make_int3(params->numAngles, params->numRows, params->numCols);
		float* dev_g = 0;
		if (cpu_to_gpu)
		{
			dev_g = copyProjectionDataToGPU(g, params, params->whichGPU);
		}
		else
		{
			dev_g = g;
		}

		float* dev_w_view = 0;
		if (w_view != NULL)
		{
			if (hipSuccess != hipMalloc((void**)&dev_w_view, params->numAngles * params->numCols * sizeof(float)))
				fprintf(stderr, "hipMalloc failed!\n");
			if (hipMemcpy(dev_w_view, w_view, params->numAngles * params->numCols * sizeof(float), hipMemcpyHostToDevice))
				fprintf(stderr, "hipMemcpy failed!\n");
		}
		float* dev_w_ray = 0;
		if (w_ray != NULL)
		{
			if (hipSuccess != hipMalloc((void**)&dev_w_ray, params->numRows * params->numCols * sizeof(float)))
				fprintf(stderr, "hipMalloc failed!\n");
			if (hipMemcpy(dev_w_ray, w_ray, params->numRows * params->numCols * sizeof(float), hipMemcpyHostToDevice))
				fprintf(stderr, "hipMemcpy failed!\n");
		}

		dim3 dimBlock = setBlockSize(N);
		dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
			int(ceil(double(N.z) / double(dimBlock.z))));
		applyWeightsKernel <<< dimGrid, dimBlock >>> (dev_g, dev_w_view, dev_w_ray, N);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "kernel failed!\n");
			fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
			fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
		}

		if (cpu_to_gpu)
		{
			//printf("pulling projections off GPU...\n");
			pullProjectionDataFromGPU(g, params, dev_g, params->whichGPU);
		}

		if (dev_w_view != 0)
			hipFree(dev_w_view);
		if (dev_w_ray != 0)
			hipFree(dev_w_ray);
		if (w_ray != NULL)
			free(w_ray);
		if (w_view != NULL)
			free(w_view);
		if (cpu_to_gpu == true && dev_g != 0)
			hipFree(dev_g);

		return true;
	}
}

bool applyPostRampFilterWeights_GPU(float* g, parameters* params, bool cpu_to_gpu)
{
	float* w_ray = setInverseConeWeight(params); // numRows X numCols
	if (w_ray == NULL)
		return true;
	else
	{
		hipSetDevice(params->whichGPU);
		hipError_t cudaStatus;

		int3 N = make_int3(params->numAngles, params->numRows, params->numCols);
		float* dev_g = 0;
		if (cpu_to_gpu)
		{
			dev_g = copyProjectionDataToGPU(g, params, params->whichGPU);
		}
		else
		{
			dev_g = g;
		}

		float* dev_w_view = 0;
		float* dev_w_ray = 0;
		if (w_ray != NULL)
		{
			if (hipSuccess != hipMalloc((void**)&dev_w_ray, params->numRows * params->numCols * sizeof(float)))
				fprintf(stderr, "hipMalloc failed!\n");
			if (hipMemcpy(dev_w_ray, w_ray, params->numRows * params->numCols * sizeof(float), hipMemcpyHostToDevice))
				fprintf(stderr, "hipMemcpy failed!\n");
		}

		dim3 dimBlock = setBlockSize(N);
		dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
			int(ceil(double(N.z) / double(dimBlock.z))));
		applyWeightsKernel <<< dimGrid, dimBlock >>> (dev_g, dev_w_view, dev_w_ray, N);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "kernel failed!\n");
			fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
			fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
		}

		if (cpu_to_gpu)
		{
			//printf("pulling projections off GPU...\n");
			pullProjectionDataFromGPU(g, params, dev_g, params->whichGPU);
		}

		if (dev_w_ray != 0)
			hipFree(dev_w_ray);
		if (w_ray != NULL)
			free(w_ray);
		if (cpu_to_gpu == true && dev_g != 0)
			hipFree(dev_g);

		return true;
	}
}

bool applyPreRampFilterWeights(float* g, parameters* params, bool cpu_to_gpu)
{
	if (params->whichGPU < 0)
		return applyPreRampFilterWeights_CPU(g, params);
	else
		return applyPreRampFilterWeights_GPU(g, params, cpu_to_gpu);
}

bool applyPostRampFilterWeights(float* g, parameters* params, bool cpu_to_gpu)
{
	if (params->whichGPU < 0)
		return applyPostRampFilterWeights_CPU(g, params);
	else
		return applyPostRampFilterWeights_GPU(g, params, cpu_to_gpu);
}

bool convertARTtoERT(float* g, parameters* params, bool cpu_to_gpu, bool doInverse)
{
	if (params->whichGPU < 0)
		return convertARTtoERT_CPU(g, params, doInverse);
	else
	{
		hipSetDevice(params->whichGPU);
		hipError_t cudaStatus;

		int3 N = make_int3(params->numAngles, params->numRows, params->numCols);
		float* dev_g = 0;
		if (cpu_to_gpu)
			dev_g = copyProjectionDataToGPU(g, params, params->whichGPU);
		else
			dev_g = g;

		float muCoeff = params->muCoeff;
		if (doInverse)
			muCoeff *= -1.0;

		dim3 dimBlock = setBlockSize(N);
		dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
			int(ceil(double(N.z) / double(dimBlock.z))));
		convertARTtoERTkernel <<< dimGrid, dimBlock >>> (dev_g, muCoeff, params->muRadius, params->pixelWidth, params->u_0(), N);

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "kernel failed!\n");
			fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
			fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
		}

		if (cpu_to_gpu)
			pullProjectionDataFromGPU(g, params, dev_g, params->whichGPU);

		if (cpu_to_gpu == true && dev_g != 0)
			hipFree(dev_g);

		return true;
	}
}

bool convertARTtoERT_CPU(float* g, parameters* params, bool doInverse)
{
	float muCoeff = params->muCoeff;
	if (doInverse)
		muCoeff *= -1.0;
	for (int iphi = 0; iphi < params->numAngles; iphi++)
	{
		for (int iv = 0; iv < params->numRows; iv++)
		{
			for (int iu = 0; iu < params->numCols; iu++)
			{
				float u = params->u(iu);

				if (fabs(u) < params->muRadius)
					g[uint64(iphi) * uint64(params->numRows * params->numCols) + uint64(iv * params->numCols + iu)] *= exp(muCoeff * sqrt(params->muRadius * params->muRadius - u * u));
			}
		}
	}
	return true;
}
