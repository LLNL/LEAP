#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "hip/hip_runtime.h"
#include <string.h>

__global__ void cosKernel(float* lhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] = cos(lhs[iz * dim.x * dim.y + iy * dim.x + ix]);
}

__global__ void sinKernel(float* lhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] = sin(lhs[iz * dim.x * dim.y + iy * dim.x + ix]);
}

__global__ void expKernel(float* lhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] = expf(lhs[iz * dim.x * dim.y + iy * dim.x + ix]);
}

__global__ void negExpKernel(float* lhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] = expf(-lhs[iz * dim.x * dim.y + iy * dim.x + ix]);
}

__global__ void setToConstantKernel(float* lhs, const float c, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] = c;
}

__global__ void equalKernel(float* lhs, const float* rhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] = rhs[iz * dim.x * dim.y + iy * dim.x + ix];
}

__global__ void multiplyKernel(float* lhs, const float* rhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] *= rhs[iz * dim.x * dim.y + iy * dim.x + ix];
}

__global__ void divideKernel(float* lhs, const float* rhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;
    const float rhs_val = rhs[iz * dim.x * dim.y + iy * dim.x + ix];

    if (rhs_val == 0.0f)
        lhs[iz * dim.x * dim.y + iy * dim.x + ix] = 1.0f;
    else
        lhs[iz * dim.x * dim.y + iy * dim.x + ix] *= rhs_val;
}

__global__ void addKernel(float* lhs, const float* rhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] += rhs[iz * dim.x * dim.y + iy * dim.x + ix];
}

__global__ void addKernel(float* lhs, const float rhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] += rhs;
}

__global__ void subKernel(float* lhs, const float* rhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] -= rhs[iz * dim.x * dim.y + iy * dim.x + ix];
}

__global__ void scaleKernel(float* lhs, const float c, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] *= c;
}

__global__ void scalarAddKernel(float* lhs, const float c, const float* rhs, const int3 dim)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int iz = threadIdx.z + blockIdx.z * blockDim.z;

    if (ix >= dim.x || iy >= dim.y || iz >= dim.z)
        return;

    lhs[iz * dim.x * dim.y + iy * dim.x + ix] += c*rhs[iz * dim.x * dim.y + iy * dim.x + ix];
}

__global__ void sumKernel(const float* x, float* sum_x, const int3 N)
{
    if (threadIdx.x > 0)
        return;

    *sum_x = 0.0f;
    for (int i = 0; i < N.x; i++)
    {
        for (int j = 0; j < N.y; j++)
        {
            for (int k = 0; k < N.z; k++) *sum_x += x[i * N.y * N.z + j * N.z + k];
        }
    }
}

__global__ void innerProductKernel(const float* x, const float* y, float* sum_x, const int3 N)
{
    if (threadIdx.x > 0)
        return;

    *sum_x = 0.0f;
    for (int i = 0; i < N.x; i++)
    {
        for (int j = 0; j < N.y; j++)
        {
            for (int k = 0; k < N.z; k++) *sum_x += x[i * N.y * N.z + j * N.z + k] * y[i * N.y * N.z + j * N.z + k];
        }
    }
}

__global__ void sum_2D(const float* x, float* sum_x, int3 N)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= N.x)
        return;

    const float* x_slice = &x[i * N.y * N.z];
    float accum = 0.0f;
    for (int j = 0; j < N.y; j++)
    {
        for (int k = 0; k < N.z; k++)
            accum += x_slice[j * N.z + k];
    }
    sum_x[i] = accum;
}

__global__ void innerProductKernel_2D(const float* x, const float* y, float* sum_x, const int3 N)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= N.x)
        return;

    const float* x_slice = &x[i * N.y * N.z];
    const float* y_slice = &y[i * N.y * N.z];
    float accum = 0.0f;
    for (int j = 0; j < N.y; j++)
    {
        for (int k = 0; k < N.z; k++)
            accum += x_slice[j * N.z + k] * y_slice[j * N.z + k];
    }
    sum_x[i] = accum;
}

__global__ void sum_1D(const float* x, float* sum_x, int N)
{
    if (threadIdx.x > 0)
        return;

    *sum_x = 0.0f;
    for (int i = 0; i < N; i++)
        *sum_x += x[i];
}

__global__ void weightedInnerProductKernel(const float* x, const float* w, const float* y, float* sum_x, const int3 N)
{
    if (threadIdx.x > 0)
        return;

    *sum_x = 0.0f;
    for (int i = 0; i < N.x; i++)
    {
        for (int j = 0; j < N.y; j++)
        {
            for (int k = 0; k < N.z; k++) *sum_x += x[i * N.y * N.z + j * N.z + k] * y[i * N.y * N.z + j * N.z + k] * w[i * N.y * N.z + j * N.z + k];
        }
    }
}

int numberOfGPUs()
{
    int num_gpus = 0;
    hipError_t err = hipGetDeviceCount(&num_gpus);
    if (err == hipSuccess)
        return num_gpus;
    else
        return 0;
}

float getAvailableGPUmemory(int whichGPU)
{
    if (whichGPU >= 0)
    {
        hipSetDevice(whichGPU);
        std::size_t free_byte;
        std::size_t total_byte;
        hipMemGetInfo(&free_byte, &total_byte);
        return float(double(free_byte) / pow(2.0, 30.0));
    }
    else
        return 0.0;
}

hipError_t setToConstant(float* dev_lhs, const float c, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    setToConstantKernel <<< dimGrid, dimBlock >>> (dev_lhs, c, N);
    return hipPeekAtLastError();
}

hipError_t equal(float* dev_lhs, const float* dev_rhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    equalKernel <<< dimGrid, dimBlock >>> (dev_lhs, dev_rhs, N);
    return hipPeekAtLastError();
}

hipError_t multiply(float* dev_lhs, const float* dev_rhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    multiplyKernel<<< dimGrid, dimBlock >>> (dev_lhs, dev_rhs, N);
    return hipPeekAtLastError();
}

hipError_t divide(float* dev_lhs, const float* dev_rhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    divideKernel <<< dimGrid, dimBlock >>> (dev_lhs, dev_rhs, N);
    return hipPeekAtLastError();
}

hipError_t add(float* dev_lhs, const float* dev_rhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    addKernel <<< dimGrid, dimBlock >>> (dev_lhs, dev_rhs, N);
    return hipPeekAtLastError();
}

hipError_t add(float* dev_lhs, const float c, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    addKernel <<< dimGrid, dimBlock >>> (dev_lhs, c, N);
    return hipPeekAtLastError();
}

hipError_t sub(float* dev_lhs, const float* dev_rhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    subKernel <<< dimGrid, dimBlock >>> (dev_lhs, dev_rhs, N);
    return hipPeekAtLastError();
}

hipError_t scale(float* dev_lhs, const float c, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    scaleKernel <<< dimGrid, dimBlock >>> (dev_lhs, c, N);
    return hipPeekAtLastError();
}

hipError_t scalarAdd(float* dev_lhs, const float c, const float* dev_rhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    scalarAddKernel <<< dimGrid, dimBlock >>> (dev_lhs, c, dev_rhs, N);
    return hipPeekAtLastError();
}

hipError_t cosFcn(float* dev_lhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    cosKernel <<< dimGrid, dimBlock >>> (dev_lhs, N);
    return hipPeekAtLastError();
}

hipError_t sinFcn(float* dev_lhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    sinKernel <<< dimGrid, dimBlock >>> (dev_lhs, N);
    return hipPeekAtLastError();
}

hipError_t expFcn(float* dev_lhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    expKernel <<< dimGrid, dimBlock >>> (dev_lhs, N);
    return hipPeekAtLastError();
}

hipError_t negExpFcn(float* dev_lhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    negExpKernel <<< dimGrid, dimBlock >>> (dev_lhs, N);
    return hipPeekAtLastError();
}

float sum(const float* dev_lhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    hipError_t cudaStatus;
    float* dev_sum = 0;
    if ((cudaStatus = hipMalloc((void**)&dev_sum, 1 * sizeof(float))) != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!\n");
        return 0.0;
    }
    //sumKernel<<<1,1>>>(dev_lhs, dev_sum, N);

    //*
    float* dev_sum_1D = 0;
    if ((cudaStatus = hipMalloc((void**)&dev_sum_1D, N.x * sizeof(float))) != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!\n");
        return 0.0;
    }
    int blockSize = 8;
    int gridSize = int(ceil(double(N.x) / double(blockSize)));
    sum_2D <<< gridSize, blockSize >>> (dev_lhs, dev_sum_1D, N);
    sum_1D <<< 1, 1 >>> (dev_sum_1D, dev_sum, N.x);
    hipFree(dev_sum_1D);
    //*/

    float retVal = 0.0;
    hipMemcpy(&retVal, dev_sum, sizeof(float), hipMemcpyDeviceToHost);

    if (dev_sum != 0)
    {
        hipFree(dev_sum);
    }
    return retVal;
}

float innerProduct(const float* dev_lhs, const float* dev_rhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    hipError_t cudaStatus;
    float* dev_sum = 0;
    if ((cudaStatus = hipMalloc((void**)&dev_sum, 1 * sizeof(float))) != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!\n");
        return 0.0;
    }
    //innerProductKernel <<<1, 1 >>> (dev_lhs, dev_rhs, dev_sum, N);

    //*
    float* dev_sum_1D = 0;
    if ((cudaStatus = hipMalloc((void**)&dev_sum_1D, N.x * sizeof(float))) != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!\n");
        return 0.0;
    }
    int blockSize = 8;
    int gridSize = int(ceil(double(N.x) / double(blockSize)));
    innerProductKernel_2D <<< gridSize, blockSize >>> (dev_lhs, dev_rhs, dev_sum_1D, N);
    sum_1D <<< 1, 1 >>> (dev_sum_1D, dev_sum, N.x);
    hipFree(dev_sum_1D);
    //*/

    float retVal = 0.0;
    hipMemcpy(&retVal, dev_sum, sizeof(float), hipMemcpyDeviceToHost);

    if (dev_sum != 0)
    {
        hipFree(dev_sum);
    }
    return retVal;
}

float weightedInnerProduct(const float* dev_lhs, const float* dev_w, const float* dev_rhs, const int3 N, int whichGPU)
{
    hipSetDevice(whichGPU);
    hipError_t cudaStatus;
    float* dev_sum = 0;
    if ((cudaStatus = hipMalloc((void**)&dev_sum, 1 * sizeof(float))) != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!\n");
        return 0.0;
    }
    weightedInnerProductKernel <<<1, 1 >>> (dev_lhs, dev_w, dev_rhs, dev_sum, N);

    float retVal = 0.0;
    hipMemcpy(&retVal, dev_sum, sizeof(float), hipMemcpyDeviceToHost);

    if (dev_sum != 0)
    {
        hipFree(dev_sum);
    }
    return retVal;
}

dim3 setBlockSize(int3 N)
{
	dim3 dimBlock(8, 8, 8);  // needs to be optimized
	if (N.z < 8)
	{
		dimBlock.x = 16;
		dimBlock.y = 16;
		dimBlock.z = 1;
	}
	else if (N.y < 8)
	{
		dimBlock.x = 16;
		dimBlock.y = 1;
		dimBlock.z = 16;
	}
	else if (N.x < 8)
	{
		dimBlock.x = 1;
		dimBlock.y = 16;
		dimBlock.z = 16;
	}
	return dimBlock;
}

dim3 setGridSize(int3 N, dim3 dimBlock)
{
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))), int(ceil(double(N.z) / double(dimBlock.z))));
    return dimGrid;
}

dim3 setBlockSize(int4 N)
{
    return setBlockSize(make_int3(N.x, N.y, N.z));
}

dim3 setGridSize(int4 N, dim3 dimBlock)
{
    return setGridSize(make_int3(N.x, N.y, N.z), dimBlock);
}

hipArray* loadTexture(hipTextureObject_t& tex_object, float* dev_data, const int4 N_txt, bool useExtrapolation, bool useLinearInterpolation, bool swapFirstAndLastDimensions)
{
    int3 N = make_int3(N_txt.x, N_txt.y, N_txt.z);
    if (swapFirstAndLastDimensions)
    {
        N.x = N_txt.z;
        N.z = N_txt.x;
    }
    return loadTexture(tex_object, dev_data, N, useExtrapolation, useLinearInterpolation);
}

hipArray* loadTexture(hipTextureObject_t& tex_object, float* dev_data, const int3 N_txt, bool useExtrapolation, bool useLinearInterpolation, bool swapFirstAndLastDimensions)
{
    int3 N = make_int3(N_txt.x, N_txt.y, N_txt.z);
    if (swapFirstAndLastDimensions)
    {
        N.x = N_txt.z;
        N.z = N_txt.x;
    }
    return loadTexture(tex_object, dev_data, N, useExtrapolation, useLinearInterpolation);
}

hipArray* loadTexture(hipTextureObject_t& tex_object, float* dev_data, const int4 N_txt, bool useExtrapolation, bool useLinearInterpolation)
{
    int3 N3 = make_int3(N_txt.x, N_txt.y, N_txt.z);
    return loadTexture(tex_object, dev_data, N3, useExtrapolation, useLinearInterpolation);
}

hipArray* loadTexture(hipTextureObject_t& tex_object, float* dev_data, const int3 N_txt, bool useExtrapolation, bool useLinearInterpolation)
{
  if (dev_data == nullptr)
    return nullptr;
  hipArray* d_data_array = nullptr;

  // Allocate 3D array memory
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  hipMalloc3DArray(&d_data_array, &channelDesc, make_hipExtent(N_txt.z, N_txt.y, N_txt.x));
 
  // Bind 3D array to texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = (hipArray_t)d_data_array;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = false;  // Texture coordinates normalization

  if (useExtrapolation)
  {
    texDesc.addressMode[0] = (hipTextureAddressMode)hipAddressModeClamp;
    texDesc.addressMode[1] = (hipTextureAddressMode)hipAddressModeClamp;
    texDesc.addressMode[2] = (hipTextureAddressMode)hipAddressModeClamp;
  }
  else
  {
    texDesc.addressMode[0] = (hipTextureAddressMode)hipAddressModeBorder;
    texDesc.addressMode[1] = (hipTextureAddressMode)hipAddressModeBorder;
    texDesc.addressMode[2] = (hipTextureAddressMode)hipAddressModeBorder;
  }

  if (useLinearInterpolation)
  {
      texDesc.filterMode = (hipTextureFilterMode)hipFilterModeLinear;
  }
  else
  {
      texDesc.filterMode = (hipTextureFilterMode)hipFilterModePoint;
  }
  hipCreateTextureObject(&tex_object, &resDesc, &texDesc, nullptr);

  // Update the texture memory
  hipMemcpy3DParms cudaparams = {0};
  cudaparams.extent = make_hipExtent(N_txt.z, N_txt.y, N_txt.x);
  cudaparams.kind = hipMemcpyDeviceToDevice;
  cudaparams.srcPos = make_hipPos(0, 0, 0);
  cudaparams.srcPtr = make_hipPitchedPtr(dev_data, N_txt.z * sizeof(float), N_txt.z, N_txt.y);
  cudaparams.dstPos = make_hipPos(0, 0, 0);
  cudaparams.dstArray = (hipArray_t)d_data_array;
  hipMemcpy3D(&cudaparams);
  return d_data_array;
}

float* copyProjectionDataToGPU(float* g, parameters* params, int whichGPU)
{
	hipSetDevice(whichGPU);

	int N = params->numAngles * params->numRows * params->numCols;

	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Copy volume data to GPU
	float* dev_g = 0;
	if (hipMalloc((void**)&dev_g, N * sizeof(float)) != hipSuccess)
	{
		fprintf(stderr, "hipMalloc(projection) failed!\n");
		return NULL;
	}
	if (hipMemcpy(dev_g, g, N * sizeof(float), hipMemcpyHostToDevice))
	{
		fprintf(stderr, "hipMemcpy(projection) failed!\n");
		return NULL;
	}

	return dev_g;
}

bool pullProjectionDataFromGPU(float* g, parameters* params, float* dev_g, int whichGPU)
{
	hipSetDevice(whichGPU);
	hipError_t cudaStatus;

	int N = params->numAngles * params->numRows * params->numCols;

	cudaStatus = hipMemcpy(g, dev_g, N * sizeof(float), hipMemcpyDeviceToHost);
	if (hipSuccess != cudaStatus)
	{
		fprintf(stderr, "failed to copy projection data back to host!\n");
		fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
		fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
		return false;
	}
	else
		return true;
}

float* copyVolumeDataToGPU(float* f, parameters* params, int whichGPU)
{
	hipSetDevice(whichGPU);

	int N = params->numX * params->numY * params->numZ;

	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Copy volume data to GPU
	float* dev_f = 0;
	if (hipMalloc((void**)&dev_f, N * sizeof(float)) != hipSuccess)
	{
		fprintf(stderr, "hipMalloc(volume) failed!\n");
		return NULL;
	}
	if (hipMemcpy(dev_f, f, N * sizeof(float), hipMemcpyHostToDevice))
	{
		fprintf(stderr, "hipMemcpy(volume) failed!\n");
		return NULL;
	}

	return dev_f;
}

bool pullVolumeDataFromGPU(float* f, parameters* params, float* dev_f, int whichGPU)
{
	hipSetDevice(whichGPU);
	hipError_t cudaStatus;
	int N = params->numX * params->numY * params->numZ;
	cudaStatus = hipMemcpy(f, dev_f, N * sizeof(float), hipMemcpyDeviceToHost);
	if (hipSuccess != cudaStatus)
	{
		fprintf(stderr, "failed to copy volume data back to host!\n");
		fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
		fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
		return false;
	}
	else
		return true;
}

float* copy3DdataToGPU(float* g, int3 N, int whichGPU)
{
	hipSetDevice(whichGPU);

	int N_prod = N.x * N.y * N.z;

	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Copy volume data to GPU
	float* dev_g = 0;
	if (hipMalloc((void**)&dev_g, N_prod * sizeof(float)) != hipSuccess)
	{
		fprintf(stderr, "hipMalloc(volume) failed!\n");
		return NULL;
	}
	if (hipMemcpy(dev_g, g, N_prod * sizeof(float), hipMemcpyHostToDevice))
	{
		fprintf(stderr, "hipMemcpy(volume) failed!\n");
		return NULL;
	}

	return dev_g;
}

bool pull3DdataFromGPU(float* g, int3 N, float* dev_g, int whichGPU)
{
	hipSetDevice(whichGPU);
	hipError_t cudaStatus;
	int N_prod = N.x * N.y * N.z;
	cudaStatus = hipMemcpy(g, dev_g, N_prod * sizeof(float), hipMemcpyDeviceToHost);
	if (hipSuccess != cudaStatus)
	{
		fprintf(stderr, "failed to copy volume data back to host!\n");
		fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
		fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
		return false;
	}
	else
		return true;
}

float* copyAngleArrayToGPU(parameters* params)
{
    hipSetDevice(params->whichGPU);
    hipError_t cudaStatus;
    float* dev_phis = 0;
    if (hipSuccess != hipMalloc((void**)&dev_phis, params->numAngles * sizeof(float)))
        fprintf(stderr, "hipMalloc failed!\n");
    if (hipMemcpy(dev_phis, params->phis, params->numAngles * sizeof(float), hipMemcpyHostToDevice))
        fprintf(stderr, "hipMemcpy(phis) failed!\n");
    return dev_phis;
}

bool setProjectionGPUparams(parameters* params, int4& N, float4& T, float4& startVals, bool doNormalize)
{
    if (params == NULL)
        return false;
    else
    {
        N.x = params->numAngles; N.y = params->numRows; N.z = params->numCols;
        T.x = params->T_phi(); T.y = params->pixelHeight; T.z = params->pixelWidth;
        startVals.x = params->phi_0(); startVals.y = params->v_0(); startVals.z = params->u_0();

        if (params->geometry == parameters::CONE)
        {
            N.w = params->numAngles;
            T.w = params->helicalPitch;
            startVals.w = params->z_source_offset;
        }
        else
        {
            N.w = params->numAngles;
            T.w = 0.0;
            startVals.w = 0.0;
        }
        if (doNormalize)
        {
            if (params->geometry == parameters::CONE)
            {
                T.y = T.y / params->sdd;
                T.z = T.z / params->sdd;
                startVals.y = startVals.y / params->sdd;
                startVals.z = startVals.z / params->sdd;
            }
            else if (params->geometry == parameters::FAN)
            {
                T.z = T.z / params->sdd;
                startVals.z = startVals.z / params->sdd;
            }
        }

        return true;
    }
}

bool setVolumeGPUparams(parameters* params, int4& N, float4& T, float4& startVals)
{
    if (params == NULL)
        return false;
    else
    {
        N.x = params->numX; N.y = params->numY; N.z = params->numZ;
        T.x = params->voxelWidth; T.y = params->voxelWidth; T.z = params->voxelHeight;
        startVals.x = params->x_0(); startVals.y = params->y_0(); startVals.z = params->z_0();
        return true;
    }
}