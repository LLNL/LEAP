#include "hip/hip_runtime.h"

#include "ramp_filter.cuh"
#include "ramp_filter_cpu.h"

#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "cuda_utils.h"

#define INCLUDE_CUFFT
#ifndef PI
#define PI 3.141592653589793
#endif

#ifdef INCLUDE_CUFFT
#include <hipfft/hipfft.h>


__global__ void multiply2DRampFilterKernel(hipfftComplex* F, const float* H, int3 N)
{
    // int k = threadIdx.x;
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k > 0) return;
    // if (k > N.z - 1)
    //	return;
    for (int j = 0; j < N.y; j++)
    {
        for (int i = 0; i < N.x; i++)
        {
            F[k * N.x * N.y + j * N.x + i].x *= H[j * N.x + i];
            F[k * N.x * N.y + j * N.x + i].y *= H[j * N.x + i];
        }
    }
}

__global__ void setPaddedDataKernel(float* data_padded, float* data, int3 N, int N_pad, int startView, int endView, int numExtrapolate)
{
    int j = threadIdx.x;
    int i = blockIdx.x + startView;
    if (i > endView || j > N.y - 1)
        return;
    float* data_padded_block = &data_padded[(i - startView) * N_pad * N.y + j * N_pad];
    float* data_block = &data[i * N.z * N.y + j * N.z];
    for (int k = 0; k < N.z; k++)
        data_padded_block[k] = data_block[k];

    for (int k = N.z; k < N_pad; k++)
        data_padded_block[k] = 0.0;

    if (numExtrapolate > 0)
    {
        const float leftVal = data_block[0];
        const float rightVal = data_block[N.z - 1];
        for (int k = N.z; k < N.z + numExtrapolate; k++)
            data_padded_block[k] = rightVal;
        for (int k = N_pad - numExtrapolate; k < N_pad; k++)
            data_padded_block[k] = leftVal;
    }
}

__global__ void multiplyRampFilterKernel(hipfftComplex* G, const float* H, int3 N)
{
    int j = threadIdx.x;
    int i = blockIdx.x;
    if (i > N.x - 1 || j > N.y - 1)
        return;
    for (int k = 0; k < N.z; k++)
    {
        G[i * N.y * N.z + j * N.z + k].x *= H[k];
        G[i * N.y * N.z + j * N.z + k].y *= H[k];
    }
}

__global__ void setFilteredDataKernel(float* data_padded, float* data, int3 N, int N_pad, int startView, int endView)
{
    int j = threadIdx.x;
    int i = blockIdx.x + startView;
    if (i > endView || j > N.y - 1)
        return;
    float* data_padded_block = &data_padded[(i - startView) * N_pad * N.y + j * N_pad];
    float* data_block = &data[i * N.z * N.y + j * N.z];
    for (int k = 0; k < N.z; k++)
        data_block[k] = data_padded_block[k];
}

float* rampFilterFrequencyResponseMagnitude(int N, parameters* params)
{
    float T = params->pixelWidth;
    bool isCurved = false;
    if (params->geometry == parameters::FAN || params->geometry == parameters::CONE)
    {
        T *= params->sod / params->sdd;
        if (params->detectorType == parameters::CURVED)
            isCurved = true;
    }

    int rampID = 2;

    hipError_t cudaStatus;
    double* h_d = rampImpulseResponse(N, T, rampID);
    float* h = new float[N];
    for (int i = 0; i < N; i++)
    {
        h[i] = h_d[i];

        if (i != 0 && isCurved == true)
        {
            double s = timeSamples(i, N) * T / params->sod;
            double temp = s / sin(s);
            h[i] *= temp * temp;
        }
    }
    delete[] h_d;

    // Make cuFFT Plans
    hipfftResult result;
    hipfftHandle forward_plan;
    if (HIPFFT_SUCCESS != hipfftPlan1d(&forward_plan, N, HIPFFT_R2C, 1))
    {
        fprintf(stderr, "Failed to plan 1d r2c fft");
        return NULL;
    }

    float* dev_h = 0;
    if (cudaStatus = hipMalloc((void**)&dev_h, N * sizeof(float)))
    {
        fprintf(stderr, "hipMalloc(padded projection data) failed!\n");
        return NULL;
    }
    cudaStatus = hipMemcpy(dev_h, h, N * sizeof(float), hipMemcpyHostToDevice);

    // Make data for the result of the FFT
    int N_over2 = N / 2 + 1;
    hipfftComplex* dev_H = 0;
    if (cudaStatus = hipMalloc((void**)&dev_H, N_over2 * sizeof(hipfftComplex)))
    {
        fprintf(stderr, "hipMalloc(Fourier transform of ramp filter) failed!\n");
        return NULL;
    }

    // FFT
    result = hipfftExecR2C(forward_plan, (hipfftReal*)dev_h, dev_H);
    hipDeviceSynchronize();

    // get result
    hipfftComplex* H_ramp = new hipfftComplex[N_over2];
    cudaStatus = hipMemcpy(H_ramp, dev_H, N_over2 * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    float* H_real = new float[N_over2];
    for (int i = 0; i < N_over2; i++)
    {
        H_real[i] = H_ramp[i].x / float(N);
    }

    // Clean up
    hipfftDestroy(forward_plan);
    hipFree(dev_h);
    hipFree(dev_H);
    delete[] h;
    delete[] H_ramp;

    return H_real;
}

bool rampFilter1D(float*& g, parameters* params, bool cpu_to_gpu, float scalar)
{
    bool retVal = true;
    hipSetDevice(params->whichGPU);
    hipError_t cudaStatus;

    float* dev_g = 0;
    if (cpu_to_gpu)
    {
        dev_g = copyProjectionDataToGPU(g, params, params->whichGPU);
    }
    else
    {
        dev_g = g;
    }

    // PUT CODE HERE
    int N_H = int(pow(2.0, ceil(log2(2 * params->numCols))));
    int N_H_over2 = N_H / 2 + 1;
    float* H_real = rampFilterFrequencyResponseMagnitude(N_H, params);
    if (scalar != 1.0)
    {
        for (int i = 0; i < N_H_over2; i++)
            H_real[i] *= scalar;
    }

    //int N_viewChunk = params->numAngles;
    int N_viewChunk = params->numAngles / 40; // number of views in a chunk (needs to be optimized)
    int numChunks = int(ceil(double(params->numAngles) / double(N_viewChunk)));

    // Make cuFFT Plans
    hipfftResult result;
    hipfftHandle forward_plan;
    if (HIPFFT_SUCCESS != hipfftPlan1d(&forward_plan, N_H, HIPFFT_R2C, N_viewChunk * params->numRows))
    {
        fprintf(stderr, "Failed to plan 1d r2c fft (size %d)\n", N_H);
        return false;
    }
    hipfftHandle backward_plan;
    if (HIPFFT_SUCCESS != hipfftPlan1d(&backward_plan, N_H, HIPFFT_C2R, N_viewChunk * params->numRows)) // do I use N_H_over2?
    {
        fprintf(stderr, "Failed to plan 1d c2r ifft\n");
        return false;
    }
    //return true;

    float* dev_g_pad = 0;
    if (cudaStatus = hipMalloc((void**)&dev_g_pad, N_viewChunk * params->numRows * N_H * sizeof(float)))
    {
        fprintf(stderr, "hipMalloc(padded projection data) failed!\n");
        retVal = false;
    }

    // Make data for the result of the FFT
    hipfftComplex* dev_G = 0;
    if (cudaStatus = hipMalloc((void**)&dev_G, N_viewChunk * params->numRows * N_H_over2 * sizeof(hipfftComplex)))
    {
        fprintf(stderr, "hipMalloc(Fourier transform of padded projection data) failed!\n");
        retVal = false;
    }

    // Copy filter to device
    float* dev_H = 0;
    if (hipSuccess != hipMalloc((void**)&dev_H, N_H_over2 * sizeof(float)))
        fprintf(stderr, "hipMalloc failed!\n");
    if (H_real == NULL)
        printf("H_real is NULL!!!\n");
    cudaStatus = hipMemcpy(dev_H, H_real, N_H_over2 * sizeof(float), hipMemcpyHostToDevice);
    if (hipSuccess != cudaStatus)
    {
        fprintf(stderr, "hipMemcpy(H) failed!\n");
        fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
        fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
        retVal = false;
    }
    int3 dataSize; dataSize.x = N_viewChunk; dataSize.y = params->numRows; dataSize.z = N_H_over2;
    int3 origSize; origSize.x = params->numAngles; origSize.y = params->numRows; origSize.z = params->numCols;

    int numExtrapolate = 0;

    if (retVal == true)
    {
        for (int iChunk = 0; iChunk < numChunks; iChunk++)
        {
            int startView = iChunk * N_viewChunk;
            int endView = min(params->numAngles - 1, startView + N_viewChunk - 1);

            setPaddedDataKernel <<< endView - startView + 1, params->numRows >>> (dev_g_pad, dev_g, origSize, N_H, startView, endView, numExtrapolate);
            hipDeviceSynchronize();

            // FFT
            result = hipfftExecR2C(forward_plan, (hipfftReal*)dev_g_pad, dev_G);

            // Multiply Filter
            multiplyRampFilterKernel <<< N_viewChunk, params->numRows >>> (dev_G, dev_H, dataSize);
            hipDeviceSynchronize();

            // IFFT
            result = hipfftExecC2R(backward_plan, (hipfftComplex*)dev_G, (hipfftReal*)dev_g_pad);

            setFilteredDataKernel <<< endView - startView + 1, params->numRows >>> (dev_g_pad, dev_g, origSize, N_H, startView, endView);
            hipDeviceSynchronize();
        }

        if (cpu_to_gpu)
        {
            // Copy result back to host
            cudaStatus = hipMemcpy(g, dev_g, params->numAngles * params->numRows * params->numCols * sizeof(float), hipMemcpyDeviceToHost);
            if (hipSuccess != cudaStatus)
            {
                fprintf(stderr, "failed to copy result back to host!\n");
                fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
                fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
            }
        }
    }

    // Clean up
    hipfftDestroy(forward_plan);
    hipfftDestroy(backward_plan);
    hipFree(dev_g_pad);
    if (cpu_to_gpu)
        hipFree(dev_g);
    hipFree(dev_H);
    hipFree(dev_G);
    delete[] H_real;

    return retVal;
}

bool rampFilter2D(float*& f, parameters* params, bool cpu_to_gpu)
{
    if (cpu_to_gpu == false)
    {
        printf("Error: current implementation of rampFilter2D requires that data reside on the CPU\n");
        return false;
    }

    int N_x = params->numX;
    int N_y = params->numY;
    int N_z = params->numZ;

    // Pad and then find next power of 2
    int N_H1 = int(pow(2.0, ceil(log2(2 * max(N_y, N_x)))));
    int N_H2 = N_H1;
    int N_H2_over2 = N_H2 / 2 + 1;

    hipSetDevice(params->whichGPU);
    bool retVal = true;

    int smoothingLevel = 0;

    // Make cuFFT Plans
    hipfftResult result;
    hipfftHandle forward_plan;
    if (HIPFFT_SUCCESS != hipfftPlan2d(&forward_plan, N_H1, N_H2, HIPFFT_R2C))
    {
        fprintf(stderr, "Failed to plan 2d r2c fft");
        return false;
    }
    hipfftHandle backward_plan;
    if (HIPFFT_SUCCESS != hipfftPlan2d(&backward_plan, N_H1, N_H2, HIPFFT_C2R))  // do I use N_H_over2?
    {
        fprintf(stderr, "Failed to plan 2d c2r ifft");
        return false;
    }

    float* paddedSlice = (float*)malloc(sizeof(float) * N_H1 * N_H2);
    // Make zero-padded array, copy data to 1st half of array and set remaining slots to zero
    hipError_t cudaStatus;
    float* dev_f_pad = 0;
    if (cudaStatus = hipMalloc((void**)&dev_f_pad, N_H1 * N_H2 * sizeof(float)))
    {
        fprintf(stderr, "hipMalloc(padded volume data) failed!\n");
        retVal = false;
    }

    // Make data for the result of the FFT
    hipfftComplex* dev_F = 0;
    if (cudaStatus = hipMalloc((void**)&dev_F, N_H1 * N_H2_over2 * sizeof(hipfftComplex)))
    {
        fprintf(stderr, "hipMalloc(Fourier transform of padded volume data) failed!\n");
        retVal = false;
    }

    // Copy filter to device
    float* H = rampFrequencyResponse2D(N_H1, 1.0, 1.0, smoothingLevel);  // FIXME?
    float* dev_H = 0;
    if (hipSuccess != hipMalloc((void**)&dev_H, N_H1 * N_H2_over2 * sizeof(float)))
        fprintf(stderr, "hipMalloc failed!\n");
    cudaStatus = hipMemcpy(dev_H, H, N_H1 * N_H2_over2 * sizeof(float), hipMemcpyHostToDevice);
    if (hipSuccess != cudaStatus)
    {
        fprintf(stderr, "hipMemcpy(H) failed!\n");
        fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
        fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
        retVal = false;
    }

    for (int k = 0; k < N_z; k++)
    {
        if (params->volumeDimensionOrder == parameters::XYZ)
        {
            for (int j = 0; j < N_H1; j++)
            {
                int j_source = j;
                if (j >= N_y)
                {
                    if (j - N_y < N_H1 - j)
                        j_source = N_y - 1;
                    else
                        j_source = 0;
                }
                for (int i = 0; i < N_H2; i++)
                {
                    int i_source = i;
                    if (i >= N_x)
                    {
                        if (i - N_x < N_H2 - i)
                            i_source = N_x - 1;
                        else
                            i_source = 0;
                    }
                    paddedSlice[j * N_H2 + i] = f[i_source * N_y * N_z + j_source*N_z + k];
                }
            }
        }
        else //if (params->volumeDimensionOrder == parameters::ZYX)
        {
            float* f_slice = &f[k * N_x * N_y];
            for (int j = 0; j < N_H1; j++)
            {
                int j_source = j;
                if (j >= N_y)
                {
                    if (j - N_y < N_H1 - j)
                        j_source = N_y - 1;
                    else
                        j_source = 0;
                }
                for (int i = 0; i < N_H2; i++)
                {
                    int i_source = i;
                    if (i >= N_x)
                    {
                        if (i - N_x < N_H2 - i)
                            i_source = N_x - 1;
                        else
                            i_source = 0;
                    }
                    paddedSlice[j * N_H2 + i] = f_slice[j_source * N_x + i_source];
                }
            }
        }
        if (hipMemcpy(dev_f_pad, paddedSlice, N_H1 * N_H2 * sizeof(float), hipMemcpyHostToDevice))
        {
            fprintf(stderr, "hipMemcpy(padded volume data) failed!\n");
            retVal = false;
        }

        // FFT
        result = hipfftExecR2C(forward_plan, (hipfftReal*)dev_f_pad, dev_F);

        // Multiply Filter
        int3 dataSize;
        dataSize.z = N_z;
        dataSize.y = N_H1;
        dataSize.x = N_H2_over2;
        multiply2DRampFilterKernel<<<1, 1>>>(dev_F, dev_H, dataSize);

        // IFFT
        result = hipfftExecC2R(backward_plan, (hipfftComplex*)dev_F, (hipfftReal*)dev_f_pad);

        // Copy result back to host
        if (retVal)
        {
            cudaStatus = hipMemcpy(paddedSlice, dev_f_pad, N_H1 * N_H2 * sizeof(float), hipMemcpyDeviceToHost);
            if (hipSuccess != cudaStatus)
            {
                fprintf(stderr, "failed to copy result back to host!\n");
                fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
                fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
                retVal = false;
            }
            if (params->volumeDimensionOrder == parameters::XYZ)
            {
                for (int j = 0; j < N_y; j++)
                {
                    for (int i = 0; i < N_x; i++)
                    {
                        f[i * N_y * N_z + j * N_z + k] = paddedSlice[j * N_H2 + i] / float(N_H1 * N_H2);
                    }
                }
            }
            else
            {
                float* f_slice = &f[k * N_x * N_y];
                for (int j = 0; j < N_y; j++)
                {
                    for (int i = 0; i < N_x; i++)
                    {
                        f_slice[j * N_x + i] = paddedSlice[j * N_H2 + i] / float(N_H1 * N_H2);
                    }
                }
            }
        }
    }

    // Clean up
    hipfftDestroy(forward_plan);
    hipfftDestroy(backward_plan);
    hipFree(dev_f_pad);
    hipFree(dev_H);
    hipFree(dev_F);
    free(H);
    free(paddedSlice);

    return retVal;
}
#else
bool rampFilter1D(float*& g, parameters* params, bool cpu_to_gpu)
{
    //printf("CUFFT libraries not available!\n");
    //return false;
    return rampFilter1D_cpu(g, params);
}

bool rampFilter2D(float*& f, parameters* params, bool cpu_to_gpu)
{
    printf("CUFFT libraries not available!\n");
    return false;
}
#endif
