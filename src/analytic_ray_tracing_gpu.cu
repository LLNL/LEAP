#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright 2023-2024 Kyle Champley
// See the LICENSE file for details.
// SPDX-License-Identifier: MIT
//
// LivermorE AI Projector for Computed Tomography (LEAP)
// CT simulation via analytic ray tracing
////////////////////////////////////////////////////////////////////////////////

#include <math.h>
#include "analytic_ray_tracing_gpu.cuh"
#include "analytic_ray_tracing.h"
#include "hip/hip_runtime.h"
//#include ""
#include "cuda_utils.h"

#ifndef PI
#define PI 3.141592653589793f
#endif

#ifndef OUT_OF_BOUNDS
#define OUT_OF_BOUNDS 1.0e12f;
#endif

//enum geometry_list { CONE = 0, PARALLEL = 1, FAN = 2, MODULAR = 3, CONE_PARALLEL = 4 };

__constant__ int d_oversampling;
__constant__ int d_geometry;
__constant__ int d_CONE;
__constant__ int d_PARALLEL;
__constant__ int d_FAN;
__constant__ int d_MODULAR;
__constant__ int d_CONE_PARALLEL;
__constant__ int d_detectorType;
__constant__ int d_FLAT;
__constant__ int d_CURVED;
__constant__ float d_sod;
__constant__ float d_sdd;
__constant__ float d_tau;
__constant__ int4 d_N_g;
__constant__ float4 d_T_g;
__constant__ float4 d_startVal_g;

//enum objectType_list { ELLIPSOID = 0, PARALLELEPIPED = 1, CYLINDER_X = 2, CYLINDER_Y = 3, CYLINDER_Z = 4, CONE_X = 5, CONE_Y = 6, CONE_Z = 7 };
#define d_ELLIPSOID 0
#define d_PARALLELEPIPED 1
#define d_CYLINDER_X 2
#define d_CYLINDER_Y 3
#define d_CYLINDER_Z 4
#define d_CONE_X  5
#define d_CONE_Y 6
#define d_CONE_Z 7

__device__ float u(const int i)
{
    return float(i) * d_T_g.z + d_startVal_g.z;
}

__device__ float v(const int i)
{
    return float(i) * d_T_g.y + d_startVal_g.y;
}

__device__ float z_source(const float phi, const int k)
{
	if (d_geometry == d_CONE_PARALLEL)
	{
		//const float alpha = asin(u(k) / d_sod) + asin(d_tau / d_sod);
		return (phi + asin(u(k) / d_sod) + asin(d_tau / d_sod)) * d_T_g.w + d_startVal_g.w;
	}
	else
	    return phi * d_T_g.w + d_startVal_g.w;
}

__device__ float3 setSourcePosition(const float phi, const int iProj, const int iRow, const int iCol, const float dv, const float du)
{
    const float cos_phi = cos(phi);
    const float sin_phi = sin(phi);

    if (d_geometry == d_PARALLEL)
    {
        return make_float3(-(u(iCol) + du) * sin_phi,
            (u(iCol) + du) * cos_phi,
            v(iRow) + dv);
    }
    else if (d_geometry == d_FAN)
    {
        return make_float3(d_sod * cos_phi + d_tau * sin_phi,
            d_sod * sin_phi - d_tau * cos_phi,
            v(iRow) + dv);
    }
    else if (d_geometry == d_CONE)
    {
        return make_float3(d_sod * cos_phi + d_tau * sin_phi,
            d_sod * sin_phi - d_tau * cos_phi,
            z_source(phi, 0));
    }
    else if (d_geometry == d_CONE_PARALLEL)
    {
        const float s = u(iCol) + du;
        const float sqrt_R2_minus_s2 = sqrtf(d_sod * d_sod - s * s);
        return make_float3(-s * sin_phi + sqrt_R2_minus_s2 * cos_phi,
            s * cos_phi + sqrt_R2_minus_s2 * sin_phi,
            z_source(phi, iCol));
    }
    else
        return make_float3(0.0f, 0.0f, 0.0f);
}

__device__ float3 setTrajectory(const float phi, const int iProj, const int iRow, const int iCol, const float dv, const float du)
{
    const float u_val = u(iCol) + du;
    const float v_val = v(iRow) + dv;

    const float cos_phi = cos(phi);
    const float sin_phi = sin(phi);

    if (d_geometry == d_PARALLEL)
    {
        const float3 r = make_float3(-cos_phi, -sin_phi, 0.0f);
		const float r_mag_inv = rsqrtf(r.x * r.x + r.y * r.y + r.z * r.z);
		return make_float3(r.x * r_mag_inv, r.y * r_mag_inv, r.z * r_mag_inv);
    }
    else if (d_geometry == d_FAN)
    {
		const float3 r = make_float3(-(cos_phi + u_val * sin_phi), -(sin_phi - u_val * cos_phi), 0.0f);
		const float r_mag_inv = rsqrtf(r.x * r.x + r.y * r.y + r.z * r.z);
		return make_float3(r.x * r_mag_inv, r.y * r_mag_inv, r.z * r_mag_inv);
    }
    else if (d_geometry == d_CONE)
    {
        if (d_detectorType == d_CURVED)
        {
			const float3 r = make_float3(-cos(phi - u_val), -sin(phi - u_val), v_val);
			const float r_mag_inv = rsqrtf(r.x * r.x + r.y * r.y + r.z * r.z);
			return make_float3(r.x * r_mag_inv, r.y * r_mag_inv, r.z * r_mag_inv);
        }
        else
        {
			const float3 r = make_float3(-(cos_phi + u_val * sin_phi), -(sin_phi - u_val * cos_phi), v_val);
			const float r_mag_inv = rsqrtf(r.x * r.x + r.y * r.y + r.z * r.z);
			return make_float3(r.x * r_mag_inv, r.y * r_mag_inv, r.z * r_mag_inv);
        }
    }
    else if (d_geometry == d_CONE_PARALLEL)
    {
		const float3 r = make_float3(-cos_phi, -sin_phi, v_val);
		const float r_mag_inv = rsqrtf(r.x * r.x + r.y * r.y + r.z * r.z);
		return make_float3(r.x * r_mag_inv, r.y * r_mag_inv, r.z * r_mag_inv);
    }
    else
		return make_float3(0.0f, 0.0f, 0.0f);
}

__device__ bool parametersOfIntersection_1D(float2& ts, float p, float r)
{
	// finds ts such that p+t*r = +-1
	if (fabs(r) < 1e-12f)
	{
		if (fabs(p) < 1.0f)
		{
			ts.x = -OUT_OF_BOUNDS;
			ts.y = OUT_OF_BOUNDS;

			return true;
		}
		else
			return false;
	}
	else
	{
		if (r > 0.0f)
		{
			ts.x = (-1.0f - p) / r;
			ts.y = (1.0f - p) / r;
		}
		else
		{
			ts.y = (-1.0f - p) / r;
			ts.x = (1.0f - p) / r;
		}
		return true;
	}
}


__device__ bool parametersOfClippingPlaneIntersections(float2& ts, float3 p, float3 r, geometricSolid* solid)
{
	for (int i = 0; i < solid->numClippingPlanes; i++)
	{
		const float p_dot_n = solid->clippingPlanes[i][0] * p.x + solid->clippingPlanes[i][1] * p.y + solid->clippingPlanes[i][2] * p.z;
		const float r_dot_n = solid->clippingPlanes[i][0] * r.x + solid->clippingPlanes[i][1] * r.y + solid->clippingPlanes[i][2] * r.z;
		if (fabs(r_dot_n) < 1.0e-12f)
		{
			if (p_dot_n < solid->clippingPlanes[i][3])
				return false;
		}
		else if (r_dot_n > 0.0f)
		{
			const float temp = (solid->clippingPlanes[i][3] - p_dot_n) / r_dot_n;
			// restriction: t > temp
			if (ts.y < temp)
				return false;
			if (temp > ts.x)
				ts.x = temp;
		}
		else
		{
			// restriction: t < temp
			const float temp = (solid->clippingPlanes[i][3] - p_dot_n) / r_dot_n;
			if (ts.x > temp)
				return false;
			if (ts.y > temp)
				ts.y = temp;
		}
	}

	return true;
}

__device__ bool intersectionEndPoints_centeredAndNormalized(float3& p, float3& r, float2& ts, geometricSolid* solid)
{
	ts.x = -OUT_OF_BOUNDS;
	ts.y = ts.x;

	// r != (0,0,1)
	const float r_dot_r = r.x * r.x + r.y * r.y + r.z * r.z;
	const float p_dot_r = r.x * p.x + r.y * p.y + r.z * p.z;

	if (solid->type == d_ELLIPSOID)
	{
		float disc = p_dot_r * p_dot_r + r_dot_r * (1.0f - (p.x * p.x + p.y * p.y + p.z * p.z));
		if (disc > 0.0f)
		{
			disc = sqrtf(disc);
			ts.x = (-p_dot_r - disc) / r_dot_r;
			ts.y = (-p_dot_r + disc) / r_dot_r;
		}
		else
			return false;
	}
	else if (solid->type == d_PARALLELEPIPED)
	{
		float2 tx;
		float2 ty;
		float2 tz;
		if (parametersOfIntersection_1D(tx, p.x, r.x) == true)
		{
			if (parametersOfIntersection_1D(ty, p.y, r.y) == true)
			{
				if (parametersOfIntersection_1D(tz, p.z, r.z) == true)
				{
					ts.x = max(max(tx.x, ty.x), tz.x);
					ts.y = min(min(tx.y, ty.y), tz.y);
				}
				else
					return false;
			}
			else
				return false;
		}
		else
			return false;
	}
	else if (solid->type == d_CYLINDER_Z)
	{
		//double r_dot_r_2D = r.x * r.x + r.y * r.y; // 3
		const float r_dot_r_2D = r_dot_r - r.z * r.z; // 2
		const float p_dor_r_2D = p_dot_r - p.z * r.z; // 2
		float disc = p_dor_r_2D * p_dor_r_2D - r_dot_r_2D * (p.x * p.x + p.y * p.y - 1.0f); // 7
		if (disc > 0.0f)
		{
			disc = sqrtf(disc);
			const float tmin = (-p_dor_r_2D - disc) / r_dot_r_2D; // 2
			const float tmax = (-p_dor_r_2D + disc) / r_dot_r_2D; // 2

			float2 tz;
			if (parametersOfIntersection_1D(tz, p.z, r.z) == true)
			{
				ts.x = max(tmin, tz.x);
				ts.y = min(tmax, tz.y);
			}
			else
				return false;
		}
		else if (r.x == 0.0f && r.y == 0.0f && p.x * p.x + p.y * p.y <= 1.0f)
		{
			return parametersOfIntersection_1D(ts, p.z, r.z);
		}
		else
			return false;
	}
	else if (solid->type == d_CYLINDER_X) // ellipsoidal cross sections parallel to x-y axis
	{
		const float r_dot_r_2D = r_dot_r - r.x * r.x;
		float disc = (p.z * r.z + p.y * r.y) * (p.z * r.z + p.y * r.y) - r_dot_r_2D * (p.z * p.z + p.y * p.y - 1.0f);
		if (disc > 0.0f)
		{
			disc = sqrtf(disc);
			const float tmin = (-(p.z * r.z + r.y * p.y) - disc) / r_dot_r_2D;
			const float tmax = (-(p.z * r.z + r.y * p.y) + disc) / r_dot_r_2D;

			float2 tz;
			if (parametersOfIntersection_1D(tz, p.x, r.x) == true)
			{
				ts.x = max(tmin, tz.x);
				ts.y = min(tmax, tz.y);
			}
			else
				return false;
		}
		else if (r.y == 0.0f && r.z == 0.0f && p.y * p.y + p.z * p.z <= 1.0f)
		{
			return parametersOfIntersection_1D(ts, p.x, r.x);
		}
		else
			return false;
	}
	else if (solid->type == d_CYLINDER_Y) // ellipsoidal cross sections parallel to x-y axis
	{
		const float r_dot_r_2D = r_dot_r - r.y * r.y;
		float disc = (p.x * r.x + p.z * r.z) * (p.x * r.x + p.z * r.z) - r_dot_r_2D * (p.x * p.x + p.z * p.z - 1.0f);
		if (disc > 0.0f)
		{
			disc = sqrtf(disc);
			const float tmin = (-(p.x * r.x + r.z * p.z) - disc) / r_dot_r_2D;
			const float tmax = (-(p.x * r.x + r.z * p.z) + disc) / r_dot_r_2D;

			float2 tz;
			if (parametersOfIntersection_1D(tz, p.y, r.y) == true)
			{
				ts.x = max(tmin, tz.x);
				ts.y = min(tmax, tz.y);
			}
			else
				return false;
		}
		else if (r.x == 0.0f && r.z == 0.0f && p.x * p.x + p.z * p.z <= 1.0f)
		{
			return parametersOfIntersection_1D(ts, p.y, r.y);
		}
		else
			return false;
	}
	else if (solid->type == d_CONE_Z)
	{
		float a = r.x * r.x + r.y * r.y - r.z * r.z;
		const float b_half = p.x * r.x + p.y * r.y - p.z * r.z;
		const float c = p.x * p.x + p.y * p.y - p.z * p.z;
		float disc = b_half * b_half - a * c;

		if (disc > 0.0f)
		{
			disc = sqrtf(disc);
			float tmin = (-b_half - disc) / a;
			float tmax = (-b_half + disc) / a;
			if (tmin > tmax)
			{
				a = tmin;
				tmin = tmax;
				tmax = a;
			}

			const float theShift = 0.5f * (solid->clipCone.y + solid->clipCone.x);
			const float theScale = 0.5f * (solid->clipCone.y - solid->clipCone.x);

			float2 tz;
			if (parametersOfIntersection_1D(tz, (p.z - theShift) / theScale, r.z / theScale) == true)
			{
				ts.x = max(tmin, tz.x);
				ts.y = min(tmax, tz.y);
			}
			else
				return false;
		}
		else
			return false;
	}
	else if (solid->type == d_CONE_X)
	{
		float a = r.z * r.z + r.y * r.y - r.x * r.x;
		const float b_half = p.z * r.z + p.y * r.y - p.x * r.x;
		const float c = p.z * p.z + p.y * p.y - p.x * p.x;
		float disc = b_half * b_half - a * c;

		if (disc > 0.0f)
		{
			disc = sqrtf(disc);
			float tmin = (-b_half - disc) / a;
			float tmax = (-b_half + disc) / a;
			if (tmin > tmax)
			{
				a = tmin;
				tmin = tmax;
				tmax = a;
			}

			const float theShift = 0.5f * (solid->clipCone.y + solid->clipCone.x);
			const float theScale = 0.5f * (solid->clipCone.y - solid->clipCone.x);

			float2 tz;
			if (parametersOfIntersection_1D(tz, (p.x - theShift) / theScale, r.x / theScale) == true)
			{
				ts.x = max(tmin, tz.x);
				ts.y = min(tmax, tz.y);
			}
			else
				return false;
		}
		else
			return false;
	}
	else if (solid->type == d_CONE_Y)
	{
		float a = r.x * r.x + r.z * r.z - r.y * r.y;
		const float b_half = p.x * r.x + p.z * r.z - p.y * r.y;
		const float c = p.x * p.x + p.z * p.z - p.y * p.y;
		float disc = b_half * b_half - a * c;

		if (disc > 0.0f)
		{
			disc = sqrtf(disc);
			float tmin = (-b_half - disc) / a;
			float tmax = (-b_half + disc) / a;
			if (tmin > tmax)
			{
				a = tmin;
				tmin = tmax;
				tmax = a;
			}

			const float theShift = 0.5f * (solid->clipCone.y + solid->clipCone.x);
			const float theScale = 0.5f * (solid->clipCone.y - solid->clipCone.x);

			float2 tz;
			if (parametersOfIntersection_1D(tz, (p.y - theShift) / theScale, r.y / theScale) == true)
			{
				if (fabs(r.y) > 1.0e-12f)
				{
					bool isInside_0 = false;
					float x_val, y_val, z_val;
					x_val = p.x + tz.x * r.x; x_val *= x_val;
					y_val = p.y + tz.x * r.y; y_val *= y_val;
					z_val = p.z + tz.x * r.z; z_val *= z_val;
					if (x_val + z_val <= y_val)
						isInside_0 = true;

					bool isInside_1 = false;
					x_val = p.x + tz.y * r.x; x_val *= x_val;
					y_val = p.y + tz.y * r.y; y_val *= y_val;
					z_val = p.z + tz.y * r.z; z_val *= z_val;
					if (x_val + z_val <= y_val)
						isInside_1 = true;
					if (isInside_0 == true)
					{
						if (isInside_1 == true)
						{
							//ts.x = max(tmin, tz.x);
							//ts.y = min(tmax, tz.y);
							ts.x = tz.x;
							ts.y = tz.y;
						}
						else
						{
							//insiders: tz.x, tmin, tmax
							if (tz.x <= tmin && tmin <= tz.y)
							{
								ts.x = tz.x;
								ts.y = tmin;
							}
							else
							{
								ts.x = tz.x;
								ts.y = tmax;
							}
						}
					}
					else
					{
						if (isInside_1 == true)
						{
							//insiders: tz.y, tmin, tmax
							if (tz.x <= tmin && tmin <= tz.y)
							{
								ts.x = tmin;
								ts.y = tz.y;
							}
							else
							{
								ts.x = tmax;
								ts.y = tz.y;
							}
						}
						else
						{
							// insiders: tmin, tmax
							if (tz.x <= tmin && tmax <= tz.y)
							{
								ts.x = tmin;
								ts.y = tmax;
							}
							else
							{
								ts.x = -OUT_OF_BOUNDS;
								ts.y = ts.x;
								return false;
							}
						}
					}
				}
				else
				{
					ts.x = tmin;
					ts.y = tmax;
				}
			}
		}
	}
	else
	{
		ts.x = -OUT_OF_BOUNDS;
		ts.y = -OUT_OF_BOUNDS;
		return false;
	}

	if (ts.x >= ts.y)
	{
		ts.x = -OUT_OF_BOUNDS;
		ts.y = ts.x;
		return false;
	}

	return true;
}

__device__ bool intersectionEndPoints(float3& p, float3& r, float2& ts, geometricSolid* solid)
{
    // assume ||r|| == 1 && r != (0,0,1) && axis[i] > 0 for i=0,1,2
    // alpha is rotation around x-y axis, currently there is no rotation for x-z or y-z axes
    float3 q;
    float3 Minv_r;

    if (solid->isRotated == false)
    {
        // Scale; (9,0) ops
        q.x = (p.x - solid->centers.x) / solid->radii.x;
        q.y = (p.y - solid->centers.y) / solid->radii.y;
        q.z = (p.z - solid->centers.z) / solid->radii.z;

        Minv_r.x = r.x / solid->radii.x;
        Minv_r.y = r.y / solid->radii.y;
        Minv_r.z = r.z / solid->radii.z;
    }
    else
    {
        float3 temp;

        // Shift
        q.x = p.x - solid->centers.x;
        q.y = p.y - solid->centers.y;
        q.z = p.z - solid->centers.z;

        // Rotate and Scale; (36, 0) ops
        temp.x = (q.x * solid->A[0 * 3 + 0] + q.y * solid->A[0 * 3 + 1] + q.z * solid->A[0 * 3 + 2]) / solid->radii.x;
        temp.y = (q.x * solid->A[1 * 3 + 0] + q.y * solid->A[1 * 3 + 1] + q.z * solid->A[1 * 3 + 2]) / solid->radii.y;
        temp.z = (q.x * solid->A[2 * 3 + 0] + q.y * solid->A[2 * 3 + 1] + q.z * solid->A[2 * 3 + 2]) / solid->radii.z;

        q.x = temp.x;
        q.y = temp.y;
        q.z = temp.z;

        Minv_r.x = (r.x * solid->A[0 * 3 + 0] + r.y * solid->A[0 * 3 + 1] + r.z * solid->A[0 * 3 + 2]) / solid->radii.x;
        Minv_r.y = (r.x * solid->A[1 * 3 + 0] + r.y * solid->A[1 * 3 + 1] + r.z * solid->A[1 * 3 + 2]) / solid->radii.y;
        Minv_r.z = (r.x * solid->A[2 * 3 + 0] + r.y * solid->A[2 * 3 + 1] + r.z * solid->A[2 * 3 + 2]) / solid->radii.z;
    }

    if (intersectionEndPoints_centeredAndNormalized(q, Minv_r, ts, solid) == false)
        return false;
    if (parametersOfClippingPlaneIntersections(ts, p, r, solid) == false)
    {
        ts.x = -OUT_OF_BOUNDS;
        ts.y = ts.x;
        return false;
    }
    if (ts.y > ts.x)
        return true;
    else
        return false;
}

__device__ void sort(float* v, int count)
{
	for (int i = 0; i < count; i++)
	{
		for (int j = i + 1; j < count; j++)
		{
			if (v[i] > v[j])
			{  // swap?
				const float tmp = v[i];
				v[i] = v[j];
				v[j] = tmp;
			}
		}
	}
}

__device__ float lineIntegral_geometricSolids(float3 p, float3 r, geometricSolid* solids, const int numObjects, float* floatData, int* intData)
{
	//*
	//vector<float> endPoints;
	//vector<int> objectIndices;
	//float* endPoints = (float*)malloc(size_t(2 * numObjects) * sizeof(float));
	//int* objectIndices = (int*)malloc(size_t(numObjects) * sizeof(int));
	//float* intersection_0 = (float*)malloc(size_t(2*numObjects) * sizeof(float));
	//float* intersection_1 = &intersection_0[numObjects];

	float* endPoints = &floatData[0];
	float* intersection_0 = &floatData[2 * numObjects];
	float* intersection_1 = &floatData[3 * numObjects];
	int* objectIndices = intData;

	int count = 0;
	//float* intersection_0 = &intersections[0];
	//float* intersection_1 = &intersections[numObjects];
	for (int i = 0; i < numObjects; i++)
	{
		float2 ts;
		if (intersectionEndPoints(p, r, ts, &solids[i]))
		{
			endPoints[2 * count + 0] = ts.x;
			endPoints[2 * count + 1] = ts.y;
			//endPoints.push_back(ts.x);
			//endPoints.push_back(ts.y);
			intersection_0[i] = ts.x;
			intersection_1[i] = ts.y;
			//objectIndices.push_back(i);
			objectIndices[count] = i;
			count += 1;
			//printf("intersection: %f to %f\n", ts[0], ts[1]);
		}
		else
		{
			//printf("no intersection (%f, %f)\n", ts[0], ts[1]);
			intersection_0[i] = OUT_OF_BOUNDS;
			intersection_1[i] = OUT_OF_BOUNDS;
		}
	}
	float retVal = 0.0f;
	if (count > 0)
	{
		sort(endPoints, 2 * count);
		//sort(endPoints.begin(), endPoints.end());
		for (int i = 0; i < 2*count - 1; i++)
		{
			// Consider the interval (allPoints[i], allPoints[i+1])
			const float midPoint = (endPoints[i + 1] + endPoints[i]) * 0.5f;
			//for (int j = int(objects.size())-1; j >= 0; j--)
			for (int ind = count - 1; ind >= 0; ind--)
			{
				const int j = objectIndices[ind];
				//if (objects[j].val != 0.0)
				{
					// Find which object this interval belongs to
					if (intersection_0[j] <= midPoint && midPoint <= intersection_1[j])
					{
						//if (isnan(arealDensities[j]))
						//	arealDensities[j] = 0.0;
						//arealDensities[j] += objects[j].val * (endPoints[i + 1] - endPoints[i]);
						retVal += solids[j].val * (endPoints[i + 1] - endPoints[i]);
						break;
					}
				}
			}
		}
	}
	//free(endPoints);
	//free(objectIndices);
	//free(intersection_0);
	return retVal;
	//*/
}

__global__ void rayTracingKernel_modular(float* g, const float* phis, geometricSolid* solids, const int numObjects, float* floatData, int* intData, const uint64 ichunk, const int chunkSize, const float* sourcePositions, const float* moduleCenters, const float* rowVectors, const float* colVectors)
{
	//const int i = threadIdx.x + blockIdx.x * blockDim.x;
	//const int j = threadIdx.y + blockIdx.y * blockDim.y;
	//const int k = threadIdx.z + blockIdx.z * blockDim.z;
	const int iprocess = threadIdx.x + blockIdx.x * blockDim.x;

	uint64 ind = ichunk * chunkSize + iprocess;
	int k = ind % d_N_g.z;
	ind = (ind - k) / d_N_g.z;
	int j = ind % d_N_g.y;
	int i = (ind - j) / d_N_g.y;

	//const int k = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= d_N_g.x || j >= d_N_g.y || k >= d_N_g.z)
		return;

	const float* sourcePosition = &sourcePositions[3 * i];
	const float* moduleCenter = &moduleCenters[3 * i];
	const float* v_vec = &rowVectors[3 * i];
	const float* u_vec = &colVectors[3 * i];

	const float v_val = v(j);
	const float u_val = u(k);

	const float3 sourcePos = make_float3(sourcePosition[0], sourcePosition[1], sourcePosition[2]);
	const float3 detPos = make_float3(moduleCenter[0] + u_val * u_vec[0] + v_val * v_vec[0], moduleCenter[1] + u_val * u_vec[1] + v_val * v_vec[1], moduleCenter[2] + u_val * u_vec[2] + v_val * v_vec[2]);

	const float phi = phis[i];
	if (d_oversampling <= 1)
	{
		const float dv = 0.0f;
		const float du = 0.0f;

		const float r_mag_inv = rsqrtf((detPos.x - sourcePos.x) * (detPos.x - sourcePos.x) + (detPos.y - sourcePos.y) * (detPos.y - sourcePos.y) + (detPos.z - sourcePos.z) * (detPos.z - sourcePos.z));
		const float3 r = make_float3((detPos.x - sourcePos.x) * r_mag_inv, (detPos.y - sourcePos.y) * r_mag_inv, (detPos.z - sourcePos.z) * r_mag_inv);

		//const float val = lineIntegral_geometricSolids(sourcePos, r, solids, numObjects, &allIntersections[k*2*numObjects]);
		const float val = lineIntegral_geometricSolids(sourcePos, r, solids, numObjects, &floatData[iprocess * 4 * numObjects], &intData[iprocess * numObjects]);
		g[uint64(i) * uint64(d_N_g.z * d_N_g.y) + uint64(j * d_N_g.z + k)] = val;
	}
	else
	{
		const float T_v_os = d_T_g.y / float(d_oversampling + 1);
		const float T_u_os = d_T_g.z / float(d_oversampling + 1);

		const int os_radius = (d_oversampling - 1) / 2;

		float accum = 0.0;
		for (int j_os = -os_radius; j_os <= os_radius; j_os++)
		{
			const float dv = j_os * T_v_os;

			for (int k_os = -os_radius; k_os <= os_radius; k_os++)
			{
				const float du = k_os * T_u_os;

				const float3 detPos_mod = make_float3(detPos.x + du * u_vec[0] + dv * v_vec[0], detPos.y + du * u_vec[1] + dv * v_vec[1], detPos.z + du * u_vec[2] + dv * v_vec[2]);
				const float r_mag_inv = rsqrtf((detPos_mod.x - sourcePos.x) * (detPos_mod.x - sourcePos.x) + (detPos_mod.y - sourcePos.y) * (detPos_mod.y - sourcePos.y) + (detPos_mod.z - sourcePos.z) * (detPos_mod.z - sourcePos.z));
				const float3 r = make_float3((detPos_mod.x - sourcePos.x) * r_mag_inv, (detPos_mod.y - sourcePos.y) * r_mag_inv, (detPos_mod.z - sourcePos.z) * r_mag_inv);

				accum += expf(-lineIntegral_geometricSolids(sourcePos, r, solids, numObjects, &floatData[iprocess * 4 * numObjects], &intData[iprocess * numObjects]));
			}
		}

		g[uint64(i) * uint64(d_N_g.z * d_N_g.y) + uint64(j * d_N_g.z + k)] = -log(accum / float(d_oversampling * d_oversampling));
	}
}

__global__ void rayTracingKernel(float* g, const float* phis, geometricSolid* solids, const int numObjects, float* floatData, int* intData, const uint64 ichunk, const int chunkSize)
{
    //const int i = threadIdx.x + blockIdx.x * blockDim.x;
    //const int j = threadIdx.y + blockIdx.y * blockDim.y;
    //const int k = threadIdx.z + blockIdx.z * blockDim.z;
	const int iprocess = threadIdx.x + blockIdx.x * blockDim.x;

	uint64 ind = ichunk * chunkSize + iprocess;
	int k = ind % d_N_g.z;
	ind = (ind - k) / d_N_g.z;
	int j = ind % d_N_g.y;
	int i = (ind - j) / d_N_g.y;

	//const int k = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= d_N_g.x || j >= d_N_g.y || k >= d_N_g.z)
        return;

	const float phi = phis[i];
	if (d_oversampling <= 1)
	{
		const float dv = 0.0f;
		const float du = 0.0f;

		const float3 sourcePos = setSourcePosition(phi, i, j, k, dv, du);
		const float3 r = setTrajectory(phi, i, j, k, dv, du);

		//const float val = lineIntegral_geometricSolids(sourcePos, r, solids, numObjects, &allIntersections[k*2*numObjects]);
		const float val = lineIntegral_geometricSolids(sourcePos, r, solids, numObjects, &floatData[iprocess * 4 * numObjects], &intData[iprocess * numObjects]);
		g[uint64(i) * uint64(d_N_g.z * d_N_g.y) + uint64(j * d_N_g.z + k)] = val;
	}
	else
	{
		const float T_v_os = d_T_g.y / float(d_oversampling + 1);
		const float T_u_os = d_T_g.z / float(d_oversampling + 1);

		const int os_radius = (d_oversampling - 1) / 2;

		float accum = 0.0;
		for (int j_os = -os_radius; j_os <= os_radius; j_os++)
		{
			const float dv = j_os * T_v_os;

			for (int k_os = -os_radius; k_os <= os_radius; k_os++)
			{
				const float du = k_os * T_u_os;

				const float3 sourcePos = setSourcePosition(phi, i, j, k, dv, du);
				const float3 r = setTrajectory(phi, i, j, k, dv, du);

				accum += expf(-lineIntegral_geometricSolids(sourcePos, r, solids, numObjects, &floatData[iprocess * 4 * numObjects], &intData[iprocess * numObjects]));
			}
		}

		g[uint64(i) * uint64(d_N_g.z * d_N_g.y) + uint64(j * d_N_g.z + k)] = -log(accum / float(d_oversampling * d_oversampling));
	}
}

void setConstantMemoryGeometryParameters(parameters* params, int oversampling)
{
	hipError_t cudaStatus;

    int4 N_g; float4 T_g; float4 startVal_g;
    setProjectionGPUparams(params, N_g, T_g, startVal_g, true);

	hipMemcpyToSymbol(HIP_SYMBOL(d_oversampling), &oversampling, sizeof(int));

    int geometry = params->geometry;
    int CONE = params->CONE;
    int PARALLEL = params->PARALLEL;
    int FAN = params->FAN;
    int MODULAR = params->MODULAR;
    int CONE_PARALLEL = params->CONE_PARALLEL;
    int detectorType = params->detectorType;
    int FLAT = params->FLAT;
    int CURVED = params->CURVED;
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_geometry), &geometry, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_CONE), &CONE, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_PARALLEL), &PARALLEL, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_FAN), &FAN, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_MODULAR), &MODULAR, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_CONE_PARALLEL), &CONE_PARALLEL, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(d_detectorType), &detectorType, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_FLAT), &FLAT, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_CURVED), &CURVED, sizeof(int));

    float sod = params->sod;
    float sdd = params->sdd;
    float tau = params->tau;
    hipMemcpyToSymbol(HIP_SYMBOL(d_sod), &sod, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_sdd), &sdd, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_tau), &tau, sizeof(float));
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_N_g), &N_g, sizeof(int4));
    hipMemcpyToSymbol(HIP_SYMBOL(d_T_g), &T_g, sizeof(float4));
    hipMemcpyToSymbol(HIP_SYMBOL(d_startVal_g), &startVal_g, sizeof(float4));
}

bool rayTrace_gpu(float* g, parameters* params, phantom* aPhantom, bool data_on_cpu, int oversampling)
{
    if (g == NULL || params == NULL || params->geometryDefined() == false)
        return false;
	oversampling = max(1, min(oversampling, 11));
	if (oversampling % 2 == 0)
		oversampling += 1;
	oversampling = max(1, min(oversampling, 11));

    hipSetDevice(params->whichGPU);
    hipError_t cudaStatus;

	setConstantMemoryGeometryParameters(params, oversampling);

    float* dev_g = 0;
    int4 N_g; float4 T_g; float4 startVal_g;
    setProjectionGPUparams(params, N_g, T_g, startVal_g, true);

    if (data_on_cpu)
    {
        if ((cudaStatus = hipMalloc((void**)&dev_g, params->projectionData_numberOfElements() * sizeof(float))) != hipSuccess)
        {
            fprintf(stderr, "hipMalloc(projections) failed!\n");
        }
    }
    else
        dev_g = g;

    float* dev_phis = copyAngleArrayToGPU(params);

	float* dev_sourcePositions = 0;
	float* dev_moduleCenters = 0;
	float* dev_rowVectors = 0;
	float* dev_colVectors = 0;
	if (params->geometry == parameters::MODULAR)
	{
		if (hipSuccess != hipMalloc((void**)&dev_sourcePositions, 3 * params->numAngles * sizeof(float)))
			fprintf(stderr, "hipMalloc failed!\n");
		if (hipMemcpy(dev_sourcePositions, params->sourcePositions, 3 * params->numAngles * sizeof(float), hipMemcpyHostToDevice))
			fprintf(stderr, "hipMemcpy(sourcePositions) failed!\n");

		if (hipSuccess != hipMalloc((void**)&dev_moduleCenters, 3 * params->numAngles * sizeof(float)))
			fprintf(stderr, "hipMalloc failed!\n");
		if (hipMemcpy(dev_moduleCenters, params->moduleCenters, 3 * params->numAngles * sizeof(float), hipMemcpyHostToDevice))
			fprintf(stderr, "hipMemcpy(moduleCenters) failed!\n");

		if (hipSuccess != hipMalloc((void**)&dev_rowVectors, 3 * params->numAngles * sizeof(float)))
			fprintf(stderr, "hipMalloc failed!\n");
		if (hipMemcpy(dev_rowVectors, params->rowVectors, 3 * params->numAngles * sizeof(float), hipMemcpyHostToDevice))
			fprintf(stderr, "hipMemcpy(rowVectors) failed!\n");

		if (hipSuccess != hipMalloc((void**)&dev_colVectors, 3 * params->numAngles * sizeof(float)))
			fprintf(stderr, "hipMalloc failed!\n");
		if (hipMemcpy(dev_colVectors, params->colVectors, 3 * params->numAngles * sizeof(float), hipMemcpyHostToDevice))
			fprintf(stderr, "hipMemcpy(colVectors) failed!\n");
	}

	int numObjects = int(aPhantom->objects.size());
	geometricSolid* dev_solids = 0;
	geometricSolid* solids = new geometricSolid[numObjects];
	for (int i = 0; i < numObjects; i++)
	{
		solids[i].type = aPhantom->objects[i].type;
		solids[i].centers = make_float3(aPhantom->objects[i].centers[0], aPhantom->objects[i].centers[1], aPhantom->objects[i].centers[2]);
		solids[i].radii = make_float3(aPhantom->objects[i].radii[0], aPhantom->objects[i].radii[1], aPhantom->objects[i].radii[2]);
		solids[i].val = aPhantom->objects[i].val;
		for (int j = 0; j < 9; j++)
			solids[i].A[j] = aPhantom->objects[i].A[j];
		for (int j = 0; j < 6; j++)
		{
			for (int k = 0; k < 4; k++)
				solids[i].clippingPlanes[j][k] = aPhantom->objects[i].clippingPlanes[j][k];
		}
		solids[i].isRotated = aPhantom->objects[i].isRotated;
		solids[i].numClippingPlanes = aPhantom->objects[i].numClippingPlanes;
		solids[i].clipCone.x = aPhantom->objects[i].clipCone[0];
		solids[i].clipCone.y = aPhantom->objects[i].clipCone[1];
	}
	if ((cudaStatus = hipMalloc((void**)&dev_solids, numObjects * sizeof(geometricSolid))) != hipSuccess)
	{
		fprintf(stderr, "hipMalloc(phantom data) failed!\n");
	}
	//if ((cudaStatus = hipMemcpy(dev_g, g, N * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess)
	if ((cudaStatus = hipMemcpy(dev_solids, solids, numObjects * sizeof(geometricSolid), hipMemcpyHostToDevice)) != hipSuccess)
	{
		fprintf(stderr, "failed to copy phantom data to device!\n");
		fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
		fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
	}
	delete[] solids;

	int num_gpu_cores = max(1024, getSPcores(params->whichGPU));
	if (params->projectionData_numberOfElements() < uint64(num_gpu_cores))
		num_gpu_cores = int(params->projectionData_numberOfElements());
	uint64 numChunks = uint64(ceil(double(params->projectionData_numberOfElements()) / double(num_gpu_cores)));
	//printf("number of cores = %d, number of chunks = %d\n", num_gpu_cores, int(numChunks));

	int blockSize = 8;
	int numBlocks = int(ceil(double(num_gpu_cores) / double(blockSize)));
	int numDataCopies = numBlocks * blockSize;

	float* dev_floatData = 0;
	if ((cudaStatus = hipMalloc((void**)&dev_floatData, numDataCopies * 4 * numObjects * sizeof(float))) != hipSuccess)
	{
		fprintf(stderr, "hipMalloc(phantom data) failed!\n");
	}
	int* dev_intData = 0;
	if ((cudaStatus = hipMalloc((void**)&dev_intData, numDataCopies * numObjects * sizeof(int))) != hipSuccess)
	{
		fprintf(stderr, "hipMalloc(phantom data) failed!\n");
	}

	for (uint64 ichunk = 0; ichunk < numChunks; ichunk++)
	{
		if (params->geometry == parameters::MODULAR)
		{
			rayTracingKernel_modular <<< numBlocks, blockSize >>> (dev_g, dev_phis, dev_solids, numObjects, dev_floatData, dev_intData, ichunk, num_gpu_cores, dev_sourcePositions, dev_moduleCenters, dev_rowVectors, dev_colVectors);
		}
		else
		{
			rayTracingKernel <<< numBlocks, blockSize >>> (dev_g, dev_phis, dev_solids, numObjects, dev_floatData, dev_intData, ichunk, num_gpu_cores);
		}
		//for (int j = 0; j < N_g.y; j++)
		//	rayTracingKernel <<< numBlocks, blockSize >>> (dev_g, dev_phis, dev_solids, numObjects, dev_floatData, dev_intData, i, j);
		//float* g, const float* phis, geometricSolid* solids, const int numObjects, float* allIntersections
	}

    // pull result off GPU
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "kernel failed!\n");
        fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
        fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
    }

    if (data_on_cpu)
        pullProjectionDataFromGPU(g, params, dev_g, params->whichGPU);
    else
        g = dev_g;

    // Clean up
    hipFree(dev_phis);
	hipFree(dev_solids);
	hipFree(dev_floatData);
	hipFree(dev_intData);
	if (dev_sourcePositions != 0)
		hipFree(dev_sourcePositions);
	if (dev_moduleCenters != 0)
		hipFree(dev_moduleCenters);
	if (dev_rowVectors != 0)
		hipFree(dev_rowVectors);
	if (dev_colVectors != 0)
		hipFree(dev_colVectors);
    if (data_on_cpu)
    {
        if (dev_g != 0)
            hipFree(dev_g);
    }

    return true;
}
