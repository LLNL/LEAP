#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright 2023-2024 Kyle Champley
// See the LICENSE file for details.
// SPDX-License-Identifier: MIT
//
// LivermorE AI Projector for Computed Tomography (LEAP)
// GPU-based geometric calibration routines
////////////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include "cuda_utils.h"
#include "hip/hip_runtime.h"
#include "leap_defines.h"
#include "log.h"
#include "geometric_calibration.cuh"

#include <iostream>
#include <vector>

//d_data_txt, dev_cost, N, T, startVal, dev_phis, params->sod, params->sdd, params->tau, Delta_t, Delta_s, Delta_tilt
__global__ void consistencyCostKernel(hipTextureObject_t g, float* cost, const int3 N, const float3 T, const float3 startVal, const float* phis, const float sod, const float sdd, const float tau, const float Delta_tilt)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int iv = threadIdx.y + blockIdx.y * blockDim.y;
    //const int m = threadIdx.y + blockIdx.y * blockDim.y;
    //const int n = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || iv >= N.y)
        return;

    const float cos_psi = cos(Delta_tilt);
    const float sin_psi = sin(Delta_tilt);
    const float3 Rpsi_e3 = make_float3(0.0f, -sin_psi, cos_psi);

    const int padding = 1000*0;

    uint64 ind = uint64(i) * uint64(N.y) + uint64(iv);

    const float cos_phi_i = cos(phis[i]);
    const float sin_phi_i = sin(phis[i]);
    const float2 s_i = make_float2(sod * cos_phi_i + tau * sin_phi_i, sod * sin_phi_i - tau * cos_phi_i);

    // if psi == 0, (-sin_phi_i, cos_phi_i, 0)
    const float3 Rphi_Rpsi_e2_i = make_float3(-sin_phi_i * cos_psi, cos_phi_i * cos_psi, sin_psi);

    const float T_u_inv = sdd / T.z;
    const float T_v_inv = sdd / T.y;
    const float u_0 = startVal.z / sdd;
    const float u_end = T.z / sdd * (N.y-1) + u_0;
    const float u_0_edge = u_0 - 0.5f * T.z/sdd;
    const float u_end_edge = u_end + 0.5f * T.z/sdd;
    const float v_0 = startVal.y / sdd;

    const float2 r_left_i = make_float2(-cos_phi_i - sin_phi_i * u_0_edge, -sin_phi_i + cos_phi_i * u_0_edge);
    const float2 r_right_i = make_float2(-cos_phi_i - sin_phi_i * u_end_edge, -sin_phi_i + cos_phi_i * u_end_edge);
    //const float2 r_left_i = make_float2(-cos_phi_i + Rphi_Rpsi_e2_i.x * u_0_edge, -sin_phi_i + Rphi_Rpsi_e2_i.y * u_0_edge);
    //const float2 r_right_i = make_float2(-cos_phi_i + Rphi_Rpsi_e2_i.x * u_end_edge, -sin_phi_i + Rphi_Rpsi_e2_i.y * u_end_edge);

    //u_arg_i = (-sin_phi_i * vox.x + cos_phi_i * vox.y + tau) * v_denom_inv_i;
    //u_arg_i = ((vox.x - tau * sin_phi_i) * Rphi_Rpsi_e2_i.x + (vox.y + tau * cos_phi_i) * Rphi_Rpsi_e2_i.y) * v_denom_inv_i;

    float cost_i = 0.0;
    const float maxAngleDiff = 150.0f * PI / 180.0f;
    const float minAngleDiff = 35.0f * PI / 180.0f;
    for (int j = 0; j < N.x; j++)
    {
        float phi_diff = fabs(phis[i] - phis[j]);
        phi_diff -= floor(phi_diff / (2.0 * PI)) * phi_diff;
        if (j == i || phi_diff > maxAngleDiff || phi_diff < minAngleDiff)
            continue;

        const float cos_phi_j = cos(phis[j]);
        const float sin_phi_j = sin(phis[j]);

        const float3 Rphi_Rpsi_e2_j = make_float3(-sin_phi_j * cos_psi, cos_phi_j * cos_psi, sin_psi);

        const float2 s_j = make_float2(sod * cos_phi_j + tau * sin_phi_j, sod * sin_phi_j - tau * cos_phi_j);
        const float dist_inv = rsqrtf((s_j.x - s_i.x) * (s_j.x - s_i.x) + (s_j.y - s_i.y) * (s_j.y - s_i.y));

        const float2 r_left_j = make_float2(-cos_phi_j - sin_phi_j * u_0_edge, -sin_phi_j + cos_phi_j * u_0_edge);
        const float2 r_right_j = make_float2(-cos_phi_j - sin_phi_j * u_end_edge, -sin_phi_j + cos_phi_j * u_end_edge);
        //const float2 r_left_j = make_float2(-cos_phi_j + Rphi_Rpsi_e2_j.x * u_0_edge, -sin_phi_j + Rphi_Rpsi_e2_j.y * u_0_edge);
        //const float2 r_right_j = make_float2(-cos_phi_j + Rphi_Rpsi_e2_j.x * u_end_edge, -sin_phi_j + Rphi_Rpsi_e2_j.y * u_end_edge);

        const float2 virtual_colVec = make_float2((s_j.x - s_i.x) * dist_inv, (s_j.y - s_i.y) * dist_inv);
        const float2 virtual_normal = make_float2(virtual_colVec.y, -virtual_colVec.x);

        const float s_i_dot_n = s_i.x * virtual_normal.x + s_i.y * virtual_normal.y;
        const float s_j_dot_n = s_j.x * virtual_normal.x + s_j.y * virtual_normal.y;

        
        // calculate D_virt_i, D_virt_j
        const float D_virt_i = fabs(s_i_dot_n);
        const float D_virt_j = fabs(s_j_dot_n);

        const float T_u_virt = T.z * D_virt_i / sdd;
        const float T_u_virt_inv = 1.0f / T_u_virt;
        const float T_v_virt = T.y * D_virt_i / sdd;
        const float u_0_virt = T_u_virt / T.z * startVal.z;
        const float v_0_virt = T_v_virt / T.y * startVal.y;

        const float v_virt = iv * T_v_virt + v_0_virt;

        float t, u_arg_A, u_arg_B;
        // Index range for view i
        t = -s_i_dot_n / (r_left_i.x * virtual_normal.x + r_left_i.y * virtual_normal.y);
        u_arg_A = ((s_i.x + t * r_left_i.x) * virtual_colVec.x + (s_i.y + t * r_left_i.y) * virtual_colVec.y - u_0_virt) * T_u_virt_inv;

        t = -s_i_dot_n / (r_right_i.x * virtual_normal.x + r_right_i.y * virtual_normal.y);
        u_arg_B = ((s_i.x + t * r_right_i.x) * virtual_colVec.x + (s_i.y + t * r_right_i.y) * virtual_colVec.y - u_0_virt) * T_u_virt_inv;
        const int iu_lo_i = int(floor(min(u_arg_A, u_arg_B))) - 2;
        const int iu_hi_i = int(ceil(max(u_arg_A, u_arg_B))) + 2;

        // Index range for view j
        t = -s_j_dot_n / (r_left_j.x * virtual_normal.x + r_left_j.y * virtual_normal.y);
        u_arg_A = ((s_j.x + t * r_left_j.x) * virtual_colVec.x + (s_j.y + t * r_left_j.y) * virtual_colVec.y - u_0_virt) * T_u_virt_inv;
        
        t = -s_j_dot_n / (r_right_j.x * virtual_normal.x + r_right_j.y * virtual_normal.y);
        u_arg_B = ((s_j.x + t * r_right_j.x) * virtual_colVec.x + (s_j.y + t * r_right_j.y) * virtual_colVec.y - u_0_virt) * T_u_virt_inv;
        const int iu_lo_j = int(floor(min(u_arg_A, u_arg_B))) - 2;
        const int iu_hi_j = int(ceil(max(u_arg_A, u_arg_B))) + 2;

        float accum_i = 0.0f;
        for (int iu = iu_lo_i-padding; iu <= iu_hi_i+padding; iu++)
        {
            const float u_virt = iu * T_u_virt + u_0_virt;
            const float3 vox = make_float3(virtual_colVec.x * u_virt, virtual_colVec.y * u_virt, v_virt); // backproject to this point

            const float integrandWeight_i = rsqrtf(D_virt_i * D_virt_i + v_virt * v_virt + u_virt * u_virt);

            const float v_denom_inv_i = 1.0f / (sod - cos_phi_i * vox.x - sin_phi_i * vox.y);
            float u_arg_i, v_arg_i;
            if (Delta_tilt == 0.0f)
            {
                u_arg_i = (-sin_phi_i * vox.x + cos_phi_i * vox.y + tau) * v_denom_inv_i;
                v_arg_i = vox.z * v_denom_inv_i;
            }
            else
            {
                u_arg_i = ((vox.x - tau * sin_phi_i) * Rphi_Rpsi_e2_i.x + (vox.y + tau * cos_phi_i) * Rphi_Rpsi_e2_i.y) * v_denom_inv_i;
                v_arg_i = ((vox.x - tau * sin_phi_i) * Rpsi_e3.x + (vox.y + tau * cos_phi_i) * Rpsi_e3.y) * v_denom_inv_i;
            }

            accum_i += integrandWeight_i * tex3D<float>(g, (u_arg_i - u_0) * T_u_inv + 0.5f, (v_arg_i - v_0) * T_v_inv + 0.5f, i + 0.5f);
        }

        float accum_j = 0.0f;
        for (int iu = iu_lo_j-padding; iu <= iu_hi_j+padding; iu++)
        {
            const float u_virt = iu * T_u_virt + u_0_virt;
            const float3 vox = make_float3(virtual_colVec.x * u_virt, virtual_colVec.y * u_virt, v_virt); // backproject to this point

            const float integrandWeight_j = rsqrtf(D_virt_j * D_virt_j + v_virt * v_virt + u_virt * u_virt);

            const float v_denom_inv_j = 1.0f / (sod - cos_phi_j * vox.x - sin_phi_j * vox.y);
            //const float u_arg_j = (-sin_phi_j * vox.x + cos_phi_j * vox.y + tau) * v_denom_inv_j;
            //const float v_arg_j = vox.z * v_denom_inv_j;
            float u_arg_j, v_arg_j;
            if (Delta_tilt == 0.0f)
            {
                u_arg_j = (-sin_phi_j * vox.x + cos_phi_j * vox.y + tau) * v_denom_inv_j;
                v_arg_j = vox.z * v_denom_inv_j;
            }
            else
            {
                u_arg_j = ((vox.x - tau * sin_phi_j) * Rphi_Rpsi_e2_j.x + (vox.y + tau * cos_phi_j) * Rphi_Rpsi_e2_j.y) * v_denom_inv_j;
                v_arg_j = ((vox.x - tau * sin_phi_j) * Rpsi_e3.x + (vox.y + tau * cos_phi_j) * Rpsi_e3.y) * v_denom_inv_j;
            }

            accum_j += integrandWeight_j * tex3D<float>(g, (u_arg_j - u_0) * T_u_inv + 0.5f, (v_arg_j - v_0) * T_v_inv + 0.5f, j + 0.5f);
        }
        cost_i += (accum_i - accum_j) * (accum_i - accum_j);
    }

    cost[ind] = cost_i;
}

float consistencyCost(float* g, parameters* params, bool data_on_cpu, float Delta_centerRow, float Delta_centerCol, float Delta_tau, float Delta_tilt)
{
    if (g == NULL || params == NULL)
        return -1.0;
    /*
    if (data_on_cpu == false) // FIXME
    {
        LOG(logERROR, "geometric_calibration", "consistencyCost") << "Currently only implemented for data on the CPU!" << std::endl;
        return -1.0;
    }
    //*/
    if (params->geometry != parameters::CONE || params->detectorType != parameters::FLAT || params->helicalPitch != 0.0)
    {
        LOG(logERROR, "geometric_calibration", "consistencyCost") << "Consistency metric only works for axial flat panel cone-beam geometries!" << std::endl;
        return -1.0;
    }

    // find projections spaced by 40 degrees
    float angularSeparation = 40.0 * PI / 180.0;
    //float angularSeparation = 20.0 * PI / 180.0;
    std::vector<int> proj_inds;
    int ind = 0;
    float phi_cur = params->phis[ind];
    proj_inds.push_back(ind);
    for (int i = 1; i < params->numAngles; i++)
    {
        float phi_diff = fabs(phi_cur - params->phis[i]);
        phi_diff -= floor(phi_diff / (2.0 * PI)) * phi_diff;
        if (phi_diff >= angularSeparation)
        {
            proj_inds.push_back(i);
            phi_cur = params->phis[i];
        }
    }
    int numAngles_subset = int(proj_inds.size());
    if (numAngles_subset <= 1)
    {
        LOG(logERROR, "geometric_calibration", "consistencyCost") << "Insufficient angular coverage!" << std::endl;
        return -1.0;
    }

    hipError_t cudaStatus;
    hipSetDevice(params->whichGPU);

    uint64 projectionSize = uint64(params->numRows) * uint64(params->numCols);

    float* dev_g_subset = 0;
    if (hipSuccess != (cudaStatus = hipMalloc((void**)&dev_g_subset, uint64(numAngles_subset) * projectionSize * sizeof(float))))
    {
        fprintf(stderr, "hipMalloc failed!\n");
        printf("hipMalloc Error: %s\n", hipGetErrorString(cudaStatus));
        return -1.0;
    }

    float* phis = new float[numAngles_subset];
    for (int ind = 0; ind < numAngles_subset; ind++)
    {
        float* dev_g_subset_ind = &dev_g_subset[uint64(ind)* projectionSize];
        float* g_ind = &g[uint64(proj_inds[ind]) * projectionSize];
        if (data_on_cpu)
        {
            if ((cudaStatus = hipMemcpy(dev_g_subset_ind, g_ind, projectionSize * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess)
            {
                fprintf(stderr, "hipMemcpy(projection) failed!\n");
                printf("hipMemcpy Error: %s\n", hipGetErrorString(cudaStatus));
                hipFree(dev_g_subset);
                delete[] phis;
                return -1.0;
            }
        }
        else
        {
            if ((cudaStatus = hipMemcpy(dev_g_subset_ind, g_ind, projectionSize * sizeof(float), hipMemcpyDeviceToDevice)) != hipSuccess)
            {
                fprintf(stderr, "hipMemcpy(projection) failed!\n");
                printf("hipMemcpy Error: %s\n", hipGetErrorString(cudaStatus));
                hipFree(dev_g_subset);
                delete[] phis;
                return -1.0;
            }
        }
        phis[ind] = params->phis[proj_inds[ind]];
    }
    float* dev_phis = copy1DdataToGPU(phis, numAngles_subset, params->whichGPU);
    delete[] phis;

    bool normalizeConeAndFanCoordinateFunctions_save = params->normalizeConeAndFanCoordinateFunctions;
    params->normalizeConeAndFanCoordinateFunctions = false;
    int3 N = make_int3(numAngles_subset, params->numRows, params->numCols);
    float3 T = make_float3(params->T_phi(), params->pixelHeight, params->pixelWidth);
    float3 startVal = make_float3(params->phis[0], params->v(0) - Delta_centerRow * params->pixelHeight, params->u(0) - Delta_centerCol * params->pixelWidth);
    float tau = params->tau + Delta_tau;
    params->normalizeConeAndFanCoordinateFunctions = normalizeConeAndFanCoordinateFunctions_save;

    // Copy to texture
    // FIXME: should copy directly from CPU to 3D array
    hipTextureObject_t d_data_txt = NULL;
    hipArray* d_data_array = loadTexture(d_data_txt, dev_g_subset, N, false, true);
    
    // Reuse dev_g_subset for cost values
    float* dev_cost = dev_g_subset;
    setToConstant(dev_g_subset, 0.0, N, params->whichGPU);
    
    dim3 dimBlock(min(8, N.x), min(8, N.y));
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))));
    //dim3 dimBlock = setBlockSize(N);
    //dim3 dimGrid = setGridSize(N, dimBlock);

    consistencyCostKernel <<< dimGrid, dimBlock >>> (d_data_txt, dev_cost, N, T, startVal, dev_phis, params->sod, params->sdd, tau, Delta_tilt*PI/180.0);
    //float retVal = sum(dev_g_subset, N, params->whichGPU);
    float retVal = sum(dev_g_subset, make_int3(N.x, N.y, 1), params->whichGPU);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "kernel failed!\n");
        fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
        fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
    }

    // Clean up
    hipFreeArray(d_data_array);
    hipDestroyTextureObject(d_data_txt);
    hipFree(dev_g_subset);
    hipFree(dev_phis);

    return retVal;
}
