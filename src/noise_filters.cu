#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright 2023-2024 Kyle Champley
// See the LICENSE file for details.
// SPDX-License-Identifier: MIT
//
// LivermorE AI Projector for Computed Tomography (LEAP)
// CUDA-based thresholded median filter and 3D low pass filter calculations
////////////////////////////////////////////////////////////////////////////////
#include "noise_filters.cuh"

#include <math.h>

#include "cuda_utils.h"
#include "hip/hip_runtime.h"
//#include ""
#include "parameters.h"

__constant__ bool d_DO_HIGH_PASS;

__global__ void azimuthalBlurKernel(float* f, float* f_filtered, const int3 N, const float3 T, const float3 startVal, const int N_phi_max, const float filterWidth, const float rFOVsq)
{
    // return;
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;

    const float x = i * T.x + startVal.x;
    const float y = j * T.y + startVal.y;

    if (x * x + y * y > rFOVsq)
    {
        f[uint64(k) * uint64(N.x * N.y) + uint64(j * N.x + i)] = 0.0f;
        return;
    }

    // const float z = k * T.z + startVal.z;

    const float r = sqrt(x * x + y * y);
    const float one_over_Tx = 1.0f / T.x;
    int N_phi;
    float val = 0.0f;

    const float* f_slice = &f[uint64(k) * uint64(N.x * N.y)];

    const int N_xy = N.x * N.y;

    // do filtering
    if (filterWidth >= 360.0f)
    {
        float T_phi = atan(T.x / r);
        N_phi = max(4, min(N_phi_max, 2 * (int)(ceil(3.141592653589793f / T_phi))));
        T_phi = 2.0f * 3.141592653589793f / ((float)N_phi);
        for (int l = 0; l < N_phi; l += 2)
        {
            const float phi = T_phi * l + 0.5f * T_phi;
            const int ix_A = int(0.5f + (r * cos(phi) - startVal.x) * one_over_Tx);
            const int iy_A = int(0.5f + (r * sin(phi) - startVal.y) * one_over_Tx);
            const int ix_B = int(0.5f + (r * cos(phi + T_phi) - startVal.x) * one_over_Tx);
            const int iy_B = int(0.5f + (r * sin(phi + T_phi) - startVal.y) * one_over_Tx);

            const int ind_A = iy_A * N.x + ix_A;
            const int ind_B = iy_B * N.x + ix_B;
            if (0 <= ind_A && ind_A < N_xy)
                val += f_slice[ind_A];
            if (0 <= ind_B && ind_B < N_xy)
                val += f_slice[ind_B];
            // val += read_imagef(f, sampler, (float2)((r * sin(phi) - clf_y_0(f_info)) * one_over_Tx + 0.5f, (r *
            // cos(phi) - clf_x_0(f_info)) * one_over_Tx + 0.5f))
            //+ read_imagef(f, sampler, (float2)((r * sin(phi + T_phi) - clf_y_0(f_info)) * one_over_Tx + 0.5f, (r *
            //cos(phi + T_phi) - clf_x_0(f_info)) * one_over_Tx + 0.5f));
        }
    }
    else
    {
        float T_phi = atan(T.x / r);
        N_phi = max(4, min(N_phi_max, 2 * (int)(ceil((3.141592653589793f / 360.0f) * filterWidth / T_phi))));
        T_phi = (3.141592653589793f / 180.0f) * filterWidth / ((float)N_phi);
        const float psi = atan2(y, x) + 0.5f * T_phi * N_phi;
        for (int l = 0; l < N_phi; l += 2)
        {
            const float phi = T_phi * l + 0.5f * T_phi - psi;
            const int ix_A = int(0.5f + (r * cos(phi) - startVal.x) * one_over_Tx);
            const int iy_A = int(0.5f + (-r * sin(phi) - startVal.y) * one_over_Tx);
            const int ix_B = int(0.5f + (r * cos(phi + T_phi) - startVal.x) * one_over_Tx);
            const int iy_B = int(0.5f + (-r * sin(phi + T_phi) - startVal.y) * one_over_Tx);
            const int ind_A = iy_A * N.x + ix_A;
            const int ind_B = iy_B * N.x + ix_B;
            if (0 <= ind_A && ind_A < N_xy)
                val += f_slice[ind_A];
            if (0 <= ind_B && ind_B < N_xy)
                val += f_slice[ind_B];

            // val += read_imagef(f, sampler, (float2)((-r * sin(phi) - clf_y_0(f_info)) * one_over_Tx + 0.5f, (r *
            // cos(phi) - clf_x_0(f_info)) * one_over_Tx + 0.5f))
            //+ read_imagef(f, sampler, (float2)((-r * sin(phi + T_phi) - clf_y_0(f_info)) * one_over_Tx + 0.5f, (r *
            //cos(phi + T_phi) - clf_x_0(f_info)) * one_over_Tx + 0.5f));
        }
    }

    f_filtered[uint64(k) * uint64(N.x * N.y) + uint64(j * N.x + i)] = val / ((float)N_phi);
}

__global__ void badPixelCorrectionKernel(float* g, float* badPixelMap, const int3 N, const int windowRadius)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;

    if (badPixelMap[j*N.z+k] != 1.0f) // pixel is good; do nothing
        return;

    uint64 iProj = uint64(i) * uint64(N.y) * uint64(N.z);
    float* aProj = &g[iProj];

    float v[49];
    int ind = 0;
    for (int dj = -windowRadius; dj <= windowRadius; dj++)
    {
        const int j_shift = max(0, min(j + dj, N.y - 1));
        for (int dk = -windowRadius; dk <= windowRadius; dk++)
        {
            const int k_shift = max(0, min(k + dk, N.z - 1));
            if (badPixelMap[j_shift * N.z + k_shift] != 1.0f) // pixel is good, store it
            {
                v[ind] = aProj[j_shift * N.z + k_shift];
                ind += 1;
            }
        }
    }

    if (ind == 1)
    {
        aProj[j * N.z + k] = v[0];
    }
    else if (ind == 2)
    {
        aProj[j * N.z + k] = 0.5f * (v[0] + v[1]);
    }
    else if (ind > 2)
    {
        // 3 ==> 2
        // 4 ==> 3
        // 5 ==> 3
        // 6 ==> 4 (need 2 and 3)
        // 7 ==> 4
        // 8 ==> 5
        // 9 ==> 5
        const int ind_mid = (ind - (ind % 2)) / 2 + 1;

        // bubble-sort for first half of samples
        for (int i = 0; i < ind_mid; i++)
        {
            for (int j = i + 1; j < ind; j++)
            {
                if (v[i] > v[j])
                {  // swap?
                    const float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }
        if (ind % 2 == 0)
            aProj[j * N.z + k] = 0.5f * (v[ind_mid - 1] + v[ind_mid - 2]);
        else
            aProj[j * N.z + k] = v[ind_mid-1];
    }
}

__global__ void medianFilter2DKernel(float* f, float* f_filtered, const int3 N, const float threshold, const int windowRadius, const float signalThreshold)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;

    uint64 i_slice = uint64(i) * uint64(N.y) * uint64(N.z);
    float* f_slice = &f[i_slice];
    if (windowRadius == 1)
    {
        float v[9];
        int ind = 0;
        for (int dj = -windowRadius; dj <= windowRadius; dj++)
        {
            const int j_shift = max(0, min(j + dj, N.y - 1));
            for (int dk = -windowRadius; dk <= windowRadius; dk++)
            {
                const int k_shift = max(0, min(k + dk, N.z - 1));
                v[ind] = f_slice[j_shift * N.z + k_shift];
                ind += 1;
            }
        }
        const float curVal = v[4];
        if (signalThreshold > 0.0f && curVal > signalThreshold)
        {
            f_filtered[i_slice + uint64(j * N.z + k)] = curVal;
            return;
        }

        // bubble-sort for first 5 samples
        for (int i = 0; i < 5; i++)
        {
            for (int j = i + 1; j < 9; j++)
            {
                if (v[i] > v[j])
                {  // swap?
                    const float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }
        if (fabs(curVal - v[4]) >= threshold * fabs(v[4]))
            f_filtered[i_slice + uint64(j * N.z + k)] = v[4];
        else
            f_filtered[i_slice + uint64(j * N.z + k)] = curVal;
    }
    else if (windowRadius == 2)
    {
        float v[25];
        int ind = 0;
        for (int dj = -windowRadius; dj <= windowRadius; dj++)
        {
            const int j_shift = max(0, min(j + dj, N.y - 1));
            for (int dk = -windowRadius; dk <= windowRadius; dk++)
            {
                const int k_shift = max(0, min(k + dk, N.z - 1));
                v[ind] = f_slice[j_shift * N.z + k_shift];
                ind += 1;
            }
        }
        const float curVal = v[12];
        if (signalThreshold > 0.0f && curVal > signalThreshold)
        {
            f_filtered[i_slice + uint64(j * N.z + k)] = curVal;
            return;
        }

        // bubble-sort for first 13 samples
        for (int i = 0; i < 13; i++)
        {
            for (int j = i + 1; j < 25; j++)
            {
                if (v[i] > v[j])
                {  // swap?
                    const float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }
        if (fabs(curVal - v[12]) >= threshold * fabs(v[12]))
            f_filtered[i_slice + uint64(j * N.z + k)] = v[12];
        else
            f_filtered[i_slice + uint64(j * N.z + k)] = curVal;
    }
    else if (windowRadius == 3)
    {
        float v[49];
        int ind = 0;
        for (int dj = -windowRadius; dj <= windowRadius; dj++)
        {
            const int j_shift = max(0, min(j + dj, N.y - 1));
            for (int dk = -windowRadius; dk <= windowRadius; dk++)
            {
                const int k_shift = max(0, min(k + dk, N.z - 1));
                v[ind] = f_slice[j_shift * N.z + k_shift];
                ind += 1;
            }
        }
        const float curVal = v[24];
        if (signalThreshold > 0.0f && curVal > signalThreshold)
        {
            f_filtered[i_slice + uint64(j * N.z + k)] = curVal;
            return;
        }

        // bubble-sort for first 25 samples
        for (int i = 0; i < 25; i++)
        {
            for (int j = i + 1; j < 49; j++)
            {
                if (v[i] > v[j])
                {  // swap?
                    const float tmp = v[i];
                    v[i] = v[j];
                    v[j] = tmp;
                }
            }
        }
        if (fabs(curVal - v[24]) >= threshold * fabs(v[24]))
            f_filtered[i_slice + uint64(j * N.z + k)] = v[24];
        else
            f_filtered[i_slice + uint64(j * N.z + k)] = curVal;
    }
}

__global__ void meanFilterKernel(float* f, float* f_filtered, int3 N, const int r, int sliceStart, int sliceEnd)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;
    if (i < sliceStart || i > sliceEnd)
    {
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = 0.0f;
        return;
    }

    const int di_min = -min(i, r);
    const int di_max = min(N.x - 1 - i, r);

    const int dj_min = -min(j, r);
    const int dj_max = min(N.y - 1 - j, r);

    const int dk_min = -min(k, r);
    const int dk_max = min(N.z - 1 - k, r);

    float x = 0.0f;
    for (int di = di_min; di <= di_max; di++)
    {
        for (int dj = dj_min; dj <= dj_max; dj++)
        {
            for (int dk = dk_min; dk <= dk_max; dk++)
            {
                x += f[uint64(i + di) * uint64(N.z * N.y) + uint64((j + dj) * N.z + (k + dk))];
            }
        }
    }
    f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = x / float((di_max - di_min + 1) * (dj_max - dj_min + 1) * (dk_max - dk_min + 1));
}

__global__ void varianceFilterKernel(float* f, float* f_filtered, int3 N, const int r, int sliceStart, int sliceEnd)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;
    if (i < sliceStart || i > sliceEnd)
    {
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = 0.0f;
        return;
    }

    const int di_min = -min(i, r);
    const int di_max = min(N.x - 1 - i, r);

    const int dj_min = -min(j, r);
    const int dj_max = min(N.y - 1 - j, r);

    const int dk_min = -min(k, r);
    const int dk_max = min(N.z - 1 - k, r);

    const float weight = 1.0f / float((di_max - di_min + 1) * (dj_max - dj_min + 1) * (dk_max - dk_min + 1));

    float x = 0.0f;
    float xx = 0.0f;
    for (int di = di_min; di <= di_max; di++)
    {
        for (int dj = dj_min; dj <= dj_max; dj++)
        {
            for (int dk = dk_min; dk <= dk_max; dk++)
            {
                const float curVal = f[uint64(i + di) * uint64(N.z * N.y) + uint64((j + dj) * N.z + (k + dk))];
                x += curVal;
                xx += curVal * curVal;
            }
        }
    }

    const float meanI = x * weight;
    const float varI = xx * weight - meanI * meanI;

    f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = varI;
}

__global__ void medianFilterKernel(float* f, float* f_filtered, int3 N, float threshold, const float signalThreshold, int sliceStart, int sliceEnd)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;
    if (i < sliceStart || i > sliceEnd)
    {
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = 0.0f;
        return;
    }

    float v[27];
    int ind = 0;
    for (int di = -1; di <= 1; di++)
    {
        const int i_shift = max(0, min(i + di, N.x - 1));
        for (int dj = -1; dj <= 1; dj++)
        {
            const int j_shift = max(0, min(j + dj, N.y - 1));
            for (int dk = -1; dk <= 1; dk++)
            {
                const int k_shift = max(0, min(k + dk, N.z - 1));
                v[ind] = f[uint64(i_shift) * uint64(N.y * N.z) + uint64(j_shift * N.z + k_shift)];
                ind += 1;
            }
        }
    }
    const float curVal = v[13];
    if (signalThreshold > 0.0f && curVal > signalThreshold)
    {
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = curVal;
        return;
    }

    // bubble-sort for first 14 samples
    for (int i = 0; i < 14; i++)
    {
        for (int j = i + 1; j < 27; j++)
        {
            if (v[i] > v[j])
            {  // swap?
                const float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }
    // fabs(curVal-v[13])/fabs(curVal) > threshold
    if (fabs(curVal - v[13]) >= threshold * fabs(v[13]))
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = v[13];
    else
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = curVal;
}

__global__ void medianFilterKernel_5x5(float* f, float* f_filtered, int3 N, float threshold, const float signalThreshold, int sliceStart, int sliceEnd)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;
    if (i < sliceStart || i > sliceEnd)
    {
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = 0.0f;
        return;
    }


    //f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = f[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)];
    //return;

    /*
    float v[125];
    int ind = 0;
    for (int di = -2; di <= 2; di++)
    {
        const int i_shift = max(0, min(i + di, N.x - 1));
        for (int dj = -2; dj <= 2; dj++)
        {
            const int j_shift = max(0, min(j + dj, N.y - 1));
            for (int dk = -2; dk <= 2; dk++)
            {
                const int k_shift = max(0, min(k + dk, N.z - 1));
                v[ind] = f[uint64(i_shift) * uint64(N.y * N.z) + uint64(j_shift * N.z + k_shift)];
                ind += 1;
            }
        }
    }
    const float curVal = v[62];

    // bubble-sort for first 63 samples
    for (int i = 0; i < 63; i++)
    {
        for (int j = i + 1; j < 125; j++)
        {
            if (v[i] > v[j])
            {  // swap?
                const float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }
    // fabs(curVal-v[62])/fabs(curVal) > threshold
    if (fabs(curVal - v[62]) >= threshold * fabs(v[62]))
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = v[62];
    else
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = curVal;
    //*/

    //*
    float v[75];
    int ind = 0;
    for (int di = -1; di <= 1; di++)
    {
        const int i_shift = max(0, min(i + di, N.x - 1));
        for (int dj = -2; dj <= 2; dj++)
        {
            const int j_shift = max(0, min(j + dj, N.y - 1));
            for (int dk = -2; dk <= 2; dk++)
            {
                const int k_shift = max(0, min(k + dk, N.z - 1));
                v[ind] = f[uint64(i_shift) * uint64(N.y * N.z) + uint64(j_shift * N.z + k_shift)];
                ind += 1;
            }
        }
    }
    const float curVal = v[37];
    if (signalThreshold > 0.0f && curVal > signalThreshold)
    {
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = curVal;
        return;
    }

    // bubble-sort for first 38 samples
    for (int i = 0; i < 38; i++)
    {
        for (int j = i + 1; j < 75; j++)
        {
            if (v[i] > v[j])
            {  // swap?
                const float tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
        }
    }
    // fabs(curVal-v[37])/fabs(curVal) > threshold
    if (fabs(curVal - v[37]) >= threshold * fabs(v[37]))
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = v[37];
    else
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = curVal;
    //*/
}

__global__ void BlurFilterKernel(float* f, float* f_filtered, int3 N, float FWHM, const int sliceStart, const int sliceEnd)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;
    if (i < sliceStart || i > sliceEnd)
    {
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = 0.0f;
        return;
    }

    const int pixelRadius = int(floor(FWHM));
    const float denom = 1.0f / FWHM;

    float val = 0.0f;
    float sum = 0.0f;
    for (int di = -pixelRadius; di <= pixelRadius; di++)
    {
        const int i_shift = max(0, min(i + di, N.x - 1));
        //if (i + di < 0 || i + di > N.x - 1)
        //    continue;
        for (int dj = -pixelRadius; dj <= pixelRadius; dj++)
        {
            const int j_shift = max(0, min(j + dj, N.y - 1));
            //if (j + dj < 0 || j + dj > N.y - 1)
            //    continue;
            for (int dk = -pixelRadius; dk <= pixelRadius; dk++)
            {
                const int k_shift = max(0, min(k + dk, N.z - 1));
                //if (k + dk < 0 || k + dk > N.z - 1)
                //    continue;

                const float theWeight = 0.5f +
                    0.5f * cosf(3.141592653589793f* min(sqrtf(float(di * di + dj * dj + dk * dk)) * denom, 1.0f));

                if (theWeight > 0.0001f)
                {
                    val += theWeight * f[uint64(i_shift) * uint64(N.y * N.z) + uint64(j_shift * N.z + k_shift)];
                    sum += theWeight;
                }
            }
        }
    }

    if (d_DO_HIGH_PASS)
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = f[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] - val / sum;
    else
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = val / sum;
}

__global__ void BlurFilter2DKernel(float* f, float* f_filtered, int3 N, float FWHM)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;

    //const float sigma = FWHM / (2.0f * sqrt(2.0f * log(2.0f)));
    // FWHM = 2*sqrt(2*log(2))*sigma
    //const int pixelRadius = int(ceil(sqrt(2.0f * log(10.0f)) * sigma));
    //const float denom = 1.0f / (2.0f * sigma * sigma);
    const int pixelRadius = int(floor(FWHM));
    const float denom = 1.0f / FWHM;

    float val = 0.0f;
    float sum = 0.0f;

    float* f_slice = &f[uint64(i) * uint64(N.y * N.z)];
    for (int dj = -pixelRadius; dj <= pixelRadius; dj++)
    {
        const int j_shift = max(0, min(j + dj, N.y - 1));
        //const float j_dist_sq = float((j - j_shift) * (j - j_shift));
        for (int dk = -pixelRadius; dk <= pixelRadius; dk++)
        {
            const int k_shift = max(0, min(k + dk, N.z - 1));
            //const float k_dist_sq = float((k - k_shift) * (k - k_shift));

            //const float theWeight = exp(-denom * (j_dist_sq + k_dist_sq));

            const float theWeight = 0.5f +
                0.5f * cosf(3.141592653589793f * min(sqrtf(float(dj * dj + dk * dk)) * denom, 1.0f));

            if (theWeight > 0.0001f)
            {
                val += theWeight * f_slice[uint64(j_shift * N.z + k_shift)];
                sum += theWeight;
            }
        }
    }

    if (d_DO_HIGH_PASS)
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = f[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] - val / sum;
    else
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = val / sum;
}

__global__ void BlurFilter1DKernel(float* f, float* f_filtered, int3 N, float FWHM)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;

    //const float sigma = FWHM / (2.0f * sqrt(2.0f * log(2.0f)));
    // FWHM = 2*sqrt(2*log(2))*sigma
    //const int pixelRadius = int(ceil(sqrt(2.0f * log(10.0f)) * sigma));
    //const float denom = 1.0f / (2.0f * sigma * sigma);
    const int pixelRadius = int(floor(FWHM));
    const float denom = 1.0f / FWHM;

    float val = 0.0;
    float sum = 0.0;
    for (int di = -pixelRadius; di <= pixelRadius; di++)
    {
        const int i_shift = max(0, min(i + di, N.x - 1));

        //const float theWeight = exp(-denom * float((i - i_shift) * (i - i_shift)));
        const float theWeight = 0.5f +
            0.5f * cosf(3.141592653589793f * min(fabsf(di) * denom, 1.0f));

        if (theWeight > 0.0001f)
        {
            val += theWeight * f[uint64(i_shift) * uint64(N.y * N.z) + uint64(j * N.z + k)];
            sum += theWeight;
        }
    }

    if (d_DO_HIGH_PASS)
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = f[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] - val / sum;
    else
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = val / sum;
}

//########################################################################################################################################################
__global__ void BlurFilterKernel_txt(hipTextureObject_t f, float* f_filtered, int3 N, float FWHM, const int sliceStart, const int sliceEnd)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;
    if (i < sliceStart || i > sliceEnd)
    {
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = 0.0f;
        return;
    }

    const int pixelRadius = int(floor(FWHM));
    const float denom = 1.0f / FWHM;

    float val = 0.0f;
    float sum = 0.0f;
    for (int di = -pixelRadius; di <= pixelRadius; di++)
    {
        const int i_shift = max(0, min(i + di, N.x - 1));
        //const int i_shift = i + di;
        //if (i + di < 0 || i + di > N.x - 1)
        //    continue;
        for (int dj = -pixelRadius; dj <= pixelRadius; dj++)
        {
            const int j_shift = max(0, min(j + dj, N.y - 1));
            //const int j_shift = j + dj;
            //if (j + dj < 0 || j + dj > N.y - 1)
            //    continue;
            for (int dk = -pixelRadius; dk <= pixelRadius; dk++)
            {
                const int k_shift = max(0, min(k + dk, N.z - 1));
                //const int k_shift = k + dk;
                //if (k + dk < 0 || k + dk > N.z - 1)
                //    continue;

                const float theWeight = 0.5f +
                    0.5f * cosf(3.141592653589793f * min(sqrtf(float(di * di + dj * dj + dk * dk)) * denom, 1.0f));

                if (theWeight > 0.0001f)
                {
                    //val += theWeight * f[uint64(i_shift) * uint64(N.y * N.z) + uint64(j_shift * N.z + k_shift)];
                    val += theWeight * tex3D<float>(f, k_shift, j_shift, i_shift);
                    sum += theWeight;
                }
            }
        }
    }

    if (d_DO_HIGH_PASS)
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = tex3D<float>(f, k, j, i) - val / sum;
    else
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = val / sum;
}

__global__ void BlurFilter2DKernel_txt(hipTextureObject_t f, float* f_filtered, int3 N, float FWHM)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;

    //const float sigma = FWHM / (2.0f * sqrt(2.0f * log(2.0f)));
    // FWHM = 2*sqrt(2*log(2))*sigma
    //const int pixelRadius = int(ceil(sqrt(2.0f * log(10.0f)) * sigma));
    //const float denom = 1.0f / (2.0f * sigma * sigma);
    const int pixelRadius = int(floor(FWHM));
    const float denom = 1.0f / FWHM;

    float val = 0.0f;
    float sum = 0.0f;

    //float* f_slice = &f[uint64(i) * uint64(N.y * N.z)];
    for (int dj = -pixelRadius; dj <= pixelRadius; dj++)
    {
        const int j_shift = max(0, min(j + dj, N.y - 1));
        //const float j_dist_sq = float((j - j_shift) * (j - j_shift));
        for (int dk = -pixelRadius; dk <= pixelRadius; dk++)
        {
            const int k_shift = max(0, min(k + dk, N.z - 1));
            //const float k_dist_sq = float((k - k_shift) * (k - k_shift));

            //const float theWeight = exp(-denom * (j_dist_sq + k_dist_sq));

            const float theWeight = 0.5f +
                0.5f * cosf(3.141592653589793f * min(sqrtf(float(dj * dj + dk * dk)) * denom, 1.0f));

            if (theWeight > 0.0001f)
            {
                //val += theWeight * f_slice[uint64(j_shift * N.z + k_shift)];
                val += theWeight * tex3D<float>(f, k_shift, j_shift, i);
                sum += theWeight;
            }
        }
    }

    if (d_DO_HIGH_PASS)
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = tex3D<float>(f, k, j, i) - val / sum;
    else
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = val / sum;
}

__global__ void BlurFilter1DKernel_txt(hipTextureObject_t f, float* f_filtered, int3 N, float FWHM)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N.x || j >= N.y || k >= N.z) return;

    //const float sigma = FWHM / (2.0f * sqrt(2.0f * log(2.0f)));
    // FWHM = 2*sqrt(2*log(2))*sigma
    //const int pixelRadius = int(ceil(sqrt(2.0f * log(10.0f)) * sigma));
    //const float denom = 1.0f / (2.0f * sigma * sigma);
    const int pixelRadius = int(floor(FWHM));
    const float denom = 1.0f / FWHM;

    float val = 0.0;
    float sum = 0.0;
    for (int di = -pixelRadius; di <= pixelRadius; di++)
    {
        const int i_shift = max(0, min(i + di, N.x - 1));

        //const float theWeight = exp(-denom * float((i - i_shift) * (i - i_shift)));
        const float theWeight = 0.5f +
            0.5f * cosf(3.141592653589793f * min(fabsf(di) * denom, 1.0f));

        if (theWeight > 0.0001f)
        {
            //val += theWeight * f[uint64(i_shift) * uint64(N.y * N.z) + uint64(j * N.z + k)];
            val += theWeight * tex3D<float>(f, k, j, i_shift);
            sum += theWeight;
        }
    }

    if (d_DO_HIGH_PASS)
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = tex3D<float>(f, k, j, i) - val / sum;
    else
        f_filtered[uint64(i) * uint64(N.y * N.z) + uint64(j * N.z + k)] = val / sum;
}
//########################################################################################################################################################

void setConstantMemoryParameters(const bool doHighPass)
{
    hipMemcpyToSymbol(HIP_SYMBOL(d_DO_HIGH_PASS), &doHighPass, sizeof(bool));
}

bool blurFilter(float* f, int N_1, int N_2, int N_3, float FWHM, int numDims, bool data_on_cpu, int whichGPU, int sliceStart, int sliceEnd, float* f_out)
{
    hipSetDevice(whichGPU);
    setConstantMemoryParameters(false);
    return lowOrHighPassFilter(f, N_1, N_2, N_3, FWHM, numDims, data_on_cpu, whichGPU, sliceStart, sliceEnd, f_out);
}

bool blurFilter_txt(float* f, int N_1, int N_2, int N_3, float FWHM, int numDims, bool data_on_cpu, int whichGPU, int sliceStart, int sliceEnd, float* f_out)
{
    hipSetDevice(whichGPU);
    setConstantMemoryParameters(false);
    return lowOrHighPassFilter_txt(f, N_1, N_2, N_3, FWHM, numDims, data_on_cpu, whichGPU, sliceStart, sliceEnd, f_out);
}

bool highPassFilter(float* f, int N_1, int N_2, int N_3, float FWHM, int numDims, bool data_on_cpu, int whichGPU, int sliceStart, int sliceEnd, float* f_out)
{
    hipSetDevice(whichGPU);
    setConstantMemoryParameters(true);
    return lowOrHighPassFilter(f, N_1, N_2, N_3, FWHM, numDims, data_on_cpu, whichGPU, sliceStart, sliceEnd, f_out);
}

bool highPassFilter_txt(float* f, int N_1, int N_2, int N_3, float FWHM, int numDims, bool data_on_cpu, int whichGPU, int sliceStart, int sliceEnd, float* f_out)
{
    hipSetDevice(whichGPU);
    setConstantMemoryParameters(true);
    return lowOrHighPassFilter_txt(f, N_1, N_2, N_3, FWHM, numDims, data_on_cpu, whichGPU, sliceStart, sliceEnd, f_out);
}

bool lowOrHighPassFilter(float* f, int N_1, int N_2, int N_3, float FWHM, int numDims, bool data_on_cpu, int whichGPU, int sliceStart, int sliceEnd, float* f_out)
{
    return lowOrHighPassFilter_txt(f, N_1, N_2, N_3, FWHM, numDims, data_on_cpu, whichGPU, sliceStart, sliceEnd, f_out);
    if (f == NULL) return false;

    if (sliceStart < 0)
        sliceStart = 0;
    if (sliceEnd < 0)
        sliceEnd = N_1 - 1;
    sliceStart = max(0, min(N_1 - 1, sliceStart));
    sliceEnd = max(0, min(N_1 - 1, sliceEnd));
    if (sliceStart > sliceEnd)
        return false;

    hipSetDevice(whichGPU);
    //hipError_t cudaStatus;

    // Copy volume to GPU
    int3 N = make_int3(N_1, N_2, N_3);
    float* dev_f = 0;
    if (data_on_cpu)
        dev_f = copy3DdataToGPU(f, N, whichGPU);
    else
        dev_f = f;

    // Allocate space on GPU for the gradient
    float* dev_Df = 0;
    if (f_out != NULL && data_on_cpu == false)
    {
        dev_Df = f_out;
    }
    else
    {
        if (hipMalloc((void**)&dev_Df, uint64(N.x) * uint64(N.y) * uint64(N.z) * sizeof(float)) != hipSuccess)
        {
            fprintf(stderr, "hipMalloc(volume %d x %d x %d) failed!\n", N_1, N_2, N_3);
            return false;
        }
    }

    // Call kernel
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
                 int(ceil(double(N.z) / double(dimBlock.z))));
    if (numDims == 1)
        BlurFilter1DKernel<<<dimGrid, dimBlock>>>(dev_f, dev_Df, N, FWHM);
    else if (numDims == 2)
        BlurFilter2DKernel<<<dimGrid, dimBlock>>>(dev_f, dev_Df, N, FWHM);
    else
        BlurFilterKernel<<<dimGrid, dimBlock>>>(dev_f, dev_Df, N, FWHM, sliceStart, sliceEnd);

    // wait for GPU to finish
    hipDeviceSynchronize();

    // Clean up
    if (data_on_cpu)
    {
        // pull result off GPU
        if (f_out != NULL)
        {
            float* dev_Df_shift = &dev_Df[uint64(sliceStart) * uint64(N.y) * uint64(N.z)];
            int3 N_crop = make_int3(sliceEnd - sliceStart + 1, N_2, N_3);
            pull3DdataFromGPU(f_out, N_crop, dev_Df_shift, whichGPU);
        }
        else
            pull3DdataFromGPU(f, N, dev_Df, whichGPU);

        if (dev_f != 0)
            hipFree(dev_f);
    }
    else
    {
        if (f_out == NULL)
        {
            // copy dev_Df to dev_f
            hipMemcpy(dev_f, dev_Df, sizeof(float) * uint64(N.x) * uint64(N.y) * uint64(N.z), hipMemcpyDeviceToDevice);
            //hipDeviceSynchronize();
        }
    }
    if (dev_Df != 0 && (f_out == NULL || data_on_cpu == true))
    {
        hipFree(dev_Df);
    }

    return true;
}

bool lowOrHighPassFilter_txt(float* f, int N_1, int N_2, int N_3, float FWHM, int numDims, bool data_on_cpu, int whichGPU, int sliceStart, int sliceEnd, float* f_out)
{
    if (f == NULL) return false;

    if (sliceStart < 0)
        sliceStart = 0;
    if (sliceEnd < 0)
        sliceEnd = N_1 - 1;
    sliceStart = max(0, min(N_1 - 1, sliceStart));
    sliceEnd = max(0, min(N_1 - 1, sliceEnd));
    if (sliceStart > sliceEnd)
        return false;

    hipSetDevice(whichGPU);
    //hipError_t cudaStatus;

    // Copy volume to GPU
    int3 N = make_int3(N_1, N_2, N_3);
    float* dev_f = 0;
    if (data_on_cpu)
        dev_f = copy3DdataToGPU(f, N, whichGPU);
    else
        dev_f = f;

    hipTextureObject_t d_data_txt = NULL;
    hipArray* d_data_array = loadTexture(d_data_txt, dev_f, N, false, false);

    // Allocate space on GPU for the gradient
    float* dev_Df = 0;
    if (f_out != NULL && data_on_cpu == false)
    {
        dev_Df = f_out;
    }
    else
    {
        if (hipMalloc((void**)&dev_Df, uint64(N.x) * uint64(N.y) * uint64(N.z) * sizeof(float)) != hipSuccess)
        {
            fprintf(stderr, "hipMalloc(volume %d x %d x %d) failed!\n", N_1, N_2, N_3);
            return false;
        }
    }

    // Call kernel
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    if (numDims == 1)
        BlurFilter1DKernel_txt <<<dimGrid, dimBlock >>> (d_data_txt, dev_Df, N, FWHM);
    else if (numDims == 2)
        BlurFilter2DKernel_txt <<<dimGrid, dimBlock >>> (d_data_txt, dev_Df, N, FWHM);
    else
        BlurFilterKernel_txt <<<dimGrid, dimBlock >>> (d_data_txt, dev_Df, N, FWHM, sliceStart, sliceEnd);

    // wait for GPU to finish
    hipDeviceSynchronize();

    // Clean up
    hipFreeArray(d_data_array);
    hipDestroyTextureObject(d_data_txt);
    if (data_on_cpu)
    {
        // pull result off GPU
        if (f_out != NULL)
        {
            float* dev_Df_shift = &dev_Df[uint64(sliceStart) * uint64(N.y) * uint64(N.z)];
            int3 N_crop = make_int3(sliceEnd - sliceStart + 1, N_2, N_3);
            pull3DdataFromGPU(f_out, N_crop, dev_Df_shift, whichGPU);
        }
        else
            pull3DdataFromGPU(f, N, dev_Df, whichGPU);

        if (dev_f != 0)
            hipFree(dev_f);
    }
    else
    {
        if (f_out == NULL)
        {
            // copy dev_Df to dev_f
            hipMemcpy(dev_f, dev_Df, sizeof(float) * uint64(N.x) * uint64(N.y) * uint64(N.z), hipMemcpyDeviceToDevice);
            //hipDeviceSynchronize();
        }
    }
    if (dev_Df != 0 && (f_out == NULL || data_on_cpu == true))
    {
        hipFree(dev_Df);
    }

    return true;
}

bool momentFilter(float* f, int N_1, int N_2, int N_3, int r, int order, bool data_on_cpu, int whichGPU, int sliceStart, int sliceEnd, float* f_out)
{
    if (f == NULL) return false;

    if (sliceStart < 0)
        sliceStart = 0;
    if (sliceEnd < 0)
        sliceEnd = N_1 - 1;
    sliceStart = max(0, min(N_1 - 1, sliceStart));
    sliceEnd = max(0, min(N_1 - 1, sliceEnd));
    if (sliceStart > sliceEnd)
        return false;

    r = max(1, min(r, 100));

    hipSetDevice(whichGPU);
    //hipError_t cudaStatus;

    // Copy volume to GPU
    int3 N = make_int3(N_1, N_2, N_3);
    float* dev_f = 0;
    if (data_on_cpu)
        dev_f = copy3DdataToGPU(f, N, whichGPU);
    else
        dev_f = f;

    // Allocate space on GPU for the gradient
    float* dev_Df = 0;
    if (hipMalloc((void**)&dev_Df, uint64(N.x) * uint64(N.y) * uint64(N.z) * sizeof(float)) != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(volume %d x %d x %d) failed!\n", N_1, N_2, N_3);
        return false;
    }

    // Call kernel
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    if (order == 1)
    {
        meanFilterKernel <<< dimGrid, dimBlock >>> (dev_f, dev_Df, N, r, sliceStart, sliceEnd);
    }
    else
    {
        varianceFilterKernel <<< dimGrid, dimBlock >>> (dev_f, dev_Df, N, r, sliceStart, sliceEnd);
    }

    // wait for GPU to finish
    hipDeviceSynchronize();

    // Clean up
    if (data_on_cpu)
    {
        // pull result off GPU
        if (f_out != NULL)
        {
            float* dev_Df_shift = &dev_Df[uint64(sliceStart) * uint64(N.y) * uint64(N.z)];
            int3 N_crop = make_int3(sliceEnd - sliceStart + 1, N_2, N_3);
            pull3DdataFromGPU(f_out, N_crop, dev_Df_shift, whichGPU);
        }
        else
            pull3DdataFromGPU(f, N, dev_Df, whichGPU);

        if (dev_f != 0)
            hipFree(dev_f);
    }
    else
    {
        // copy dev_Df to dev_f
        hipMemcpy(dev_f, dev_Df, sizeof(float) * uint64(N.x) * uint64(N.y) * uint64(N.z), hipMemcpyDeviceToDevice);
    }
    if (dev_Df != 0)
    {
        hipFree(dev_Df);
    }

    return true;
}

bool medianFilter(float* f, int N_1, int N_2, int N_3, float threshold, int w, float signalThreshold, bool data_on_cpu, int whichGPU, int sliceStart, int sliceEnd, float* f_out)
{
    if (f == NULL) return false;

    if (sliceStart < 0)
        sliceStart = 0;
    if (sliceEnd < 0)
        sliceEnd = N_1 - 1;
    sliceStart = max(0, min(N_1 - 1, sliceStart));
    sliceEnd = max(0, min(N_1 - 1, sliceEnd));
    if (sliceStart > sliceEnd)
        return false;

    int windowRadius = max(1, min(2, (w - 1) / 2));

    hipSetDevice(whichGPU);
    //hipError_t cudaStatus;

    // Copy volume to GPU
    int3 N = make_int3(N_1, N_2, N_3);
    float* dev_f = 0;
    if (data_on_cpu)
        dev_f = copy3DdataToGPU(f, N, whichGPU);
    else
        dev_f = f;

    // Allocate space on GPU for the gradient
    float* dev_Df = 0;
    if (hipMalloc((void**)&dev_Df, uint64(N.x) * uint64(N.y) * uint64(N.z) * sizeof(float)) != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(volume %d x %d x %d) failed!\n", N_1, N_2, N_3);
        return false;
    }

    // Call kernel
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
                 int(ceil(double(N.z) / double(dimBlock.z))));
    if (windowRadius == 2)
    {
        medianFilterKernel_5x5 <<<dimGrid, dimBlock >>> (dev_f, dev_Df, N, threshold, signalThreshold, sliceStart, sliceEnd);
    }
    else
    {
        medianFilterKernel <<<dimGrid, dimBlock >>> (dev_f, dev_Df, N, threshold, signalThreshold, sliceStart, sliceEnd);
    }

    // wait for GPU to finish
    hipDeviceSynchronize();

    // Clean up
    if (data_on_cpu)
    {
        // pull result off GPU
        if (f_out != NULL)
        {
            float* dev_Df_shift = &dev_Df[uint64(sliceStart) * uint64(N.y) * uint64(N.z)];
            int3 N_crop = make_int3(sliceEnd - sliceStart + 1, N_2, N_3);
            pull3DdataFromGPU(f_out, N_crop, dev_Df_shift, whichGPU);
        }
        else
            pull3DdataFromGPU(f, N, dev_Df, whichGPU);

        if (dev_f != 0)
            hipFree(dev_f);
    }
    else
    {
        // copy dev_Df to dev_f
        hipMemcpy(dev_f, dev_Df, sizeof(float) * uint64(N.x) * uint64(N.y) * uint64(N.z), hipMemcpyDeviceToDevice);
    }
    if (dev_Df != 0)
    {
        hipFree(dev_Df);
    }

    return true;
}

bool medianFilter2D(float* f, int N_1, int N_2, int N_3, float threshold, int w, float signalThreshold, bool data_on_cpu, int whichGPU)
{
    if (f == NULL) return false;

    hipSetDevice(whichGPU);
    //hipError_t cudaStatus;

    // Copy volume to GPU
    int3 N = make_int3(N_1, N_2, N_3);
    float* dev_f = 0;
    if (data_on_cpu)
        dev_f = copy3DdataToGPU(f, N, whichGPU);
    else
        dev_f = f;

    // Allocate space on GPU for the gradient
    float* dev_Df = 0;
    if (hipMalloc((void**)&dev_Df, uint64(N.x) * uint64(N.y) * uint64(N.z) * sizeof(float)) != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(volume %d x %d x %d) failed!\n", N_1, N_2, N_3);
        return false;
    }

    int windowRadius = max(1, min(3, (w-1)/2));

    // Call kernel
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    medianFilter2DKernel <<< dimGrid, dimBlock >>> (dev_f, dev_Df, N, threshold, windowRadius, signalThreshold);

    // wait for GPU to finish
    hipDeviceSynchronize();

    // Clean up
    if (data_on_cpu)
    {
        // pull result off GPU
        pull3DdataFromGPU(f, N, dev_Df, whichGPU);

        if (dev_f != 0)
            hipFree(dev_f);
    }
    else
    {
        // copy dev_Df to dev_f
        hipMemcpy(dev_f, dev_Df, sizeof(float) * uint64(N.x) * uint64(N.y) * uint64(N.z), hipMemcpyDeviceToDevice);
    }
    if (dev_Df != 0)
    {
        hipFree(dev_Df);
    }

    return true;
}

bool badPixelCorrection_gpu(float* g, parameters* params, float* badPixelMap, int w, bool data_on_cpu)
{
    if (g  == NULL || params == NULL || badPixelMap == NULL) return false;

    hipSetDevice(params->whichGPU);
    //hipError_t cudaStatus;

    // Copy volume to GPU
    int3 N = make_int3(params->numAngles, params->numRows, params->numCols);
    float* dev_g = 0;
    float* dev_badPixelMap = 0;
    if (data_on_cpu)
    {
        dev_g = copy3DdataToGPU(g, N, params->whichGPU);
        dev_badPixelMap = copy3DdataToGPU(badPixelMap, make_int3(1, params->numRows, params->numCols), params->whichGPU);
    }
    else
    {
        dev_g = g;
        dev_badPixelMap = badPixelMap;
    }

    int windowRadius = max(1, min(3, (w - 1) / 2));

    // Call kernel
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    badPixelCorrectionKernel <<< dimGrid, dimBlock >>> (dev_g, dev_badPixelMap, N, windowRadius);

    // wait for GPU to finish
    hipDeviceSynchronize();

    // Clean up
    if (data_on_cpu)
    {
        // pull result off GPU
        pull3DdataFromGPU(g, N, dev_g, params->whichGPU);

        if (dev_g != 0)
            hipFree(dev_g);
        if (dev_badPixelMap != 0)
            hipFree(dev_badPixelMap);
    }

    return true;
}

bool azimuthalBlur(float* f, parameters* params, float filterWidth, bool data_on_cpu, float* f_out)
{
    if (f == NULL) return false;

    hipSetDevice(params->whichGPU);
    //hipError_t cudaStatus;

    // Copy volume to GPU
    int3 N = make_int3(params->numX, params->numY, params->numZ);
    float3 T = make_float3(params->voxelWidth, params->voxelWidth, params->voxelHeight);
    float3 startVal = make_float3(params->x_0(), params->y_0(), params->z_0());
    float* dev_f = 0;
    if (data_on_cpu)
    {
        //dev_f = copy3DdataToGPU(f, N, params->whichGPU);
        dev_f = copyVolumeDataToGPU(f, params, params->whichGPU);
    }
    else
        dev_f = f;

    int N_phi_max = max(4, 2 * int(double(max(N.x, N.y))));

    // Allocate space on GPU for the gradient
    float* dev_Df = 0;
    if (hipMalloc((void**)&dev_Df, uint64(N.x) * uint64(N.y) * uint64(N.z) * sizeof(float)) != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(volume %d x %d x %d) failed!\n", N.x, N.y, N.z);
        return false;
    }

    float rFOVsq = params->rFOV() * params->rFOV();

    // Call kernel
    dim3 dimBlock = setBlockSize(N);
    dim3 dimGrid(int(ceil(double(N.x) / double(dimBlock.x))), int(ceil(double(N.y) / double(dimBlock.y))),
        int(ceil(double(N.z) / double(dimBlock.z))));
    azimuthalBlurKernel <<< dimGrid, dimBlock >>> (dev_f, dev_Df, N, T, startVal, N_phi_max, filterWidth, rFOVsq);

    // wait for GPU to finish
    hipDeviceSynchronize();

    // Clean up
    if (data_on_cpu)
    {
        // pull result off GPU
        if (f_out != NULL)
            pull3DdataFromGPU(f_out, N, dev_Df, params->whichGPU);
        else
            pullVolumeDataFromGPU(f, params, dev_Df, params->whichGPU);

        if (dev_f != 0)
            hipFree(dev_f);
    }
    else
    {
        // copy dev_Df to dev_f
        hipMemcpy(dev_f, dev_Df, sizeof(float) * uint64(N.x) * uint64(N.y) * uint64(N.z), hipMemcpyDeviceToDevice);
    }
    if (dev_Df != 0)
    {
        hipFree(dev_Df);
    }

    return true;
}
