#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright 2022-2023 Lawrence Livermore National Security, LLC and other 
// LEAP project developers. See the LICENSE file for details.
// SPDX-License-Identifier: MIT
//
// LivermorE AI Projector for Computed Tomography (LEAP)
// cuda module for the primary projectors models in LEAP
////////////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""
#include "projectors.h"
#include "projectors_SF.cuh"
#include "cuda_utils.h"

#include "projectors_extendedSF.cuh"

//#define NUM_SLICES_PER_THREAD 1
#define NUM_SLICES_PER_THREAD 8

__constant__ float d_q_helical;
__constant__ float d_v_min_inv;
__constant__ float d_v_max_inv;
__constant__ float d_weightFcnTransition;
__constant__ float d_weightFcnParameter;
__constant__ float d_phi_start;
__constant__ float d_phi_end;

__device__ float helicalConeWeight(float v)
{
    const float abs_v_hat = (v >= 0.0f) ? v * d_v_max_inv : v * d_v_min_inv;

    if (abs_v_hat <= d_q_helical)
        return 1.0f;
    else if (abs_v_hat > 1.0f)
        return 0.0f;
    else if (abs_v_hat <= d_weightFcnTransition)
        return d_weightFcnParameter * (abs_v_hat - d_q_helical) * (abs_v_hat - d_q_helical) + 1.0f;
    else
        return -1.0f * d_weightFcnParameter * (abs_v_hat - 1.0f) * (abs_v_hat - 1.0f);
}

__global__ void parallelBeamBackprojectorKernel_SF(hipTextureObject_t g, int4 N_g, float4 T_g, float4 startVals_g, float* f, int4 N_f, float4 T_f, float4 startVals_f, float rFOVsq, float* phis, int volumeDimensionOrder)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = (threadIdx.z + blockIdx.z * blockDim.z) * NUM_SLICES_PER_THREAD;
    if (i >= N_f.x || j >= N_f.y || k >= N_f.z)
        return;
        
    const float x = i * T_f.x + startVals_f.x;
    const float y = j * T_f.y + startVals_f.y;
    const float z = k * T_f.z + startVals_f.z;
    
    uint64 ind;
    if (volumeDimensionOrder == 0)
        ind = uint64(i) * uint64(N_f.y * N_f.z) + uint64(j * N_f.z + k);
    else
        ind = uint64(k) * uint64(N_f.y * N_f.x) + uint64(j * N_f.x + i);

    if (x*x + y*y > rFOVsq)
    {
        f[ind] = 0.0;
        return;
    }
    
    const float maxWeight = T_f.x * T_f.y / T_g.z;
    const float maxWeight_inv = 1.0f / maxWeight;

    const float T_u_inv = 1.0f / T_g.z;
    const float C_num = 0.5f * T_u_inv * T_f.x;
    const float C_num_T_x = T_f.x * C_num;
    const float x_mult = x * T_u_inv;
    const float y_mult = y * T_u_inv;
    const float s_shift = -startVals_g.z * T_u_inv;

    //*
    float vals[NUM_SLICES_PER_THREAD];
    int numZ = min(NUM_SLICES_PER_THREAD, N_f.z-k);
    for (int k_offset = 0; k_offset < numZ; k_offset++)
        vals[k_offset] = 0.0f;
    for (int l = 0; l < N_g.x; l++)
    {
        //float sin_phi, cos_phi;
        //sincosf(phis[l], &sin_phi, &cos_phi);
        const float sin_phi = sinf(phis[l]);
        const float cos_phi = cosf(phis[l]);
        const float C = C_num * max(fabs(cos_phi), fabs(sin_phi));
        float s_arg = s_shift - sin_phi * x_mult + cos_phi * y_mult;
        const float ds = modf(s_arg, &s_arg);
        const float s_ind_A = s_arg - (C_num_T_x / C * max(0.0f, (min(0.5f, C + ds) + min(0.5f, C - ds)))) * maxWeight_inv + 1.5f;

        for (int k_offset = 0; k_offset < numZ; k_offset++)
            vals[k_offset] += tex3D<float>(g, s_ind_A, float(k+k_offset) + 0.5f, float(l) + 0.5f);
    }

    if (volumeDimensionOrder == 0)
    {
        for (int k_offset = 0; k_offset < numZ; k_offset++)
            f[ind + uint64(k_offset)] = vals[k_offset] * maxWeight;
    }
    else
    {
        for (int k_offset = 0; k_offset < numZ; k_offset++)
            f[ind + uint64(k_offset) * uint64(N_f.y * N_f.x)] = vals[k_offset] * maxWeight;
    }
    //*/

    /*
    float cos_phi, sin_phi, C, s_arg, ds;
    
    float val = 0.0;
    int l = 0;
    while (l < N_g.x)
    {
        if (l+3 < N_g.x)
        {
            const int l1 = l + 1;
            const int l2 = l + 2;
            const int l3 = l + 3;

			sin_phi = sin(phis[l]);
			cos_phi = cos(phis[l]);
            C = C_num * max(fabs(cos_phi), fabs(sin_phi));
            s_arg = s_shift - sin_phi * x_mult + cos_phi * y_mult;
            ds = modf(s_arg,&s_arg);
            const float s_ind_A = s_arg-(C_num_T_x / C * max(0.0f, (min(0.5f, C + ds) + min(0.5f, C - ds) )))*maxWeight_inv+1.5f;

			sin_phi = sin(phis[l1]);
			cos_phi = cos(phis[l1]);
            C = C_num * max(fabs(cos_phi), fabs(sin_phi));
            s_arg = s_shift - sin_phi * x_mult + cos_phi * y_mult;
            ds = modf(s_arg,&s_arg);
            const float s_ind_B = s_arg-(C_num_T_x / C * max(0.0f, (min(0.5f, C + ds) + min(0.5f, C - ds) )))*maxWeight_inv+1.5f;

			sin_phi = sin(phis[l2]);
			cos_phi = cos(phis[l2]);
            C = C_num * max(fabs(cos_phi), fabs(sin_phi));
            s_arg = s_shift - sin_phi * x_mult + cos_phi * y_mult;
            ds = modf(s_arg,&s_arg);
            const float s_ind_C = s_arg-(C_num_T_x / C * max(0.0f, (min(0.5f, C + ds) + min(0.5f, C - ds) )))*maxWeight_inv+1.5f;

			sin_phi = sin(phis[l3]);
			cos_phi = cos(phis[l3]);
            C = C_num * max(fabs(cos_phi), fabs(sin_phi));
            s_arg = s_shift - sin_phi * x_mult + cos_phi * y_mult;
            ds = modf(s_arg,&s_arg);
            const float s_ind_D = s_arg-(C_num_T_x / C * max(0.0f, (min(0.5f, C + ds) + min(0.5f, C - ds) )))*maxWeight_inv+1.5f;

            // Do texture mapping
            val += tex3D<float>(g,s_ind_A, float(k)+0.5f, float(l)+0.5f)
                +  tex3D<float>(g,s_ind_B, float(k)+0.5f, float(l1)+0.5f)
                +  tex3D<float>(g,s_ind_C, float(k)+0.5f, float(l2)+0.5f)
                +  tex3D<float>(g,s_ind_D, float(k)+0.5f, float(l3)+0.5f);
            l += 4;
        }
        else if (l+1 < N_g.x)
        {
            int l1 = l + 1;

			sin_phi = sin(phis[l]);
			cos_phi = cos(phis[l]);
            C = C_num * max(fabs(cos_phi), fabs(sin_phi));
            s_arg = s_shift - sin_phi * x_mult + cos_phi * y_mult;
            ds = modf(s_arg,&s_arg);
            const float s_ind_A = s_arg-(C_num_T_x / C * max(0.0f, (min(0.5f, C + ds) + min(0.5f, C - ds) )))*maxWeight_inv+1.5f;

			sin_phi = sin(phis[l1]);
			cos_phi = cos(phis[l1]);
            C = C_num * max(fabs(cos_phi), fabs(sin_phi));
            s_arg = s_shift - sin_phi * x_mult + cos_phi * y_mult;
            ds = modf(s_arg,&s_arg);
            const float s_ind_B = s_arg-(C_num_T_x / C * max(0.0f, (min(0.5f, C + ds) + min(0.5f, C - ds) )))*maxWeight_inv+1.5f;

            val += tex3D<float>(g,s_ind_A, float(k)+0.5f, float(l)+0.5f)
                +  tex3D<float>(g,s_ind_B, float(k)+0.5f, float(l1)+0.5f);
            l += 2;
        }
        else //if (l+1 < N_g.x)
        {
			sin_phi = sin(phis[l]);
			cos_phi = cos(phis[l]);
            C = C_num * max(fabs(cos_phi), fabs(sin_phi));
            s_arg = s_shift - sin_phi * x_mult + cos_phi * y_mult;
            ds = modf(s_arg,&s_arg);
            const float s_ind_A = s_arg-(C_num_T_x / C * max(0.0f, (min(0.5f, C + ds) + min(0.5f, C - ds) )))*maxWeight_inv+1.5f;

            val += tex3D<float>(g,s_ind_A, float(k)+0.5f, float(l)+0.5f);
            l += 1;
        }
    }

    f[ind] = val * maxWeight;
    //*/
}

__global__ void parallelBeamProjectorKernel_SF(float* g, int4 N_g, float4 T_g, float4 startVals_g, hipTextureObject_t f, int4 N_f, float4 T_f, float4 startVals_f, float rFOVsq, float* phis, int volumeDimensionOrder)
{
    const int l = threadIdx.x + blockIdx.x * blockDim.x;
    const int m = threadIdx.y + blockIdx.y * blockDim.y;
    const int n = threadIdx.z + blockIdx.z * blockDim.z;
    if (l >= N_g.x || m >= N_g.y || n >= N_g.z)
        return;

    const float v = m * T_g.y + startVals_g.y;
    const float u = n * T_g.z + startVals_g.z;
    
    const float sin_phi = sin(phis[l]);
    const float cos_phi = cos(phis[l]);
    
    const float n_minus_half = (float)n - 0.5f;
    const float n_plus_half = (float)n + 0.5f;
    const float l_phi = T_f.x / max(fabs(cos_phi), fabs(sin_phi));
    const float C = T_f.x * T_f.x / (2.0f * T_g.z * l_phi);

    const float ds_ind_di = -T_f.x*sin_phi / T_g.z;
    const float ds_ind_dj = T_f.y*cos_phi / T_g.z;
    const float s_ind_offset = (startVals_f.y*cos_phi - startVals_f.x*sin_phi - startVals_g.z) / T_g.z;
    // s_ind(i,j) = (float)i * ds_ind_di + (float)j * ds_ind_dj + s_ind_offset

    float g_output = 0.0f;
    if (fabs(cos_phi) > fabs(sin_phi))
    {
        const float ds_ind_dj_inv = 1.0f / ds_ind_dj;
        float shiftConstant;
        if (ds_ind_dj > 0.0f)
            shiftConstant = (n_minus_half-C) * ds_ind_dj_inv;
        else
            shiftConstant = (n_plus_half+C) * ds_ind_dj_inv;
        for (int i = 0; i < N_f.x; i++)
        {
            const float s_ind_base = (float)i * ds_ind_di +  s_ind_offset;
            const int j_min_A = (int)ceil(shiftConstant - s_ind_base * ds_ind_dj_inv);
            const float s_ind_A = s_ind_base + (float)j_min_A * ds_ind_dj;

            if (((float)i*T_f.x+startVals_f.x )*((float)i*T_f.x+startVals_f.x) + ((float)j_min_A*T_f.y+startVals_f.y )*((float)j_min_A*T_f.y+startVals_f.y) > rFOVsq)
                continue;

            const float weight_0 = max(0.0f, min(n_plus_half, s_ind_A + C) - max(n_minus_half, s_ind_A - C));
            const float weight_1 = max(0.0f, min(n_plus_half, s_ind_A + ds_ind_dj + C) - max(n_minus_half, s_ind_A + ds_ind_dj - C));
            if (volumeDimensionOrder == 0)
            {
                g_output += (weight_0 + weight_1) * tex3D<float>(f, float(m)+0.5f, float(j_min_A)+0.5f+ weight_1/(weight_0+ weight_1), float(i)+0.5f)
                    + max(0.0f, min(n_plus_half, s_ind_A + 2.0f * ds_ind_dj + C) - max(n_minus_half, s_ind_A + 2.0f * ds_ind_dj - C)) * tex3D<float>(f, float(m)+0.5f, float(j_min_A + 2) + 0.5f, float(i) + 0.5f);
            }
            else
            {
                g_output += (weight_0 + weight_1) * tex3D<float>(f, float(i) + 0.5f, float(j_min_A) + 0.5f + weight_1 / (weight_0 + weight_1), float(m) + 0.5f)
                    + max(0.0f, min(n_plus_half, s_ind_A + 2.0f * ds_ind_dj + C) - max(n_minus_half, s_ind_A + 2.0f * ds_ind_dj - C)) * tex3D<float>(f, float(i) + 0.5f, float(j_min_A + 2) + 0.5f, float(m)+0.5f);
            }
        }
    }
    else
    {
        const float ds_ind_di_inv = 1.0f / ds_ind_di;
        float shiftConstant;
        if (ds_ind_di > 0.0f)
            shiftConstant = (n_minus_half-C) * ds_ind_di_inv;
        else
            shiftConstant = (n_plus_half+C) * ds_ind_di_inv;
        for (int j = 0; j < N_f.y; j++)
        {
            const float s_ind_base = (float)j * ds_ind_dj + s_ind_offset;
            const int i_min_A = (int)ceil(shiftConstant - s_ind_base * ds_ind_di_inv);
            const float s_ind_A = s_ind_base + (float)i_min_A * ds_ind_di;

            if (((float)i_min_A*T_f.x+startVals_f.x )*((float)i_min_A*T_f.x+startVals_f.x) + ((float)j*T_f.y+startVals_f.y )*((float)j*T_f.y+startVals_f.y) > rFOVsq)
                continue;

            const float weight_0 = max(0.0f, min(n_plus_half, s_ind_A + C) - max(n_minus_half, s_ind_A - C));
            const float weight_1 = max(0.0f, min(n_plus_half, s_ind_A + ds_ind_di + C) - max(n_minus_half, s_ind_A + ds_ind_di - C));
            if (volumeDimensionOrder == 0)
            {
                g_output += (weight_0 + weight_1) * tex3D<float>(f, float(m)+0.5f, float(j)+0.5f, float(i_min_A)+0.5f + weight_1/(weight_0 + weight_1))
                    + max(0.0f, min(n_plus_half, s_ind_A + 2.0f * ds_ind_di + C) - max(n_minus_half, s_ind_A + 2.0f * ds_ind_di - C)) * tex3D<float>(f, float(m) + 0.5f, float(j) + 0.5f, float(i_min_A + 2)+0.5f);
            }
            else
            {
                g_output += (weight_0 + weight_1) * tex3D<float>(f, float(i_min_A) + 0.5f + weight_1 / (weight_0 + weight_1), float(j) + 0.5f, float(m) + 0.5f)
                    + max(0.0f, min(n_plus_half, s_ind_A + 2.0f * ds_ind_di + C) - max(n_minus_half, s_ind_A + 2.0f * ds_ind_di - C)) * tex3D<float>(f, float(i_min_A + 2) + 0.5f, float(j) + 0.5f, float(m) + 0.5f);
            }
        }
    }
    g[uint64(l) * uint64(N_g.z * N_g.y) + uint64(m * N_g.z + n)] = l_phi * g_output;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void fanBeamBackprojectorKernel_SF(hipTextureObject_t g, int4 N_g, float4 T_g, float4 startVals_g, float* f, int4 N_f, float4 T_f, float4 startVals_f, float R, float D, float tau, float rFOVsq, float* phis, int volumeDimensionOrder, bool doWeight)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = (threadIdx.z + blockIdx.z * blockDim.z)*NUM_SLICES_PER_THREAD;
    if (i >= N_f.x || j >= N_f.y || k >= N_f.z)
        return;

    uint64 ind;
    if (volumeDimensionOrder == 0)
        ind = uint64(i) * uint64(N_f.y * N_f.z) + uint64(j * N_f.z + k);
    else
        ind = uint64(k) * uint64(N_f.y * N_f.x) + uint64(j * N_f.x + i);

    const float x = i * T_f.x + startVals_f.x;
    const float y = j * T_f.y + startVals_f.y;
    const float z = k * T_f.z + startVals_f.z;
    if (x * x + y * y > rFOVsq)
    {
        f[ind] = 0.0f;
        return;
    }

    //*
    float vals[NUM_SLICES_PER_THREAD];
    int numZ = min(NUM_SLICES_PER_THREAD, N_f.z - k);
    for (int k_offset = 0; k_offset < numZ; k_offset++)
        vals[k_offset] = 0.0f;

    const float iv = (z - startVals_g.y) / T_g.y + 0.5f;
    const float T_x_over_2 = 0.5f * T_f.x;
    const float Tu_inv = 1.0f / T_g.z;

    for (int l = 0; l < N_g.x; l++)
    {
        const float L = (float)l + 0.5f;
        const float sin_phi = sin(phis[l]);
        const float cos_phi = cos(phis[l]);

        float B_x = (sin_phi < 0.0f) ? -cos_phi * T_x_over_2 : cos_phi * T_x_over_2;
        const float B_y = (cos_phi < 0.0f) ? sin_phi * T_x_over_2 : -sin_phi * T_x_over_2;

        const float x_dot_theta_perp = cos_phi * y - sin_phi * x + tau;
        const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
        const float R_minus_x_dot_theta_inv = 1.0f / R_minus_x_dot_theta;

        const float u_arg = x_dot_theta_perp * R_minus_x_dot_theta_inv;
        const float x_denom = fabs(u_arg * cos_phi - sin_phi);
        const float y_denom = fabs(u_arg * sin_phi + cos_phi);
        const float l_phi = T_f.x * sqrt(1.0f + u_arg * u_arg) / max(x_denom, y_denom);
        float A_x;
        if (x_denom > y_denom)
            A_x = fabs(sin_phi) * T_x_over_2;
        else
        {
            A_x = fabs(cos_phi) * T_x_over_2;
            B_x = B_y;
        }
        const float tau_low = ((x_dot_theta_perp - A_x) / (R_minus_x_dot_theta - B_x) - startVals_g.z) * Tu_inv;
        const float tau_high = ((x_dot_theta_perp + A_x) / (R_minus_x_dot_theta + B_x) - startVals_g.z) * Tu_inv;

        float ind_first = floor(tau_low + 0.5f); // first detector index

        const float horizontalWeights_0_A = (min(tau_high, ind_first + 1.5f) - tau_low) * l_phi;
        const float horizontalWeights_1_A = l_phi * (tau_high - tau_low) - horizontalWeights_0_A;

        const float ind_last = ind_first + 2.5f;
        ind_first = ind_first + 0.5f + max(0.0f, min(tau_high - ind_first - 0.5f, 1.0f)) * l_phi / horizontalWeights_0_A;

        const float bpWeight = doWeight ? R * R_minus_x_dot_theta_inv : 1.0f;

        for (int k_offset = 0; k_offset < numZ; k_offset++)
        {
            vals[k_offset] += (tex3D<float>(g, ind_first, iv + float(k_offset), L) * horizontalWeights_0_A
                + tex3D<float>(g, ind_last, iv + float(k_offset), L) * horizontalWeights_1_A) * bpWeight;
        }
    }

    if (volumeDimensionOrder == 0)
    {
        for (int k_offset = 0; k_offset < numZ; k_offset++)
            f[ind + uint64(k_offset)] = vals[k_offset];
    }
    else
    {
        for (int k_offset = 0; k_offset < numZ; k_offset++)
            f[ind + uint64(k_offset) * uint64(N_f.y * N_f.x)] = vals[k_offset];
    }
    //*/

    /*
    const float iv = (z - startVals_g.y) / T_g.y + 0.5f;
    const float T_x_over_2 = 0.5f * T_f.x;
    const float Tu_inv = 1.0f / T_g.z;

    float val = 0.0;
    for (int l = 0; l < N_g.x; l++)
    {
        const float L = (float)l + 0.5f;
        const float sin_phi = sin(phis[l]);
        const float cos_phi = cos(phis[l]);

        float B_x = (sin_phi < 0.0f) ? -cos_phi * T_x_over_2 : cos_phi * T_x_over_2;
        const float B_y = (cos_phi < 0.0f) ? sin_phi * T_x_over_2 : -sin_phi * T_x_over_2;

        const float x_dot_theta_perp = cos_phi * y - sin_phi * x + tau;
        const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
        const float R_minus_x_dot_theta_inv = 1.0f / R_minus_x_dot_theta;

        const float u_arg = x_dot_theta_perp * R_minus_x_dot_theta_inv;
        const float x_denom = fabs(u_arg * cos_phi - sin_phi);
        const float y_denom = fabs(u_arg * sin_phi + cos_phi);
        const float l_phi = T_f.x * sqrt(1.0f + u_arg * u_arg) / max(x_denom, y_denom);
        float A_x;
        if (x_denom > y_denom)
            A_x = fabs(sin_phi) * T_x_over_2;
        else
        {
            A_x = fabs(cos_phi) * T_x_over_2;
            B_x = B_y;
        }
        const float tau_low = ((x_dot_theta_perp - A_x) / (R_minus_x_dot_theta - B_x) - startVals_g.z) * Tu_inv;
        const float tau_high = ((x_dot_theta_perp + A_x) / (R_minus_x_dot_theta + B_x) - startVals_g.z) * Tu_inv;

        float ind_first = floor(tau_low + 0.5f); // first detector index

        const float horizontalWeights_0_A = (min(tau_high, ind_first + 1.5f) - tau_low) * l_phi;
        const float horizontalWeights_1_A = l_phi * (tau_high - tau_low) - horizontalWeights_0_A;

        const float ind_last = ind_first + 2.5f;
        ind_first = ind_first + 0.5f + max(0.0f, min(tau_high - ind_first - 0.5f, 1.0f)) * l_phi / horizontalWeights_0_A;

        const float bpWeight = doWeight ? R * R_minus_x_dot_theta_inv : 1.0f;

        val += (tex3D<float>(g, ind_first, iv, L) * horizontalWeights_0_A
            + tex3D<float>(g, ind_last, iv, L) * horizontalWeights_1_A) * bpWeight;
    }

    f[ind] = val;
    //*/
}

__global__ void fanBeamProjectorKernel_SF(float* g, int4 N_g, float4 T_g, float4 startVals_g, hipTextureObject_t f, int4 N_f, float4 T_f, float4 startVals_f, float R, float D, float tau, float rFOVsq, float* phis, int volumeDimensionOrder)
{
    const int l = threadIdx.x + blockIdx.x * blockDim.x;
    const int m = threadIdx.y + blockIdx.y * blockDim.y;
    const int n = threadIdx.z + blockIdx.z * blockDim.z;
    if (l >= N_g.x || m >= N_g.y || n >= N_g.z)
        return;

    const float v = m * T_g.y + startVals_g.y;
    const float u = n * T_g.z + startVals_g.z;

    const float sin_phi = sin(phis[l]);
    const float cos_phi = cos(phis[l]);

    const float n_minus_half = (float)n - 0.5f + startVals_g.z / T_g.z;
    const float n_plus_half = (float)n + 0.5f + startVals_g.z / T_g.z;
    const float m_minus_half = (float)m - 0.5f;
    const float m_plus_half = (float)m + 0.5f;

    const int iz = int(floor(0.5 + (v - startVals_f.z) / T_f.z));

    float g_output = 0.0f;

    if (fabs(u * cos_phi - sin_phi) > fabs(u * sin_phi + cos_phi))
    {
        const float A_x = fabs(sin_phi) * 0.5f * T_f.x;
        const float B_x = cos_phi * 0.5f * T_f.x * ((sin_phi < 0.0f) ? -1.0f : 1.0f);
        const float Tx_sin = T_f.x * sin_phi;
        const float Tx_cos = T_g.z * T_f.x * cos_phi;

        float shiftConstant, slopeConstant;
        if (u * cos_phi - sin_phi > 0.0f)
        {
            shiftConstant = (((R + B_x) * (u - 0.5f * T_g.z) - A_x - tau) / (cos_phi * (u - 0.5f * T_g.z) - sin_phi) - startVals_f.x) / T_f.x;
            slopeConstant = (-sin_phi * (u - 0.5f * T_g.z) - cos_phi) / (T_f.x * (cos_phi * (u - 0.5f * T_g.z) - sin_phi));
        }
        else
        {
            shiftConstant = (((R - B_x) * (u + 0.5f * T_g.z) + A_x - tau) / (cos_phi * (u + 0.5f * T_g.z) - sin_phi) - startVals_f.x) / T_f.x;
            slopeConstant = (sin_phi * (u + 0.5f * T_g.z) + cos_phi) / (T_f.x * (-cos_phi * (u + 0.5f * T_g.z) + sin_phi));
        }

        for (int j = 0; j < N_f.y; j++)
        {
            const float y = (float)j * T_f.y + startVals_f.y;
            const int i = (int)ceil(y * slopeConstant + shiftConstant);
            const float x = (float)i * T_f.x + startVals_f.x;

            if (x * x + y * y > rFOVsq)
                continue;

            const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
            const float num_low = tau - x * sin_phi + y * cos_phi - A_x;
            const float num_high = num_low + 2.0f * A_x;

            const float denom_low = (R_minus_x_dot_theta - B_x) * T_g.z;
            const float denom_high = (R_minus_x_dot_theta + B_x) * T_g.z;

            const float hWeight_0 = max(0.0f, min(num_high / denom_high, n_plus_half) - max(num_low / denom_low, n_minus_half));
            const float hWeight_1 = max(0.0f, min((num_high - Tx_sin) / (denom_high - Tx_cos), n_plus_half) - max((num_low - Tx_sin) / (denom_low - Tx_cos), n_minus_half));
            const float hWeight_2 = max(0.0f, 1.0f - hWeight_1 - hWeight_0);

            if (volumeDimensionOrder == 0)
            {
                g_output += tex3D<float>(f, float(iz) + 0.5f, float(j) + 0.5f, float(i) + 0.5f + hWeight_1/(hWeight_0 + hWeight_1)) * (hWeight_0 + hWeight_1)
                    + tex3D<float>(f, float(iz) + 0.5f, float(j) + 0.5f, float(i + 2)+0.5f) * hWeight_2;
            }
            else
            {
                g_output += tex3D<float>(f, float(i) + 0.5f + hWeight_1 / (hWeight_0 + hWeight_1), float(j) + 0.5f, float(iz) + 0.5f) * (hWeight_0 + hWeight_1)
                    + tex3D<float>(f, float(i + 2) + 0.5f, float(j) + 0.5f, float(iz) + 0.5f) * hWeight_2;
            }
        }
        g[uint64(l) * uint64(N_g.z * N_g.y) + uint64(m * N_g.z + n)] = T_f.x * sqrt(1.0f + u * u) / fabs(u * cos_phi - sin_phi) * g_output;
    }
    else
    {
        const float A_y = fabs(cos_phi) * 0.5f * T_f.x;
        const float B_y = sin_phi * 0.5f * T_f.x * ((cos_phi < 0.0f) ? 1.0f : -1.0f);
        const float Ty_cos = T_f.y * cos_phi;
        const float Ty_sin = T_g.z * T_f.y * sin_phi;

        float shiftConstant, slopeConstant;
        if (u * sin_phi + cos_phi >= 0.0f)
        {
            shiftConstant = (((R + B_y) * (u - 0.5f * T_g.z) - A_y - tau) / (sin_phi * (u - 0.5f * T_g.z) + cos_phi) - startVals_f.y) / T_f.y;
            slopeConstant = (sin_phi - cos_phi * (u - 0.5f * T_g.z)) / (T_f.y * (sin_phi * (u - 0.5f * T_g.z) + cos_phi));
        }
        else
        {
            shiftConstant = (((R - B_y) * (u + 0.5f * T_g.z) + A_y - tau) / (cos_phi + sin_phi * (u + 0.5f * T_g.z)) - startVals_f.y) / T_f.y;
            slopeConstant = (sin_phi - cos_phi * (u + 0.5f * T_g.z)) / (T_f.y * (cos_phi + sin_phi * (u + 0.5f * T_g.z)));
        }
        for (int i = 0; i < N_f.x; i++)
        {
            const float x = (float)i * T_f.x + startVals_f.x;
            const int j = (int)ceil(x * slopeConstant + shiftConstant);
            const float y = (float)j * T_f.y + startVals_f.y;

            if (x * x + y * y > rFOVsq)
                continue;

            const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
            const float num_low = tau - x * sin_phi + y * cos_phi - A_y;
            const float num_high = num_low + 2.0f * A_y;

            const float denom_low = (R_minus_x_dot_theta - B_y) * T_g.z;
            const float denom_high = (R_minus_x_dot_theta + B_y) * T_g.z;

            const float hWeight_0 = max(0.0f, min(num_high / denom_high, n_plus_half) - max(num_low / denom_low, n_minus_half));
            const float hWeight_1 = max(0.0f, min((num_high + Ty_cos) / (denom_high - Ty_sin), n_plus_half) - max((num_low + Ty_cos) / (denom_low - Ty_sin), n_minus_half));
            const float hWeight_2 = max(0.0f,1.0f - hWeight_1 - hWeight_0);

            if (volumeDimensionOrder == 0)
            {
                g_output += tex3D<float>(f, float(iz) + 0.5f, float(j) + 0.5f + hWeight_1/(hWeight_0 + hWeight_1), float(i) + 0.5f) * (hWeight_0 + hWeight_1)
                    + tex3D<float>(f, float(iz) + 0.5f, float(j + 2) + 0.5f, float(i) + 0.5f) * hWeight_2;
            }
            else
            {
                g_output += tex3D<float>(f, float(i) + 0.5f, float(j) + 0.5f+hWeight_1 / (hWeight_0 + hWeight_1), float(iz) + 0.5f) * (hWeight_0 + hWeight_1)
                    + tex3D<float>(f, float(i) + 0.5f, float(j + 2) + 0.5f, float(iz)+0.5f) * hWeight_2;
            }
        }
        g[uint64(l) * uint64(N_g.z * N_g.y) + uint64(m * N_g.z + n)] = T_f.x * sqrt(1.0f + u * u) / fabs(u * sin_phi + cos_phi) * g_output;
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void applyPolarWeight(float* g, int4 N_g, float4 T_g, float4 startVals_g)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N_g.x || j >= N_g.y || k >= N_g.z)
        return;

    const float v = j * T_g.y + startVals_g.y;
    g[uint64(i) * uint64(N_g.z * N_g.y) + uint64(j * N_g.z + k)] *= rsqrt(1.0f + v*v);
}

__global__ void applyInversePolarWeight(float* g, int4 N_g, float4 T_g, float4 startVals_g)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N_g.x || j >= N_g.y || k >= N_g.z)
        return;

    const float v = j * T_g.y + startVals_g.y;
    g[uint64(i) * uint64(N_g.z * N_g.y) + uint64(j * N_g.z + k)] *= sqrt(1.0f + v * v);
}

__global__ void curvedConeBeamHelicalWeightedBackprojectorKernel_SF(hipTextureObject_t g, const int4 N_g, const float4 T_g, const float4 startVals_g, float* f, const int4 N_f, const float4 T_f, const float4 startVals_f, const float R, const float D, const float tau, const float rFOVsq, const float* phis, const int volumeDimensionOrder)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N_f.x || j >= N_f.y || k >= N_f.z)
        return;

    uint64 ind;
    if (volumeDimensionOrder == 0)
        ind = uint64(i) * uint64(N_f.y * N_f.z) + uint64(j * N_f.z + k);
    else
        ind = uint64(k) * uint64(N_f.y * N_f.x) + uint64(j * N_f.x + i);

    const float x = i * T_f.x + startVals_f.x;
    const float y = j * T_f.y + startVals_f.y;
    if (x * x + y * y > rFOVsq)
    {
        f[ind] = 0.0f;
        return;
    }
    const float z = k * T_f.z + startVals_f.z;

    const float T_x_over_2 = 0.5f * T_f.x;
    const float v0_over_Tv = startVals_g.y / T_g.y;
    const float Tz_over_Tv = T_f.z / T_g.y;
    const float v_phi_x_start_num = z / T_g.y;
    const float Tv_inv = 1.0f / T_g.y;
    const float Tu_inv = 1.0f / T_g.z;

    const float atan_term = atan(2.0f * tau * R / (R * R - tau * tau));
    //const int maxTurns = int(ceil((1.0f / d_v_max_inv - 1.0f / d_v_min_inv) * (R + sqrt(x * x + y * y)) / fabs(PI * T_g.w)));

    const float twoPI_inv = 1.0f / (2.0f * PI);
    const float neg_twoPI_pitch = -2.0f * PI * T_g.w;
    const float neg_twoPI_pitch_inv = 1.0f / neg_twoPI_pitch;

    const float v_min = 1.0f / d_v_min_inv;
    const float v_max = 1.0f / d_v_max_inv;

    float val = 0.0f;

    for (int l = 0; l < N_g.x; l++)
    {
        const float phi_cur = phis[l];
        const float z_source = (phi_cur * T_g.w + startVals_g.w);
        const float z_source_over_T_v = z_source * Tv_inv;
        const float sin_phi = sin(phi_cur);
        const float cos_phi = cos(phi_cur);

        const float dist_from_source_components_x = fabs(R * cos_phi + tau * sin_phi - x);
        const float dist_from_source_components_y = fabs(R * sin_phi - tau * cos_phi - y);
        const float dist_from_source = sqrt(dist_from_source_components_x * dist_from_source_components_x + dist_from_source_components_y * dist_from_source_components_y);
        const float dist_from_source_inv = 1.0f / dist_from_source;

        const float v_arg = (z - z_source) * dist_from_source_inv;
        const float centralWeight = helicalConeWeight(v_arg);
        if (centralWeight > 0.0f)
        {
            const float L = (float)l + 0.5f;

            float B_x = (sin_phi < 0.0f) ? -cos_phi * T_x_over_2 : cos_phi * T_x_over_2;
            const float B_y = (cos_phi < 0.0f) ? sin_phi * T_x_over_2 : -sin_phi * T_x_over_2;

            const float l_phi = T_f.x * dist_from_source / max(dist_from_source_components_x, dist_from_source_components_y);

            const float x_dot_theta_perp = cos_phi * y - sin_phi * x + tau;
            const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
            const float R_minus_x_dot_theta_inv = 1.0f / R_minus_x_dot_theta;
            const float u_arg = x_dot_theta_perp * R_minus_x_dot_theta_inv;
            const float x_denom = fabs(u_arg * cos_phi - sin_phi);
            const float y_denom = fabs(u_arg * sin_phi + cos_phi);
            //const float l_phi = T_f.x * sqrt(1.0f + u_arg * u_arg) / max(x_denom, y_denom);
            float A_x;
            if (x_denom > y_denom)
                A_x = fabs(sin_phi) * T_x_over_2;
            else
            {
                A_x = fabs(cos_phi) * T_x_over_2;
                B_x = B_y;
            }
            const float tau_low = (atan((x_dot_theta_perp - A_x) / (R_minus_x_dot_theta - B_x)) - startVals_g.z) * Tu_inv;
            const float tau_high = (atan((x_dot_theta_perp + A_x) / (R_minus_x_dot_theta + B_x)) - startVals_g.z) * Tu_inv;

            float ind_first = floor(tau_low + 0.5f); // first detector index

            const float horizontalWeights_0_A = (min(tau_high, ind_first + 1.5f) - tau_low) * l_phi;
            const float horizontalWeights_1_A = l_phi * (tau_high - tau_low) - horizontalWeights_0_A;

            const float ind_last = ind_first + 2.5f;
            ind_first = ind_first + 0.5f + max(0.0f, min(tau_high - ind_first - 0.5f, 1.0f)) * l_phi / horizontalWeights_0_A;

            const float v_phi_x_step_A = Tz_over_Tv * dist_from_source_inv;
            const float v_phi_x_first = (v_phi_x_start_num - z_source_over_T_v) * dist_from_source_inv - v0_over_Tv;
            
            //const float v_phi_x = (v_phi_x_start_num + k_offset * Tz_over_Tv - z_source_over_T_v) * R_minus_x_dot_theta_inv - v0_over_Tv;
            const float v_phi_x = v_phi_x_first;

            const float row_high_A = floor(v_phi_x - 0.5f * v_phi_x_step_A + 0.5f) + 0.5f;
            const float z_high_A = v_phi_x + 0.5f * v_phi_x_step_A - row_high_A;

            // Calculate the View Redundancy Weight
            const float phi_cur_conj = phi_cur - 2.0f * atan(u_arg) + atan_term + PI;
            const float cos_phi_conj = cos(phi_cur_conj);
            const float sin_phi_conj = sin(phi_cur_conj);
            const float dist_from_source_components_x_conj = fabs(R * cos_phi_conj + tau * sin_phi_conj - x);
            const float dist_from_source_components_y_conj = fabs(R * sin_phi_conj - tau * cos_phi_conj - y);
            const float dist_from_source_conj = sqrt(dist_from_source_components_x_conj * dist_from_source_components_x_conj + dist_from_source_components_y_conj * dist_from_source_components_y_conj);
            const float dist_from_source_inv_conj = 1.0f / dist_from_source_conj;

            float sumWeights = 0.0f;

            //* NEW METHOD
            const float v_arg_shift = neg_twoPI_pitch * dist_from_source_inv;

            const float v_bound_A = (v_arg_shift > 0.0f) ? (v_min - v_arg) * dist_from_source * neg_twoPI_pitch_inv : (v_max - v_arg) * dist_from_source * neg_twoPI_pitch_inv;
            const float v_bound_B = (v_arg_shift < 0.0f) ? (v_min - v_arg) * dist_from_source * neg_twoPI_pitch_inv : (v_max - v_arg) * dist_from_source * neg_twoPI_pitch_inv;

            const int N_turns_below = max(int(ceil((d_phi_start - phi_cur) * twoPI_inv)), int(ceil(v_bound_A)));
            const int N_turns_above = min(int(floor((d_phi_end - phi_cur) * twoPI_inv)), int(floor(v_bound_B)));
            for (int iturn = N_turns_below; iturn <= N_turns_above; iturn++)
            {
                if (iturn != 0)
                    sumWeights += helicalConeWeight(v_arg + iturn * v_arg_shift);
            }

            const float v_arg_conj = (z - (phi_cur_conj * T_g.w + startVals_g.w)) * dist_from_source_inv_conj;
            const float v_arg_shift_conj = neg_twoPI_pitch * dist_from_source_inv_conj;

            const float v_bound_A_conj = (v_arg_shift_conj > 0.0f) ? (v_min - v_arg_conj) * dist_from_source_conj * neg_twoPI_pitch_inv : (v_max - v_arg_conj) * dist_from_source_conj * neg_twoPI_pitch_inv;
            const float v_bound_B_conj = (v_arg_shift_conj < 0.0f) ? (v_min - v_arg_conj) * dist_from_source_conj * neg_twoPI_pitch_inv : (v_max - v_arg_conj) * dist_from_source_conj * neg_twoPI_pitch_inv;

            const int N_turns_below_conj = max(int(ceil((d_phi_start - phi_cur_conj) * twoPI_inv)), int(ceil(v_bound_A_conj)));
            const int N_turns_above_conj = min(int(floor((d_phi_end - phi_cur_conj) * twoPI_inv)), int(floor(v_bound_B_conj)));
            for (int iturn = N_turns_below_conj; iturn <= N_turns_above_conj; iturn++)
            {
                sumWeights += helicalConeWeight(v_arg_conj + iturn * v_arg_shift_conj);
            }

            /*
            const int N_turns_below = min(maxTurns, int(ceil((phi_cur - d_phi_start) * PIINV))) + 1;
            const int N_turns_above = min(maxTurns, int(ceil((d_phi_end - phi_cur) * PIINV))) + 1;

            for (int iturn = -1; iturn >= -N_turns_below; iturn--)
            {
                if (int(-iturn) % 2 == 0)
                {
                    const float phi_turn = phi_cur + iturn * PI;
                    if (phi_turn < d_phi_start)
                        break;
                    sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * dist_from_source_inv);
                }
                else
                {
                    const float phi_turn = phi_cur + phi_conj_shift + iturn * PI;
                    if (phi_turn < d_phi_start)
                        break;
                    sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * dist_from_source_inv_conj);
                }
            }
            for (int iturn = 1; iturn <= N_turns_above; iturn++)
            {
                if (int(iturn) % 2 == 0)
                {
                    const float phi_turn = phi_cur + iturn * PI;
                    if (phi_turn > d_phi_end)
                        break;
                    sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * dist_from_source_inv);
                }
                else
                {
                    const float phi_turn = phi_cur + phi_conj_shift + iturn * PI;
                    if (phi_turn > d_phi_end)
                        break;
                    sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * dist_from_source_inv_conj);
                }
            }
            //*/
            // End-Calculate the View Redundancy Weight
            //sumWeights = centralWeight;

            const float v_weight_one = min(v_phi_x_step_A, v_phi_x_step_A - z_high_A);
            const float v_weight_two = max(0.0f, min(z_high_A, 1.0f));
            const float v_oneAndTwo = v_weight_two / (v_weight_one + v_weight_two);
            const float row_high_plus_two_A = row_high_A + 2.0f;

            if (z_high_A > 1.0f)
            {
                val += ((tex3D<float>(g, ind_first, row_high_A + v_oneAndTwo, L) * horizontalWeights_0_A
                    + tex3D<float>(g, ind_last, row_high_A + v_oneAndTwo, L) * horizontalWeights_1_A) * (v_weight_one + v_weight_two)
                    + (tex3D<float>(g, ind_first, row_high_plus_two_A, L) * horizontalWeights_0_A
                        + tex3D<float>(g, ind_last, row_high_plus_two_A, L) * horizontalWeights_1_A) * (z_high_A - 1.0f)) *centralWeight / (centralWeight + sumWeights);
            }
            else
            {
                val += ((tex3D<float>(g, ind_first, row_high_A + v_oneAndTwo, L) * horizontalWeights_0_A
                    + tex3D<float>(g, ind_last, row_high_A + v_oneAndTwo, L) * horizontalWeights_1_A) * (v_weight_one + v_weight_two)) *centralWeight / (centralWeight + sumWeights);
            }
        }
    }
    f[ind] = val;
}

__global__ void coneBeamHelicalWeightedBackprojectorKernel_SF(hipTextureObject_t g, int4 N_g, float4 T_g, float4 startVals_g, float* f, int4 N_f, float4 T_f, float4 startVals_f, float R, float D, float tau, float rFOVsq, float* phis, int volumeDimensionOrder)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (i >= N_f.x || j >= N_f.y || k >= N_f.z)
        return;

    uint64 ind;
    if (volumeDimensionOrder == 0)
        ind = uint64(i) * uint64(N_f.y * N_f.z) + uint64(j * N_f.z + k);
    else
        ind = uint64(k) * uint64(N_f.y * N_f.x) + uint64(j * N_f.x + i);

    const float x = i * T_f.x + startVals_f.x;
    const float y = j * T_f.y + startVals_f.y;
    if (x * x + y * y > rFOVsq)
    {
        f[ind] = 0.0f;
        return;
    }
    const float z = k * T_f.z + startVals_f.z;

    const float T_x_over_2 = 0.5f * T_f.x;
    const float v0_over_Tv = startVals_g.y / T_g.y;
    const float Tz_over_Tv = T_f.z / T_g.y;
    const float v_phi_x_start_num = z / T_g.y;
    const float Tv_inv = 1.0f / T_g.y;
    const float Tu_inv = 1.0f / T_g.z;

    const float v_min = 1.0f / d_v_min_inv;
    const float v_max = 1.0f / d_v_max_inv;

    float val = 0.0;

    const float atan_term = atan(2.0f * tau * R / (R * R - tau * tau));
    
    // OLD METHOD
    //const int maxTurns = int(ceil((1.0f/d_v_max_inv - 1.0f/d_v_min_inv) * (R + sqrt(x * x + y * y)) / fabs(PI * T_g.w)));

    //* NEW METHOD
    // This is the maximum number of contributions for an angle, counting every 2pi
    // It accounts for the full angular range of the measured projections and the helical pitch
    // There will need to be this many calculations for the ray and the conjugate ray
    //const int maxTurns = min(int(ceil((1.0f / d_v_max_inv - 1.0f / d_v_min_inv) * (R + sqrt(x * x + y * y)) / fabs(2.0f * PI * T_g.w))),
    //    int(ceil((d_phi_end - d_phi_start) / (2.0f * PI))))+1;
    const float twoPI_inv = 1.0f / (2.0f * PI);
    const float neg_twoPI_pitch = -2.0f * PI * T_g.w;
    const float neg_twoPI_pitch_inv = 1.0f / neg_twoPI_pitch;
    //*/

    //const float angularRange_inv = 1.0f / (d_phi_end - d_phi_start);

    for (int l = 0; l < N_g.x; l++)
    {
        const float phi_cur = phis[l];
        const float z_source = (phi_cur * T_g.w + startVals_g.w);
        const float sin_phi = sin(phi_cur);
        const float cos_phi = cos(phi_cur);

        const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
        const float R_minus_x_dot_theta_inv = 1.0f / R_minus_x_dot_theta;

        const float v_arg = (z - z_source) * R_minus_x_dot_theta_inv;
        const float centralWeight = helicalConeWeight(v_arg);
        if (centralWeight > 0.0f)
        {   
            const float L = (float)l + 0.5f;

            float B_x = (sin_phi < 0.0f) ? -cos_phi * T_x_over_2 : cos_phi * T_x_over_2;
            const float B_y = (cos_phi < 0.0f) ? sin_phi * T_x_over_2 : -sin_phi * T_x_over_2;

            const float x_dot_theta_perp = cos_phi * y - sin_phi * x + tau;
            const float u_arg = x_dot_theta_perp * R_minus_x_dot_theta_inv;
            const float x_denom = fabs(u_arg * cos_phi - sin_phi);
            const float y_denom = fabs(u_arg * sin_phi + cos_phi);
            const float l_phi = T_f.x * sqrt(1.0f + u_arg * u_arg) / max(x_denom, y_denom);
            float A_x;
            if (x_denom > y_denom)
                A_x = fabs(sin_phi) * T_x_over_2;
            else
            {
                A_x = fabs(cos_phi) * T_x_over_2;
                B_x = B_y;
            }
            const float tau_low = ((x_dot_theta_perp - A_x) / (R_minus_x_dot_theta - B_x) - startVals_g.z) * Tu_inv;
            const float tau_high = ((x_dot_theta_perp + A_x) / (R_minus_x_dot_theta + B_x) - startVals_g.z) * Tu_inv;

            float ind_first = floor(tau_low + 0.5f); // first detector index

            const float horizontalWeights_0_A = (min(tau_high, ind_first + 1.5f) - tau_low) * l_phi;
            const float horizontalWeights_1_A = l_phi * (tau_high - tau_low) - horizontalWeights_0_A;

            const float ind_last = ind_first + 2.5f;
            ind_first = ind_first + 0.5f + max(0.0f, min(tau_high - ind_first - 0.5f, 1.0f)) * l_phi / horizontalWeights_0_A;

            const float v_phi_x = (v_phi_x_start_num - z_source * Tv_inv) * R_minus_x_dot_theta_inv - v0_over_Tv;
            const float v_phi_x_step_A = Tz_over_Tv * R_minus_x_dot_theta_inv;

            const float row_high_A = floor(v_phi_x - 0.5f * v_phi_x_step_A + 0.5f) + 0.5f;
            const float z_high_A = v_phi_x + 0.5f * v_phi_x_step_A - row_high_A;

            // Calculate the View Redundancy Weight
            const float phi_cur_conj = phi_cur - 2.0f * atan(u_arg) + atan_term + PI;
            //const float R_minus_x_dot_theta_inv_conj = 1.0f / (R - x * cos(phi_cur + phi_conj_shift + PI) - y * sin(phi_cur + phi_conj_shift + PI));
            const float R_minus_x_dot_theta_conj = R - x * cos(phi_cur_conj) - y * sin(phi_cur_conj);
            const float R_minus_x_dot_theta_inv_conj = 1.0f / R_minus_x_dot_theta_conj;
            float sumWeights = 0.0f;

            //* NEW METHOD
            const float v_arg_shift = neg_twoPI_pitch * R_minus_x_dot_theta_inv;

            const float v_bound_A = (v_arg_shift > 0.0f) ? (v_min - v_arg) * R_minus_x_dot_theta * neg_twoPI_pitch_inv : (v_max - v_arg) * R_minus_x_dot_theta * neg_twoPI_pitch_inv;
            const float v_bound_B = (v_arg_shift < 0.0f) ? (v_min - v_arg) * R_minus_x_dot_theta * neg_twoPI_pitch_inv : (v_max - v_arg) * R_minus_x_dot_theta * neg_twoPI_pitch_inv;

            const int N_turns_below = max(int(ceil((d_phi_start - phi_cur) * twoPI_inv)), int(ceil(v_bound_A)));
            const int N_turns_above = min(int(floor((d_phi_end - phi_cur) * twoPI_inv)), int(floor(v_bound_B)));
            for (int iturn = N_turns_below; iturn <= N_turns_above; iturn++)
            {
                //z_source = (phi_cur * T_g.w + startVals_g.w);
                //v_arg = (z - z_source) * R_minus_x_dot_theta_inv;
                // (z - (phi_turn * T_g.w + startVals_g.w))* R_minus_x_dot_theta_inv
                // (z - ((phi_cur + iturn * 2.0f*PI) * T_g.w + startVals_g.w))* R_minus_x_dot_theta_inv
                // (z - (phi_cur*T_g.w + startVals_g.w + (iturn * 2.0f * PI) * T_g.w))* R_minus_x_dot_theta_inv
                // (z - (z_source + (iturn * 2.0f * PI) * T_g.w))* R_minus_x_dot_theta_inv
                // (z- z_source)*R_minus_x_dot_theta_inv - (iturn * 2.0f * PI) * T_g.w*R_minus_x_dot_theta_inv
                // v_arg - (iturn * 2.0f * PI) * T_g.w*R_minus_x_dot_theta_inv
                //const float phi_turn = phi_cur + iturn * 2.0f*PI;
                //if (d_phi_start <= phi_turn && phi_turn <= d_phi_end)
                //sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * R_minus_x_dot_theta_inv);
                if (iturn != 0)
                {
                    sumWeights += helicalConeWeight(v_arg + iturn * v_arg_shift);
                    //sumWeights += tex1D<float>(v_weights, (v_arg + iturn * v_arg_shift - startVals_g.y) * Tv_inv + 0.5f);
                }
            }

            const float v_arg_conj = (z - (phi_cur_conj * T_g.w + startVals_g.w)) * R_minus_x_dot_theta_inv_conj;
            const float v_arg_shift_conj = neg_twoPI_pitch * R_minus_x_dot_theta_inv_conj;

            const float v_bound_A_conj = (v_arg_shift_conj > 0.0f) ? (v_min - v_arg_conj) * R_minus_x_dot_theta_conj * neg_twoPI_pitch_inv : (v_max - v_arg_conj) * R_minus_x_dot_theta_conj * neg_twoPI_pitch_inv;
            const float v_bound_B_conj = (v_arg_shift_conj < 0.0f) ? (v_min - v_arg_conj) * R_minus_x_dot_theta_conj * neg_twoPI_pitch_inv : (v_max - v_arg_conj) * R_minus_x_dot_theta_conj * neg_twoPI_pitch_inv;

            const int N_turns_below_conj = max(int(ceil((d_phi_start - phi_cur_conj) * twoPI_inv)), int(ceil(v_bound_A_conj)));
            const int N_turns_above_conj = min(int(floor((d_phi_end - phi_cur_conj) * twoPI_inv)), int(floor(v_bound_B_conj)));
            for (int iturn = N_turns_below_conj; iturn <= N_turns_above_conj; iturn++)
            {
                //const float phi_turn = phi_cur + phi_conj_shift+PI + iturn * 2.0f * PI;
                //if (d_phi_start <= phi_turn && phi_turn <= d_phi_end)
                //sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * R_minus_x_dot_theta_inv_conj);
                sumWeights += helicalConeWeight(v_arg_conj + iturn * v_arg_shift_conj);
                //sumWeights += tex1D<float>(v_weights, (v_arg_conj + iturn * v_arg_shift_conj - startVals_g.y) * Tv_inv + 0.5f);
            }
            //sumWeights -= centralWeight;
            //*/

            // phi_cur + 2.0f*PI*k, where -N_turns_below <= k <= N_turns_above
            /* OLD METHOD
            const int N_turns_below = min(maxTurns, int(ceil((phi_cur - d_phi_start) * PIINV))) + 1;
            const int N_turns_above = min(maxTurns, int(ceil((d_phi_end - phi_cur) * PIINV))) + 1;

            for (int iturn = -1; iturn >= -N_turns_below; iturn--)
            {
                if (int(-iturn) % 2 == 0)
                {
                    const float phi_turn = phi_cur + iturn * PI;
                    if (phi_turn < d_phi_start)
                        break;
                    sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * R_minus_x_dot_theta_inv);
                    //sumWeights += 1.0f;
                }
                else
                {
                    const float phi_turn = phi_cur + phi_conj_shift + iturn * PI;
                    if (phi_turn < d_phi_start)
                        break;
                    sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * R_minus_x_dot_theta_inv_conj);
                    //sumWeights += 1.0f;
                }
            }
            for (int iturn = 1; iturn <= N_turns_above; iturn++)
            {
                if (int(iturn) % 2 == 0)
                {
                    const float phi_turn = phi_cur + iturn * PI;
                    if (phi_turn > d_phi_end)
                        break;
                    sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * R_minus_x_dot_theta_inv);
                    //sumWeights += 1.0f;
                }
                else
                {
                    const float phi_turn = phi_cur + phi_conj_shift + iturn * PI;
                    if (phi_turn > d_phi_end)
                        break;
                    sumWeights += helicalConeWeight((z - (phi_turn * T_g.w + startVals_g.w)) * R_minus_x_dot_theta_inv_conj);
                    //sumWeights += 1.0f;
                }
            }
            //*/
            //const int N_turns_below = min(maxTurns, int(ceil((phi_cur - d_phi_start) * PIINV))) + 1;
            //const int N_turns_above = min(maxTurns, int(ceil((d_phi_end - phi_cur) * PIINV))) + 1;
            //sumWeights = float(N_turns_above + N_turns_below);
            // End-Calculate the View Redundancy Weight

            //*
            const float v_weight_one = min(v_phi_x_step_A, v_phi_x_step_A - z_high_A);
            const float v_weight_two = max(0.0f, min(z_high_A, 1.0f));
            const float v_oneAndTwo = v_weight_two / (v_weight_one + v_weight_two);
            const float row_high_plus_two_A = row_high_A + 2.0f;
            val += ((tex3D<float>(g, ind_first, row_high_A + v_oneAndTwo, L) * horizontalWeights_0_A
                + tex3D<float>(g, ind_last, row_high_A + v_oneAndTwo, L) * horizontalWeights_1_A) * (v_weight_one + v_weight_two)
                + (tex3D<float>(g, ind_first, row_high_plus_two_A, L) * horizontalWeights_0_A
                    + tex3D<float>(g, ind_last, row_high_plus_two_A, L) * horizontalWeights_1_A) * max(0.0f, z_high_A - 1.0f)) * centralWeight / (centralWeight + sumWeights);
            //*/
            //val += centralWeight / (centralWeight + sumWeights);
        }
    }

    f[ind] = val;
}

__global__ void curvedConeBeamBackprojectorKernel_SF(hipTextureObject_t g, const int4 N_g, const float4 T_g, const float4 startVals_g, float* f, const int4 N_f, const float4 T_f, const float4 startVals_f, const float R, const float D, const float tau, const float rFOVsq, const float* phis, const int volumeDimensionOrder)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = (threadIdx.z + blockIdx.z * blockDim.z) * NUM_SLICES_PER_THREAD;
    if (i >= N_f.x || j >= N_f.y || k >= N_f.z)
        return;

    uint64 ind;
    if (volumeDimensionOrder == 0)
        ind = uint64(i) * uint64(N_f.y * N_f.z) + uint64(j * N_f.z + k);
    else
        ind = uint64(k) * uint64(N_f.y * N_f.x) + uint64(j * N_f.x + i);

    const float x = i * T_f.x + startVals_f.x;
    const float y = j * T_f.y + startVals_f.y;
    const float z = k * T_f.z + startVals_f.z;
    if (x * x + y * y > rFOVsq)
    {
        f[ind] = 0.0f;
        return;
    }

    const float T_x_over_2 = 0.5f * T_f.x;
    const float v0_over_Tv = startVals_g.y / T_g.y;
    const float Tz_over_Tv = T_f.z / T_g.y;
    const float v_phi_x_start_num = z / T_g.y;
    const float Tv_inv = 1.0f / T_g.y;
    const float Tu_inv = 1.0f / T_g.z;

    float vals[NUM_SLICES_PER_THREAD];
    int numZ = min(NUM_SLICES_PER_THREAD, N_f.z - k);
    for (int k_offset = 0; k_offset < numZ; k_offset++)
        vals[k_offset] = 0.0f;

    for (int l = 0; l < N_g.x; l++)
    {
        const float L = (float)l + 0.5f;
        const float z_source_over_T_v = (phis[l] * T_g.w + startVals_g.w) * Tv_inv;
        const float sin_phi = sin(phis[l]);
        const float cos_phi = cos(phis[l]);

        float B_x = (sin_phi < 0.0f) ? -cos_phi * T_x_over_2 : cos_phi * T_x_over_2;
        const float B_y = (cos_phi < 0.0f) ? sin_phi * T_x_over_2 : -sin_phi * T_x_over_2;

        const float x_dot_theta_perp = cos_phi * y - sin_phi * x + tau;
        const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
        const float R_minus_x_dot_theta_inv = 1.0f / R_minus_x_dot_theta;

        const float dist_from_source_components_x = fabs(R * cos_phi + tau * sin_phi - x);
        const float dist_from_source_components_y = fabs(R * sin_phi - tau * cos_phi - y);
        const float dist_from_source = sqrt(dist_from_source_components_x * dist_from_source_components_x + dist_from_source_components_y * dist_from_source_components_y);
        const float l_phi = T_f.x * dist_from_source / max(dist_from_source_components_x, dist_from_source_components_y);

        const float u_arg = x_dot_theta_perp * R_minus_x_dot_theta_inv;
        const float x_denom = fabs(u_arg * cos_phi - sin_phi);
        const float y_denom = fabs(u_arg * sin_phi + cos_phi);
        //const float l_phi = T_f.x * sqrt(1.0f + u_arg * u_arg) / max(x_denom, y_denom);
        float A_x;
        if (x_denom > y_denom)
            A_x = fabs(sin_phi) * T_x_over_2;
        else
        {
            A_x = fabs(cos_phi) * T_x_over_2;
            B_x = B_y;
        }
        const float tau_low = (atan((x_dot_theta_perp - A_x) / (R_minus_x_dot_theta - B_x)) - startVals_g.z) * Tu_inv;
        const float tau_high = (atan((x_dot_theta_perp + A_x) / (R_minus_x_dot_theta + B_x)) - startVals_g.z) * Tu_inv;

        float ind_first = floor(tau_low + 0.5f); // first detector index

        const float horizontalWeights_0_A = (min(tau_high, ind_first + 1.5f) - tau_low) * l_phi;
        const float horizontalWeights_1_A = l_phi * (tau_high - tau_low) - horizontalWeights_0_A;

        const float ind_last = ind_first + 2.5f;
        ind_first = ind_first + 0.5f + max(0.0f, min(tau_high - ind_first - 0.5f, 1.0f)) * l_phi / horizontalWeights_0_A;

        const float v_phi_x_step_A = Tz_over_Tv / dist_from_source;
        const float v_phi_x_first = (v_phi_x_start_num - z_source_over_T_v) / dist_from_source - v0_over_Tv;
        for (int k_offset = 0; k_offset < numZ; k_offset++)
        {
            //const float v_phi_x = (v_phi_x_start_num + k_offset * Tz_over_Tv - z_source_over_T_v) * R_minus_x_dot_theta_inv - v0_over_Tv;
            const float v_phi_x = v_phi_x_first + k_offset * v_phi_x_step_A;

            const float row_high_A = floor(v_phi_x - 0.5f * v_phi_x_step_A + 0.5f) + 0.5f;
            const float z_high_A = v_phi_x + 0.5f * v_phi_x_step_A - row_high_A;

            const float v_weight_one = min(v_phi_x_step_A, v_phi_x_step_A - z_high_A);
            const float v_weight_two = max(0.0f, min(z_high_A, 1.0f));
            const float v_oneAndTwo = v_weight_two / (v_weight_one + v_weight_two);
            const float row_high_plus_two_A = row_high_A + 2.0f;

            if (z_high_A > 1.0f)
            {
                vals[k_offset] += (tex3D<float>(g, ind_first, row_high_A + v_oneAndTwo, L) * horizontalWeights_0_A
                    + tex3D<float>(g, ind_last, row_high_A + v_oneAndTwo, L) * horizontalWeights_1_A) * (v_weight_one + v_weight_two)
                    + (tex3D<float>(g, ind_first, row_high_plus_two_A, L) * horizontalWeights_0_A
                        + tex3D<float>(g, ind_last, row_high_plus_two_A, L) * horizontalWeights_1_A) * (z_high_A - 1.0f);
            }
            else
            {
                vals[k_offset] += (tex3D<float>(g, ind_first, row_high_A + v_oneAndTwo, L) * horizontalWeights_0_A
                    + tex3D<float>(g, ind_last, row_high_A + v_oneAndTwo, L) * horizontalWeights_1_A) * (v_weight_one + v_weight_two);
            }
        }
    }

    if (volumeDimensionOrder == 0)
    {
        for (int k_offset = 0; k_offset < numZ; k_offset++)
            f[ind + uint64(k_offset)] = vals[k_offset];
    }
    else
    {
        for (int k_offset = 0; k_offset < numZ; k_offset++)
            f[ind + uint64(k_offset) * uint64(N_f.y * N_f.x)] = vals[k_offset];
    }
}

__global__ void curvedConeBeamProjectorKernel_SF(float* g, int4 N_g, float4 T_g, float4 startVals_g, hipTextureObject_t f, int4 N_f, float4 T_f, float4 startVals_f, float R, float D, float tau, float rFOVsq, float* phis, int volumeDimensionOrder)
{
    const int l = threadIdx.x + blockIdx.x * blockDim.x;
    const int m = threadIdx.y + blockIdx.y * blockDim.y;
    const int n = threadIdx.z + blockIdx.z * blockDim.z;
    if (l >= N_g.x || m >= N_g.y || n >= N_g.z)
        return;

    const float v = m * T_g.y + startVals_g.y;
    const float u = tan(n * T_g.z + startVals_g.z);

    const float sin_phi = sin(phis[l]);
    const float cos_phi = cos(phis[l]);

    const float n_minus_half = (float)n - 0.5f; // +startVals_g.z / T_g.z;
    const float n_plus_half = (float)n + 0.5f; // +startVals_g.z / T_g.z;
    const float m_minus_half = (float)m - 0.5f;
    const float m_plus_half = (float)m + 0.5f;

    const float u0_over_Tu = startVals_g.z / T_g.z;
    const float one_over_Tu = 1.0f / T_g.z;

    const float Tz_over_Tv = T_f.z / T_g.y;
    const float v0_over_Tv = startVals_g.y / T_g.y;

    const float z_source = phis[l] * T_g.w + startVals_g.w;

    const float z0_over_Tz_plus_half = startVals_f.z / T_f.z + 0.5f;
    const float z_ind_offset = -z0_over_Tz_plus_half + z_source / T_f.z;

    const float z_ind_slope = (v - 0.5f * T_g.y) / T_f.z;

    const float sourcePos_x = R * cos_phi + tau * sin_phi;
    const float sourcePos_y = R * sin_phi - tau * cos_phi;

    float g_output = 0.0f;

    if (fabs(u * cos_phi - sin_phi) > fabs(u * sin_phi + cos_phi))
    {
        const float A_x = fabs(sin_phi) * 0.5f * T_f.x;
        const float B_x = cos_phi * 0.5f * T_f.x * ((sin_phi < 0.0f) ? -1.0f : 1.0f);
        const float Tx_sin = T_f.x * sin_phi;
        //const float Tx_cos = T_g.z * T_f.x * cos_phi; // FIXME?
        const float Tx_cos = T_f.x * cos_phi; // FIXME?

        float shiftConstant, slopeConstant;
        if (u * cos_phi - sin_phi > 0.0f)
        {
            shiftConstant = (((R + B_x) * (u - 0.5f * T_g.z) - A_x - tau) / (cos_phi * (u - 0.5f * T_g.z) - sin_phi) - startVals_f.x) / T_f.x;
            slopeConstant = ((-sin_phi * (u - 0.5f * T_g.z) - cos_phi) / (cos_phi * (u - 0.5f * T_g.z) - sin_phi)) / T_f.x;
        }
        else
        {
            shiftConstant = (((R - B_x) * (u + 0.5f * T_g.z) + A_x - tau) / (cos_phi * (u + 0.5f * T_g.z) - sin_phi) - startVals_f.x) / T_f.x;
            slopeConstant = ((sin_phi * (u + 0.5f * T_g.z) + cos_phi) / (-cos_phi * (u + 0.5f * T_g.z) + sin_phi)) / T_f.x;
        }

        for (int j = 0; j < N_f.y; j++)
        {
            const float y = (float)j * T_f.y + startVals_f.y;
            const int i = (int)ceil(y * slopeConstant + shiftConstant);
            const float x = (float)i * T_f.x + startVals_f.x;

            if (x * x + y * y > rFOVsq)
                continue;

            const float v_denom_inv = rsqrtf((sourcePos_x - x) * (sourcePos_x - x) + (sourcePos_y - y) * (sourcePos_y - y));

            const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
            //const int k = (int)ceil(z_ind_slope * R_minus_x_dot_theta + z_ind_offset);
            const int k = (int)ceil(z_ind_slope / v_denom_inv + z_ind_offset);

            if (k <= -3)
            {
                continue;
            }
            if (k >= N_f.z)
            {
                continue;
            }

            const float num_low = tau - x * sin_phi + y * cos_phi - A_x;
            const float num_high = num_low + 2.0f * A_x;

            const float denom_low = R_minus_x_dot_theta - B_x;
            const float denom_high = R_minus_x_dot_theta + B_x;

            const float hWeight_0 = max(0.0f, min(atan(num_high / denom_high)* one_over_Tu- u0_over_Tu, n_plus_half) - max(atan(num_low / denom_low)* one_over_Tu- u0_over_Tu, n_minus_half));
            const float hWeight_1 = max(0.0f, min(atan((num_high - Tx_sin) / (denom_high - Tx_cos))* one_over_Tu- u0_over_Tu, n_plus_half) - max(atan((num_low - Tx_sin) / (denom_low - Tx_cos))* one_over_Tu- u0_over_Tu, n_minus_half));
            const float hWeight_2 = max(0.0f, 1.0f - hWeight_1 - hWeight_0);

            const float v_phi_x_step = Tz_over_Tv * v_denom_inv;
            const float xi_high = ((float)k - z_ind_offset) * v_phi_x_step - v0_over_Tv;

            const float vWeight_0 = (min(xi_high - m_minus_half, 1.0f)) * ((k >= 0) ? 1.0f : 0.0f);
            const float vWeight_1 = max(0.0f, min(v_phi_x_step, m_plus_half - xi_high)) * ((k >= -1 && k + 1 < N_f.z) ? 1.0f : 0.0f);
            const float vWeight_2 = max(0.0f, min(m_plus_half - xi_high - v_phi_x_step, 1.0f)) * ((k + 2 < N_f.z) ? 1.0f : 0.0f);
            const float x_12 = float(i) + 0.5f + hWeight_1 / (hWeight_0 + hWeight_1);
            const float z_12 = float(k) + 0.5f + vWeight_1 / (vWeight_0 + vWeight_1);
            if (volumeDimensionOrder == 0)
            {
                g_output += (tex3D<float>(f, z_12, float(j) + 0.5f, x_12) * (vWeight_0 + vWeight_1)
                    + tex3D<float>(f, float(k + 2) + 0.5f, float(j) + 0.5f, x_12) * vWeight_2) * (hWeight_0 + hWeight_1)
                    + (tex3D<float>(f, z_12, float(j) + 0.5f, float(i + 2) + 0.5f) * (vWeight_0 + vWeight_1)
                        + tex3D<float>(f, float(k + 2) + 0.5f, float(j) + 0.5f, float(i + 2) + 0.5f) * vWeight_2) * hWeight_2;
            }
            else
            {
                g_output += (tex3D<float>(f, x_12, float(j) + 0.5f, z_12) * (vWeight_0 + vWeight_1)
                    + tex3D<float>(f, x_12, float(j) + 0.5f, float(k + 2) + 0.5f) * vWeight_2) * (hWeight_0 + hWeight_1)
                    + (tex3D<float>(f, float(i + 2) + 0.5f, float(j) + 0.5f, z_12) * (vWeight_0 + vWeight_1)
                        + tex3D<float>(f, float(i + 2) + 0.5f, float(j) + 0.5f, float(k + 2) + 0.5f) * vWeight_2) * hWeight_2;
            }
        }
        g[uint64(l) * uint64(N_g.z * N_g.y) + uint64(m * N_g.z + n)] = T_f.x * sqrt(1.0f + u * u) / fabs(u * cos_phi - sin_phi) * g_output;
    }
    else
    {
        const float A_y = fabs(cos_phi) * 0.5f * T_f.x;
        const float B_y = sin_phi * 0.5f * T_f.x * ((cos_phi < 0.0f) ? 1.0f : -1.0f);
        const float Ty_cos = T_f.y * cos_phi;
        //const float Ty_sin = T_g.z * T_f.y * sin_phi;
        const float Ty_sin = T_f.y * sin_phi;

        float shiftConstant, slopeConstant;
        if (u * sin_phi + cos_phi >= 0.0f)
        {
            shiftConstant = (((R + B_y) * (u - 0.5f * T_g.z) - A_y - tau) / (sin_phi * (u - 0.5f * T_g.z) + cos_phi) - startVals_f.y) / T_f.y;
            slopeConstant = ((sin_phi - cos_phi * (u - 0.5f * T_g.z)) / (sin_phi * (u - 0.5f * T_g.z) + cos_phi)) / T_f.y;
        }
        else
        {
            shiftConstant = (((R - B_y) * (u + 0.5f * T_g.z) + A_y - tau) / (cos_phi + sin_phi * (u + 0.5f * T_g.z)) - startVals_f.y) / T_f.y;
            slopeConstant = ((sin_phi - cos_phi * (u + 0.5f * T_g.z)) / (cos_phi + sin_phi * (u + 0.5f * T_g.z))) / T_f.y;
        }
        for (int i = 0; i < N_f.x; i++)
        {
            const float x = (float)i * T_f.x + startVals_f.x;
            const int j = (int)ceil(x * slopeConstant + shiftConstant);
            const float y = (float)j * T_f.y + startVals_f.y;

            if (x * x + y * y > rFOVsq)
                continue;

            const float v_denom_inv = rsqrtf((sourcePos_x - x) * (sourcePos_x - x) + (sourcePos_y - y) * (sourcePos_y - y));

            const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
            //const int k = (int)ceil(z_ind_slope * R_minus_x_dot_theta + z_ind_offset);
            const int k = (int)ceil(z_ind_slope / v_denom_inv + z_ind_offset);

            if (k <= -3)
            {
                continue;
            }
            if (k >= N_f.z)
            {
                continue;
            }

            const float num_low = tau - x * sin_phi + y * cos_phi - A_y;
            const float num_high = num_low + 2.0f * A_y;

            const float denom_low = R_minus_x_dot_theta - B_y;
            const float denom_high = R_minus_x_dot_theta + B_y;

            const float hWeight_0 = max(0.0f, min(atan(num_high / denom_high)*one_over_Tu- u0_over_Tu, n_plus_half) - max(atan(num_low / denom_low)*one_over_Tu- u0_over_Tu, n_minus_half));
            const float hWeight_1 = max(0.0f, min(atan((num_high + Ty_cos) / (denom_high - Ty_sin))*one_over_Tu- u0_over_Tu, n_plus_half) - max(atan((num_low + Ty_cos) / (denom_low - Ty_sin))*one_over_Tu- u0_over_Tu, n_minus_half));
            const float hWeight_2 = max(0.0f, 1.0f - hWeight_1 - hWeight_0);

            const float v_phi_x_step = Tz_over_Tv * v_denom_inv;
            const float xi_high = ((float)k - z_ind_offset) * v_phi_x_step - v0_over_Tv;

            const float vWeight_0 = (min(xi_high - m_minus_half, 1.0f)) * ((k >= 0) ? 1.0f : 0.0f);
            const float vWeight_1 = max(0.0f, min(v_phi_x_step, m_plus_half - xi_high)) * ((k >= -1 && k + 1 < N_f.z) ? 1.0f : 0.0f);
            const float vWeight_2 = max(0.0f, min(m_plus_half - xi_high - v_phi_x_step, 1.0f)) * ((k + 2 < N_f.z) ? 1.0f : 0.0f);
            const float y_12 = float(j) + 0.5f + hWeight_1 / (hWeight_0 + hWeight_1);
            const float z_12 = float(k) + 0.5f + vWeight_1 / (vWeight_0 + vWeight_1);
            if (volumeDimensionOrder == 0)
            {
                g_output += (tex3D<float>(f, z_12, y_12, float(i) + 0.5f) * (vWeight_0 + vWeight_1)
                    + tex3D<float>(f, float(k + 2) + 0.5f, y_12, float(i) + 0.5f) * vWeight_2) * (hWeight_0 + hWeight_1)
                    + (tex3D<float>(f, z_12, float(j + 2) + 0.5f, float(i) + 0.5f) * (vWeight_0 + vWeight_1)
                        + tex3D<float>(f, float(k + 2) + 0.5f, float(j + 2) + 0.5f, float(i) + 0.5f) * vWeight_2) * hWeight_2;
            }
            else
            {
                g_output += (tex3D<float>(f, float(i) + 0.5f, y_12, z_12) * (vWeight_0 + vWeight_1)
                    + tex3D<float>(f, float(i) + 0.5f, y_12, float(k + 2) + 0.5f) * vWeight_2) * (hWeight_0 + hWeight_1)
                    + (tex3D<float>(f, float(i) + 0.5f, float(j + 2) + 0.5f, z_12) * (vWeight_0 + vWeight_1)
                        + tex3D<float>(f, float(i) + 0.5f, float(j + 2) + 0.5f, float(k + 2) + 0.5f) * vWeight_2) * hWeight_2;
            }
        }
        g[uint64(l) * uint64(N_g.z * N_g.y) + uint64(m * N_g.z + n)] = T_f.x * sqrt(1.0f + u * u) / fabs(u * sin_phi + cos_phi) * g_output;
    }
}

__global__ void coneBeamBackprojectorKernel_SF(hipTextureObject_t g, const int4 N_g, const float4 T_g, const float4 startVals_g, float* f, const int4 N_f, const float4 T_f, const float4 startVals_f, const float R, const float D, const float tau, const float rFOVsq, const float* phis, const int volumeDimensionOrder)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = (threadIdx.z + blockIdx.z * blockDim.z)*NUM_SLICES_PER_THREAD;
    if (i >= N_f.x || j >= N_f.y || k >= N_f.z)
        return;

    uint64 ind;
    if (volumeDimensionOrder == 0)
        ind = uint64(i) * uint64(N_f.y * N_f.z) + uint64(j * N_f.z + k);
    else
        ind = uint64(k) * uint64(N_f.y * N_f.x) + uint64(j * N_f.x + i);

    const float x = i * T_f.x + startVals_f.x;
    const float y = j * T_f.y + startVals_f.y;
    const float z = k * T_f.z + startVals_f.z;
    if (x * x + y * y > rFOVsq)
    {
        f[ind] = 0.0f;
        return;
    }

    const float T_x_over_2 = 0.5f * T_f.x;
    const float v0_over_Tv = startVals_g.y / T_g.y;
    const float Tz_over_Tv = T_f.z / T_g.y;
    const float v_phi_x_start_num = z / T_g.y;
    const float Tv_inv = 1.0f / T_g.y;
    const float Tu_inv = 1.0f / T_g.z;

    float vals[NUM_SLICES_PER_THREAD];
    int numZ = min(NUM_SLICES_PER_THREAD, N_f.z - k);
    for (int k_offset = 0; k_offset < numZ; k_offset++)
        vals[k_offset] = 0.0f;

    for (int l = 0; l < N_g.x; l++)
    {
        const float L = (float)l + 0.5f;
        const float z_source_over_T_v = (phis[l] * T_g.w + startVals_g.w) * Tv_inv;
        const float sin_phi = sin(phis[l]);
        const float cos_phi = cos(phis[l]);

        float B_x = (sin_phi < 0.0f) ? -cos_phi * T_x_over_2 : cos_phi * T_x_over_2;
        const float B_y = (cos_phi < 0.0f) ? sin_phi * T_x_over_2 : -sin_phi * T_x_over_2;

        const float x_dot_theta_perp = cos_phi * y - sin_phi * x + tau;
        const float R_minus_x_dot_theta = R - x * cos_phi - y * sin_phi;
        const float R_minus_x_dot_theta_inv = 1.0f / R_minus_x_dot_theta;

        const float u_arg = x_dot_theta_perp * R_minus_x_dot_theta_inv;
        const float x_denom = fabs(u_arg * cos_phi - sin_phi);
        const float y_denom = fabs(u_arg * sin_phi + cos_phi);
        const float l_phi = T_f.x * sqrt(1.0f + u_arg * u_arg) / max(x_denom, y_denom);
        float A_x;
        if (x_denom > y_denom)
            A_x = fabs(sin_phi) * T_x_over_2;
        else
        {
            A_x = fabs(cos_phi) * T_x_over_2;
            B_x = B_y;
        }
        const float tau_low = ((x_dot_theta_perp - A_x) / (R_minus_x_dot_theta - B_x) - startVals_g.z) * Tu_inv;
        const float tau_high = ((x_dot_theta_perp + A_x) / (R_minus_x_dot_theta + B_x) - startVals_g.z) * Tu_inv;

        float ind_first = floor(tau_low + 0.5f); // first detector index

        const float horizontalWeights_0_A = (min(tau_high, ind_first + 1.5f) - tau_low) * l_phi;
        const float horizontalWeights_1_A = l_phi * (tau_high - tau_low) - horizontalWeights_0_A;

        const float ind_last = ind_first + 2.5f;
        ind_first = ind_first + 0.5f + max(0.0f, min(tau_high - ind_first - 0.5f, 1.0f)) * l_phi / horizontalWeights_0_A;

        const float v_phi_x_step_A = Tz_over_Tv * R_minus_x_dot_theta_inv;
        const float v_phi_x_first = (v_phi_x_start_num - z_source_over_T_v) * R_minus_x_dot_theta_inv - v0_over_Tv;
        for (int k_offset = 0; k_offset < numZ; k_offset++)
        {
            //const float v_phi_x = (v_phi_x_start_num + k_offset * Tz_over_Tv - z_source_over_T_v) * R_minus_x_dot_theta_inv - v0_over_Tv;
            const float v_phi_x = v_phi_x_first + k_offset * v_phi_x_step_A;

            const float row_high_A = floor(v_phi_x - 0.5f * v_phi_x_step_A + 0.5f) + 0.5f;
            const float z_high_A = v_phi_x + 0.5f * v_phi_x_step_A - row_high_A;

            const float v_weight_one = min(v_phi_x_step_A, v_phi_x_step_A - z_high_A);
            const float v_weight_two = max(0.0f, min(z_high_A, 1.0f));
            const float v_oneAndTwo = v_weight_two / (v_weight_one + v_weight_two);
            const float row_high_plus_two_A = row_high_A + 2.0f;

            if (z_high_A > 1.0f)
            {
                vals[k_offset] += (tex3D<float>(g, ind_first, row_high_A + v_oneAndTwo, L) * horizontalWeights_0_A
                    + tex3D<float>(g, ind_last, row_high_A + v_oneAndTwo, L) * horizontalWeights_1_A) * (v_weight_one + v_weight_two)
                    + (tex3D<float>(g, ind_first, row_high_plus_two_A, L) * horizontalWeights_0_A
                        + tex3D<float>(g, ind_last, row_high_plus_two_A, L) * horizontalWeights_1_A) * (z_high_A - 1.0f);
            }
            else
            {
                vals[k_offset] += (tex3D<float>(g, ind_first, row_high_A + v_oneAndTwo, L) * horizontalWeights_0_A
                    + tex3D<float>(g, ind_last, row_high_A + v_oneAndTwo, L) * horizontalWeights_1_A) * (v_weight_one + v_weight_two);
            }
        }
    }

    if (volumeDimensionOrder == 0)
    {
        for (int k_offset = 0; k_offset < numZ; k_offset++)
            f[ind + uint64(k_offset)] = vals[k_offset];
    }
    else
    {
        for (int k_offset = 0; k_offset < numZ; k_offset++)
            f[ind + uint64(k_offset) * uint64(N_f.y * N_f.x)] = vals[k_offset];
    }
}

__global__ void coneBeamProjectorKernel_SF(float* g, int4 N_g, float4 T_g, float4 startVals_g, hipTextureObject_t f, int4 N_f, float4 T_f, float4 startVals_f, float R, float D, float tau, float rFOVsq, float* phis, int volumeDimensionOrder)
{
    const int l = threadIdx.x + blockIdx.x * blockDim.x;
    const int m = threadIdx.y + blockIdx.y * blockDim.y;
    const int n = threadIdx.z + blockIdx.z * blockDim.z;
    if (l >= N_g.x || m >= N_g.y || n >= N_g.z)
        return;

    const float v = m * T_g.y + startVals_g.y;
    const float u = n * T_g.z + startVals_g.z;
    
    const float sin_phi = sin(phis[l]);
    const float cos_phi = cos(phis[l]);

    const float n_minus_half = (float)n - 0.5f + startVals_g.z / T_g.z;
    const float n_plus_half = (float)n + 0.5f + startVals_g.z / T_g.z;
    const float m_minus_half = (float)m - 0.5f;
    const float m_plus_half = (float)m + 0.5f;

    const float v0_over_Tv = startVals_g.y / T_g.y;

    const float z_source = phis[l] * T_g.w + startVals_g.w;

    const float z0_over_Tz_plus_half = startVals_f.z / T_f.z + 0.5f;
    const float z_ind_offset = -z0_over_Tz_plus_half + z_source/T_f.z;

    const float z_ind_slope = (v - 0.5f*T_g.y) / T_f.z;

    float g_output = 0.0f;
    
     if (fabs(u*cos_phi-sin_phi) > fabs(u*sin_phi+cos_phi))
     {
         const float A_x = fabs(sin_phi) * 0.5f*T_f.x;
         const float B_x = cos_phi * 0.5f*T_f.x * ((sin_phi < 0.0f) ? -1.0f : 1.0f);
         const float Tx_sin = T_f.x*sin_phi;
         const float Tx_cos = T_g.z*T_f.x*cos_phi;

         float shiftConstant, slopeConstant;
         if (u*cos_phi - sin_phi > 0.0f)
         {
             shiftConstant = (((R + B_x)*(u - 0.5f*T_g.z) - A_x - tau) / (cos_phi*(u-0.5f*T_g.z)-sin_phi) - startVals_f.x) / T_f.x;
             slopeConstant = (-sin_phi*(u-0.5f*T_g.z)-cos_phi) / (T_f.x*(cos_phi*(u-0.5f*T_g.z)-sin_phi) );
         }
         else
         {
             shiftConstant = (((R - B_x)*(u + 0.5f*T_g.z) + A_x - tau) / (cos_phi*(u+0.5f*T_g.z)-sin_phi) - startVals_f.x) / T_f.x;
             slopeConstant = (sin_phi*(u+0.5f*T_g.z)+cos_phi) / (T_f.x*(-cos_phi*(u+0.5f*T_g.z)+sin_phi) );
         }

         for (int j = 0; j < N_f.y; j++)
         {
             const float y = (float)j * T_f.y + startVals_f.y;
             const int i = (int)ceil(y * slopeConstant +  shiftConstant);
             const float x = (float)i * T_f.x + startVals_f.x;

            if (x*x + y*y > rFOVsq)
                continue;

             const float R_minus_x_dot_theta = R - x*cos_phi - y*sin_phi;
             const int k = (int)ceil(  z_ind_slope*R_minus_x_dot_theta + z_ind_offset  );

             if (k <= -3)
             {
                 if (z_ind_slope*sin_phi > 0.0f)
                     break;
                 else
                     continue;
             }
             if (k >= N_f.z)
             {
                 if (z_ind_slope*sin_phi < 0.0f)
                     break;
                 else
                     continue;
             }

             const float num_low = tau - x*sin_phi + y*cos_phi - A_x;
             const float num_high = num_low + 2.0f*A_x;

             const float denom_low = (R_minus_x_dot_theta - B_x) * T_g.z;
             const float denom_high = (R_minus_x_dot_theta + B_x) * T_g.z;

             const float hWeight_0 = max(0.0f, min( num_high/denom_high, n_plus_half ) - max( num_low/denom_low, n_minus_half ) );
             const float hWeight_1 = max(0.0f, min( (num_high-Tx_sin)/(denom_high-Tx_cos), n_plus_half ) - max( (num_low-Tx_sin)/(denom_low-Tx_cos), n_minus_half ) );
             const float hWeight_2 = max(0.0f, 1.0f - hWeight_1 - hWeight_0);

             const float v_phi_x_step = T_f.z / (T_g.y*R_minus_x_dot_theta);
             const float xi_high = ((float)k - z_ind_offset) * v_phi_x_step - v0_over_Tv;

             const float vWeight_0 = (min(xi_high - m_minus_half, 1.0f)) * ((k >= 0) ? 1.0f : 0.0f);
             const float vWeight_1 = max(0.0f, min(v_phi_x_step, m_plus_half - xi_high)) * ((k >= -1 && k + 1 < N_f.z) ? 1.0f : 0.0f);
             const float vWeight_2 = max(0.0f, min(m_plus_half - xi_high - v_phi_x_step, 1.0f)) * ((k + 2 < N_f.z) ? 1.0f : 0.0f);
             const float x_12 = float(i) + 0.5f + hWeight_1 / (hWeight_0 + hWeight_1);
             const float z_12 = float(k) + 0.5f + vWeight_1 / (vWeight_0 + vWeight_1);
             if (volumeDimensionOrder == 0)
             {
                 g_output += (tex3D<float>(f, z_12, float(j) + 0.5f, x_12) * (vWeight_0 + vWeight_1)
                     + tex3D<float>(f, float(k + 2) + 0.5f, float(j) + 0.5f, x_12) * vWeight_2) * (hWeight_0 + hWeight_1)
                     + (tex3D<float>(f, z_12, float(j) + 0.5f, float(i + 2) + 0.5f) * (vWeight_0 + vWeight_1)
                         + tex3D<float>(f, float(k + 2) + 0.5f, float(j) + 0.5f, float(i + 2) + 0.5f) * vWeight_2) * hWeight_2;
             }
             else
             {
                 g_output += (tex3D<float>(f, x_12, float(j) + 0.5f, z_12) * (vWeight_0 + vWeight_1)
                 + tex3D<float>(f, x_12, float(j) + 0.5f, float(k+2) + 0.5f) * vWeight_2) * (hWeight_0 + hWeight_1)
                 + (tex3D<float>(f, float(i + 2) + 0.5f, float(j) + 0.5f, z_12) * (vWeight_0 + vWeight_1)
                 + tex3D<float>(f, float(i + 2) + 0.5f, float(j) + 0.5f, float(k + 2) + 0.5f) * vWeight_2) * hWeight_2;
             }
         }
         g[uint64(l) * uint64(N_g.z * N_g.y) + uint64(m * N_g.z + n)] = T_f.x * sqrt(1.0f+u*u) / fabs(u*cos_phi-sin_phi) * g_output;
     }
     else
     {
         const float A_y = fabs(cos_phi) * 0.5f*T_f.x;
         const float B_y = sin_phi * 0.5f*T_f.x * ((cos_phi < 0.0f) ? 1.0f : -1.0f);
         const float Ty_cos = T_f.y*cos_phi;
         const float Ty_sin = T_g.z*T_f.y*sin_phi;

         float shiftConstant, slopeConstant;
         if (u*sin_phi + cos_phi >= 0.0f)
         {
             shiftConstant = (((R + B_y)*(u - 0.5f*T_g.z) - A_y - tau) / (sin_phi*(u-0.5f*T_g.z)+cos_phi) - startVals_f.y) / T_f.y;
             slopeConstant = (sin_phi-cos_phi*(u-0.5f*T_g.z)) / (T_f.y*(sin_phi*(u-0.5f*T_g.z)+cos_phi) );
         }
         else
         {
             shiftConstant = (((R - B_y)*(u + 0.5f*T_g.z) + A_y - tau) / (cos_phi+sin_phi*(u+0.5f*T_g.z)) - startVals_f.y) / T_f.y;
             slopeConstant = (sin_phi-cos_phi*(u+0.5f*T_g.z)) / (T_f.y*(cos_phi+sin_phi*(u+0.5f*T_g.z)) );
         }
         for (int i = 0; i < N_f.x; i++)
         {
             const float x = (float)i * T_f.x + startVals_f.x;
             const int j = (int)ceil( x * slopeConstant + shiftConstant);
             const float y = (float)j * T_f.y + startVals_f.y;

            if (x*x + y*y > rFOVsq)
                continue;

             const float R_minus_x_dot_theta = R - x*cos_phi - y*sin_phi;
             const int k = (int)ceil(  z_ind_slope*R_minus_x_dot_theta + z_ind_offset  );

             if (k <= -3)
             {
                 if (z_ind_slope*cos_phi > 0.0f)
                     break;
                 else
                     continue;
             }
             if (k >= N_f.z)
             {
                 if (z_ind_slope*cos_phi < 0.0f)
                     break;
                 else
                     continue;
             }

             const float num_low = tau - x*sin_phi + y*cos_phi - A_y;
             const float num_high = num_low + 2.0f*A_y;

             const float denom_low = (R_minus_x_dot_theta - B_y) * T_g.z;
             const float denom_high = (R_minus_x_dot_theta + B_y) * T_g.z;

             const float hWeight_0 = max(0.0f, min( num_high/denom_high, n_plus_half ) - max( num_low/denom_low, n_minus_half ) );
             const float hWeight_1 = max(0.0f, min( (num_high+Ty_cos)/(denom_high-Ty_sin), n_plus_half ) - max( (num_low+Ty_cos)/(denom_low-Ty_sin), n_minus_half ) );
             const float hWeight_2 = max(0.0f, 1.0f - hWeight_1 - hWeight_0);

             const float v_phi_x_step = T_f.z / (T_g.y*R_minus_x_dot_theta);
             const float xi_high = ((float)k - z_ind_offset) * v_phi_x_step - v0_over_Tv;

             const float vWeight_0 = (min(xi_high - m_minus_half, 1.0f)) * ((k >= 0) ? 1.0f : 0.0f);
             const float vWeight_1 = max(0.0f, min(v_phi_x_step, m_plus_half - xi_high)) * ((k >= -1 && k + 1 < N_f.z) ? 1.0f : 0.0f);
             const float vWeight_2 = max(0.0f, min(m_plus_half - xi_high - v_phi_x_step, 1.0f)) * ((k + 2 < N_f.z) ? 1.0f : 0.0f);
             const float y_12 = float(j) + 0.5f + hWeight_1 / (hWeight_0 + hWeight_1);
             const float z_12 = float(k) + 0.5f + vWeight_1 / (vWeight_0 + vWeight_1);
             if (volumeDimensionOrder == 0)
             {
                 g_output += (tex3D<float>(f, z_12, y_12, float(i) + 0.5f) * (vWeight_0 + vWeight_1)
                     + tex3D<float>(f, float(k + 2) + 0.5f, y_12, float(i) + 0.5f) * vWeight_2) * (hWeight_0 + hWeight_1)
                     + (tex3D<float>(f, z_12, float(j + 2) + 0.5f, float(i) + 0.5f) * (vWeight_0 + vWeight_1)
                         + tex3D<float>(f, float(k + 2) + 0.5f, float(j + 2) + 0.5f, float(i) + 0.5f) * vWeight_2) * hWeight_2;
             }
             else
             {
                 g_output += (tex3D<float>(f, float(i) + 0.5f, y_12, z_12) * (vWeight_0 + vWeight_1)
                     + tex3D<float>(f, float(i) + 0.5f, y_12, float(k + 2) + 0.5f) * vWeight_2) * (hWeight_0 + hWeight_1)
                     + (tex3D<float>(f, float(i) + 0.5f, float(j + 2) + 0.5f, z_12) * (vWeight_0 + vWeight_1)
                     + tex3D<float>(f, float(i) + 0.5f, float(j + 2) + 0.5f, float(k + 2) + 0.5f) * vWeight_2) * hWeight_2;
             }
         }
         g[uint64(l) * uint64(N_g.z * N_g.y) + uint64(m * N_g.z + n)] = T_f.x * sqrt(1.0f+u*u) / fabs(u*sin_phi+cos_phi) * g_output;
     }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Main Routines
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*
bool project_SF_modular(float*& g, float* f, parameters* params, bool data_on_cpu)
{
    return project_SF(g, f, params, data_on_cpu);
}

bool backproject_SF_modular(float* g, float*& f, parameters* params, bool data_on_cpu)
{
    return backproject_SF(g, f, params, data_on_cpu);
}
//*/

bool project_SF_fan(float*& g, float* f, parameters* params, bool data_on_cpu)
{
    return project_SF(g, f, params, data_on_cpu);
}

bool backproject_SF_fan(float* g, float*& f, parameters* params, bool data_on_cpu)
{
    return backproject_SF(g, f, params, data_on_cpu);
}

bool project_SF_parallel(float*& g, float* f, parameters* params, bool data_on_cpu)
{
    return project_SF(g, f, params, data_on_cpu);
}


bool backproject_SF_parallel(float* g, float*& f, parameters* params, bool data_on_cpu)
{
    return backproject_SF(g, f, params, data_on_cpu);
}

bool project_SF_cone(float*& g, float* f, parameters* params, bool data_on_cpu)
{
    return project_SF(g, f, params, data_on_cpu);
}

bool backproject_SF_cone(float* g, float*& f, parameters* params, bool data_on_cpu)
{
    return backproject_SF(g, f, params, data_on_cpu);
}

bool project_SF(float *&g, float *f, parameters* params, bool data_on_cpu)
{
    if (params->voxelSizeWorksForFastSF() == false)
    {
        //printf("using extended\n");
        if (params->voxelWidth < params->default_voxelWidth() || params->voxelHeight < params->default_voxelHeight())
            return project_eSF(g, f, params, data_on_cpu);
    }
    if (g == NULL || f == NULL || params == NULL || params->allDefined() == false)
        return false;

    hipSetDevice(params->whichGPU);
    hipError_t cudaStatus;

    float* dev_g = 0;
    float* dev_f = 0;

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Allocate projection data on GPU
    int4 N_g; float4 T_g; float4 startVal_g;
    setProjectionGPUparams(params, N_g, T_g, startVal_g, true);
    
    float rFOVsq = params->rFOV()*params->rFOV();
    
    if (data_on_cpu)
    {
        if ((cudaStatus = hipMalloc((void**)&dev_g, params->projectionData_numberOfElements() * sizeof(float))) != hipSuccess)
        {
            fprintf(stderr, "hipMalloc(projections) failed!\n");
        }
    }
    else
        dev_g = g;

    float* dev_phis = copyAngleArrayToGPU(params);

    int4 N_f; float4 T_f; float4 startVal_f;
    setVolumeGPUparams(params, N_f, T_f, startVal_f);

    if (data_on_cpu)
        dev_f = copyVolumeDataToGPU(f, params, params->whichGPU);
    else
        dev_f = f;

    hipTextureObject_t d_data_txt = NULL;
    //hipArray* d_data_array = loadTexture(d_data_txt, dev_f, N_f, false, bool(params->geometry == parameters::CONE), bool(params->volumeDimensionOrder == 1));
    hipArray* d_data_array = loadTexture(d_data_txt, dev_f, N_f, false, true, bool(params->volumeDimensionOrder == 1));

    // Call Kernel
    dim3 dimBlock = setBlockSize(N_g);
    dim3 dimGrid = setGridSize(N_g, dimBlock);
    if (params->geometry == parameters::CONE)
    {
        if (params->detectorType == parameters::FLAT)
            coneBeamProjectorKernel_SF <<< dimGrid, dimBlock >>> (dev_g, N_g, T_g, startVal_g, d_data_txt, N_f, T_f, startVal_f, params->sod, params->sdd, params->tau, rFOVsq, dev_phis, params->volumeDimensionOrder);
        else
        {
            curvedConeBeamProjectorKernel_SF <<< dimGrid, dimBlock >>> (dev_g, N_g, T_g, startVal_g, d_data_txt, N_f, T_f, startVal_f, params->sod, params->sdd, params->tau, rFOVsq, dev_phis, params->volumeDimensionOrder);
        }
        applyInversePolarWeight <<< dimGrid, dimBlock >>> (dev_g, N_g, T_g, startVal_g);
    }
    else if (params->geometry == parameters::FAN)
    {
        fanBeamProjectorKernel_SF <<< dimGrid, dimBlock >>> (dev_g, N_g, T_g, startVal_g, d_data_txt, N_f, T_f, startVal_f, params->sod, params->sdd, params->tau, rFOVsq, dev_phis, params->volumeDimensionOrder);
    }
    else if (params->geometry == parameters::PARALLEL)
    {
        parallelBeamProjectorKernel_SF <<< dimGrid, dimBlock >>> (dev_g, N_g, T_g, startVal_g, d_data_txt, N_f, T_f, startVal_f, rFOVsq, dev_phis, params->volumeDimensionOrder);
    }

    // pull result off GPU
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "kernel failed!\n");
        fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
        fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
    }

    if (data_on_cpu)
        pullProjectionDataFromGPU(g, params, dev_g, params->whichGPU);
    else
        g = dev_g;

    // Clean up
    hipFreeArray(d_data_array);
    hipDestroyTextureObject(d_data_txt);
    hipFree(dev_phis);

    if (data_on_cpu)
    {
        if (dev_g != 0)
            hipFree(dev_g);
        if (dev_f != 0)
            hipFree(dev_f);
    }

    return true;
}

bool backproject_SF(float *g, float *&f, parameters* params, bool data_on_cpu)
{
    if (g == NULL || f == NULL || params == NULL || params->allDefined() == false)
        return false;
    if (params->voxelSizeWorksForFastSF() == false)
    {
        //printf("using extended\n");
        if (params->voxelWidth > params->default_voxelWidth() || params->voxelHeight > params->default_voxelHeight())
            return backproject_eSF(g, f, params, data_on_cpu);
    }

    hipSetDevice(params->whichGPU);
    hipError_t cudaStatus;

    float* dev_g = 0;
    float* dev_f = 0;

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Allocate volume data on GPU
    int4 N_f; float4 T_f; float4 startVal_f;
    setVolumeGPUparams(params, N_f, T_f, startVal_f);

    if (data_on_cpu)
    {
        //printf("mallocing %.0f elements\n", float(params->volumeData_numberOfElements()));
        if ((cudaStatus = hipMalloc((void**)&dev_f, params->volumeData_numberOfElements() * sizeof(float))) != hipSuccess)
        {
            fprintf(stderr, "hipMalloc(volume) failed!\n");
        }
    }
    else
        dev_f = f;

    float* dev_phis = copyAngleArrayToGPU(params);

    int4 N_g; float4 T_g; float4 startVal_g;
    setProjectionGPUparams(params, N_g, T_g, startVal_g, true);
    
    float rFOVsq = params->rFOV()*params->rFOV();
    
    if (data_on_cpu)
        dev_g = copyProjectionDataToGPU(g, params, params->whichGPU);
    else
        dev_g = g;

    dim3 dimBlock_g = setBlockSize(N_g);
    dim3 dimGrid_g = setGridSize(N_g, dimBlock_g);
    if (params->geometry == parameters::CONE)
    {
        applyInversePolarWeight <<< dimGrid_g, dimBlock_g >>> (dev_g, N_g, T_g, startVal_g);
    }

    hipTextureObject_t d_data_txt = NULL;
    //hipArray* d_data_array = loadTexture(d_data_txt, dev_g, N_g, false, true);
    hipArray* d_data_array = loadTexture(d_data_txt, dev_g, N_g, params->doExtrapolation, true);

    // Call Kernel
    dim3 dimBlock = setBlockSize(N_f);
    dim3 dimGrid = setGridSize(N_f, dimBlock);

    int4 N_f_mod = make_int4(N_f.x, N_f.y, int(ceil(float(N_f.z)/float(NUM_SLICES_PER_THREAD))), N_f.w);
    dim3 dimBlock_slab = setBlockSize(N_f_mod);
    dim3 dimGrid_slab = setGridSize(N_f_mod, dimBlock_slab);
    if (params->geometry == parameters::PARALLEL)
    {
        parallelBeamBackprojectorKernel_SF <<< dimGrid_slab, dimBlock_slab >>> (d_data_txt, N_g, T_g, startVal_g, dev_f, N_f, T_f, startVal_f, rFOVsq, dev_phis, params->volumeDimensionOrder);
    }
    else if (params->geometry == parameters::FAN)
    {
        fanBeamBackprojectorKernel_SF <<< dimGrid_slab, dimBlock_slab >>> (d_data_txt, N_g, T_g, startVal_g, dev_f, N_f, T_f, startVal_f, params->sod, params->sdd, params->tau, rFOVsq, dev_phis, params->volumeDimensionOrder, params->doWeightedBackprojection);
    }
    else if (params->geometry == parameters::CONE)
    {
        if (params->doWeightedBackprojection && params->helicalPitch != 0.0)
        {
            float q_helical = 0.7;
            float weightFcnParameter = -2.0 / ((1.0 - q_helical) * (1.0 - q_helical));
            float weightFcnTransition = (q_helical + 1.0) / 2.0;
            float v_min_inv = (params->v(0) - 0.5 * params->pixelHeight) / params->sdd;
            v_min_inv = 1.0 / v_min_inv;
            float v_max_inv = (params->v(params->numRows - 1) + 0.5 * params->pixelHeight) / params->sdd;
            v_max_inv = 1.0 / v_max_inv;
            float phi_start = params->get_phi_start();
            float phi_end = params->get_phi_end();

            /*
            float* v_weights = new float[params->numRows];
            for (int i = 0; i < params->numRows; i++)
            {
                float v = startVal_g.y + i * T_g.y;

                float abs_v_hat = (v >= 0.0f) ? v * v_max_inv : v * v_min_inv;

                if (abs_v_hat <= q_helical)
                    v_weights[i] = 1.0;
                else if (abs_v_hat > 1.0)
                    v_weights[i] = 0.0;
                else if (abs_v_hat <= weightFcnTransition)
                    v_weights[i] = weightFcnParameter * (abs_v_hat - q_helical) * (abs_v_hat - q_helical) + 1.0;
                else
                    v_weights[i] = -1.0 * weightFcnParameter * (abs_v_hat - 1.0) * (abs_v_hat - 1.0);
                //printf("cpu %f: %f\n", v, v_weights[i]);
            }
            hipTextureObject_t d_v_weights_txt = NULL;
            hipArray* d_v_weights_array = loadTexture1D(d_v_weights_txt, v_weights, params->numRows, false, true);
            //*/

            //printf("v_min/max = %f, %f\n", v_min, v_max);
            //printf("weight params: %f, %f\n", weightFcnParameter, weightFcnTransition);

            hipMemcpyToSymbol(HIP_SYMBOL(d_q_helical), &q_helical, sizeof(float));
            hipMemcpyToSymbol(HIP_SYMBOL(d_v_min_inv), &v_min_inv, sizeof(float));
            hipMemcpyToSymbol(HIP_SYMBOL(d_v_max_inv), &v_max_inv, sizeof(float));
            hipMemcpyToSymbol(HIP_SYMBOL(d_weightFcnTransition), &weightFcnTransition, sizeof(float));
            hipMemcpyToSymbol(HIP_SYMBOL(d_weightFcnParameter), &weightFcnParameter, sizeof(float));
            hipMemcpyToSymbol(HIP_SYMBOL(d_phi_start), &phi_start, sizeof(float));
            hipMemcpyToSymbol(HIP_SYMBOL(d_phi_end), &phi_end, sizeof(float));

            if (params->detectorType == parameters::FLAT)
                coneBeamHelicalWeightedBackprojectorKernel_SF <<< dimGrid, dimBlock >>> (d_data_txt, N_g, T_g, startVal_g, dev_f, N_f, T_f, startVal_f, params->sod, params->sdd, params->tau, rFOVsq, dev_phis, params->volumeDimensionOrder);
            else
                curvedConeBeamHelicalWeightedBackprojectorKernel_SF <<< dimGrid, dimBlock >>> (d_data_txt, N_g, T_g, startVal_g, dev_f, N_f, T_f, startVal_f, params->sod, params->sdd, params->tau, rFOVsq, dev_phis, params->volumeDimensionOrder);

            //hipFreeArray(d_v_weights_array);
            //hipDestroyTextureObject(d_v_weights_txt);
            //delete[] v_weights;
        }
        else
        {
            if (params->detectorType == parameters::FLAT)
                coneBeamBackprojectorKernel_SF <<< dimGrid_slab, dimBlock_slab >>> (d_data_txt, N_g, T_g, startVal_g, dev_f, N_f, T_f, startVal_f, params->sod, params->sdd, params->tau, rFOVsq, dev_phis, params->volumeDimensionOrder);
            else
                curvedConeBeamBackprojectorKernel_SF <<< dimGrid_slab, dimBlock_slab >>> (d_data_txt, N_g, T_g, startVal_g, dev_f, N_f, T_f, startVal_f, params->sod, params->sdd, params->tau, rFOVsq, dev_phis, params->volumeDimensionOrder);
        }
    }
    else
        return false;

    // pull result off GPU
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "kernel failed!\n");
        fprintf(stderr, "error name: %s\n", hipGetErrorName(cudaStatus));
        fprintf(stderr, "error msg: %s\n", hipGetErrorString(cudaStatus));
    }
    //*
    if (data_on_cpu)
        pullVolumeDataFromGPU(f, params, dev_f, params->whichGPU);
    else
        f = dev_f;
    //*/

    // Clean up
    hipFreeArray(d_data_array);
    hipDestroyTextureObject(d_data_txt);
    hipFree(dev_phis);

    if (data_on_cpu)
    {
        if (dev_g != 0)
            hipFree(dev_g);
        if (dev_f != 0)
            hipFree(dev_f);
    }
    else if (params->geometry == parameters::CONE)
    {
        applyPolarWeight <<< dimGrid_g, dimBlock_g >>> (dev_g, N_g, T_g, startVal_g);
        cudaStatus = hipDeviceSynchronize();
    }

    return true;
}
